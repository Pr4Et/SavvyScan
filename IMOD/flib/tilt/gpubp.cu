#include "hip/hip_runtime.h"
/*
 *  gpubp.cu -- Kernel and C code for CUDA-based backprojection, reprojection
 *               and Fourier filtering
 *
 *  Author: David Mastronarde   email: mast@colorado.edu
 *
 *  Copyright (C) 2009 by Boulder Laboratory for 3-Dimensional Electron
 *  Microscopy of Cells ("BL3DEMC") and the Regents of the University of 
 *  Colorado.  See dist/COPYRIGHT for full copyright notice.
 *
 *  $Id$
 */
#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "b3dutil.h"
#include "imodconfig.h"

#ifdef F77FUNCAP
#define gpuavailable GPUAVAILABLE
#define gpuallocarrays GPUALLOCARRAYS
#define gpuloadproj GPULOADPROJ
#define gpushiftproj GPUSHIFTPROJ
#define gpubpxtilt GPUBPXTILT
#define gpubpnox GPUBPNOX
#define gpubplocal GPUBPLOCAL
#define gpuloadlocals GPULOADLOCALS
#define gpuloadfilter GPULOADFILTER
#define gpufilterlines GPUFILTERLINES
#define gpureproject GPUREPROJECT
#define gpureprojlocal GPUREPROJLOCAL
#define gpureprojoneslice GPUREPROJONESLICE
#define gpudone GPUDONE
#else
#define gpuavailable gpuavailable_
#define gpuallocarrays gpuallocarrays_
#define gpuloadproj gpuloadproj_
#define gpushiftproj gpushiftproj_
#define gpubpxtilt gpubpxtilt_
#define gpubpnox gpubpnox_
#define gpubplocal gpubplocal_
#define gpuloadlocals gpuloadlocals_
#define gpuloadfilter gpuloadfilter_
#define gpufilterlines gpufilterlines_
#define gpureproject gpureproject_
#define gpureprojlocal gpureprojlocal_
#define gpureprojoneslice gpureprojoneslice_
#define gpudone gpudone_
#endif

#if CUDA_VERSION >= 4000
#define HAS_LAYERS
#else
#define hipArrayLayered 0
#endif

#ifdef __cplusplus
extern "C" {
  int gpuavailable(int *nGPU, float *memory, int *maxTex2D, int *maxTexLayer, 
                   int *maxTex3D, int *debug);
  int gpuallocarrays(int *width, int *nyout, int *nxProjPad, int *nyProj,
                     int *nplanes, int *nviews, int *numWarps, int *numDelz,
                     int *nfilt, int *nreproj, int *firstNpl, int *lastNpl, int *use3D);
  int gpubpnox(float *slice, float *lines, float *sinBeta, float *cosBeta,
               int *nxprj,
               float *xcenIn, float *xcen, float *ycen, float *edgefill);
  int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart);
  int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, 
                  int *loadStart);
  int gpubpxtilt(float *slice, float *sinBeta, float *cosBeta, float *sinAlpha,
                 float *cosAlpha, float *xzfac, float *yzfac, int *nxprj,
                 int *nyProj, float *xcenIn, float *xcen, float *ycen,
                 int *lslice, float *centerSlice, float *edgefill);
  int gpubplocal(float *slice, int *lslice, int *nxwarp, int *nywarp,
                 int *ixswarp, int *iyswarp, int *idxwarp, int *idywarp,
                 int *nxprj, float *xcen, float *xcenIn, float *delxx,
                 float *ycen, float *centerSlice, float *edgefill);
  int gpuloadfilter(float *lines);
  int gpuloadlocals(float *packed, int *numWarps);
  int gpufilterlines(float *lines, int *lslice, int *filterSet);
  int gpureproject(float *lines, float *sinBeta, float *cosBeta, float *sinAlpha, 
                   float *cosAlpha, float *xzfac, float *yzfac, float *delz,
                   int *lsStart, int *lsEnd, int *ithick,
                   float *xcen, float *xcenPdelxx, int *minXreproj, 
                   float *xprjOffset, float *ycen, int *minYreproj,
                   float *yprjOffset, float *centerSlice, int *ifalpha, 
                   float *pmean);
  int gpureprojoneslice(float *slice, float *lines, float *sinBeta, float *cosBeta,
                        float *ycen, int *numproj, float *pmean);
  int gpureprojlocal
  (float *lines, float *sinBeta, float *cosBeta, float *sinAlpha, float *cosAlpha,
   float *xzfac, float *yzfac, int *nxwarp, int *nywarp, int *ixswarp, 
   int *iyswarp, int *idxwarp, int *idywarp, float *warpDelz, int *nWarpDelz, 
   float *dxWarpDelz,float *xprojMin,float *xprojMax, int *lsStart, int *lsEnd,
   int *ithick, int *iview, float *xcen, float *xcenIn, float *delxx, 
   int *minXload, float *xprjOffset, float *ycenAdj, float *yprjOffset,
   float *centerSlice, float *pmean);
  void gpudone();
}
#endif

static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1);
static int testReportErr(const char *mess);
static int loadBetaInvertCos(float *cosBeta, float *sinBeta, float *costmp,
                             int num);
static int synchronizeCopySlice(float *devslc, int pitch, float *slice,
                                int width, int numLines);
static void pflush(const char *format, ...);
static void pflerr(const char *format, ...);
static void allocerr(const char *mess, int *nplanes, int *firstNpl,
                     int *lastNpl, int ifcuda);



// Offsets to positions in constant array
// For some reason 6 separate arrays did not work for xtilt case
// 7 arrays in 65536 bytes would allow 2340
#define DELTA_OFS  2200
#define MAX_TABLE (6 * DELTA_OFS)
__constant__ float tables[MAX_TABLE];
__constant__ int rpNumz[DELTA_OFS];

#define COSOFS 0
#define SINOFS (1 * DELTA_OFS)
#define CALOFS (2 * DELTA_OFS)
#define SALOFS (3 * DELTA_OFS)
#define XZFOFS (4 * DELTA_OFS)
#define YZFOFS (5 * DELTA_OFS)
#define INVOFS (2 * DELTA_OFS)
#define SINVOFS (3 * DELTA_OFS)

// Definitions for accessing the local alignments arrays with texture calls
#define F11IND 0.f
#define F21IND 1.f
#define F12IND 2.f
#define F22IND 3.f
#define F13IND 4.f
#define F23IND 5.f
#define CAIND 6.f
#define SAIND 7.f
#define CBIND 8.f
#define SBIND 9.f
#define XZFIND 10.f
#define YZFIND 11.f


// declare texture reference for 2D float textures
texture<float, 2, hipReadModeElementType> projtex2D;
texture<float, 3, hipReadModeElementType> projtex3D;
#ifdef HAS_LAYERS
texture<float, hipTextureType2DLayered> projtexLayer;
#endif
texture<float, 2, hipReadModeElementType> localtex;
texture<float, 2, hipReadModeElementType> rpSlicetex;
texture<float, 2, hipReadModeElementType> pfactex;
texture<float, 2, hipReadModeElementType> delztex;

// Static variables for device arrays
static float *devSlice = NULL;
static hipArray* devProj = NULL;
static float *devXprojFix = NULL;
static float *devXprojZ = NULL;
static float *devYprojFix = NULL;
static float *devYprojZ = NULL;
static hipArray *devLocalData = NULL;
static hipArray *devLocalPfac = NULL;
static hipArray *devDelz = NULL;
static float *devRadialFilt = NULL;
static float *devFFT = NULL;
static hipArray *devRpSlice = NULL;
static float *devReproj = NULL;

// Other static variables
static hipfftHandle sForwardPlan = 0, sInversePlan = 0;
static int sMaxGflopsDevice = -1;
static int sDeviceSelected = 0;
static size_t sSlicePitch;
static size_t sReprojPitch;
static size_t sLocalPitch;
static int sSliceThick, sSliceWidth, sNumViews, sNumProjPlanes;
static int sLsliceFirst, sNumLoadedPlanes, sNxPlane, sNyPlane, sNumFilts;
static int sCopyFilteredOK = 0;
static int *sPlaneLoaded;
static int sUse3dTexture;

/*
 *  SETUP/SHUTDOWN ROUTINES
 */

/*
 * Test whether a GPU is available, either a GPU of the given number if nGPU is
 * > 0, or the one with the best processing rate if nGPU is 0, and return the
 * memory in bytes.  Return value is 1 for success, 0 for failure.
 */
int gpuavailable(int *nGPU, float *memory, int *maxTex2D, int *maxTexLayer, int *maxTex3D,
                 int *debug)
{
  int current_device = 0;
  int device_count = 0;
  int totalCores, ind;
  float gflops;
  struct hipDeviceProp_t device_properties, best_properties;

  // The Mac mini comes through with a clock rate of 0 so allow a 0 product
  float max_gflops = -1.;
  *memory = 0;
  hipGetDeviceCount( &device_count );
  if (*debug) {
#if CUDA_VERSION >= 3000
    int version, version2;
    hipRuntimeGetVersion(&version2);
    hipDriverGetVersion(&version);
    pflush("CUDA version - driver: %d.%02d  runtime: %d.%02d\n", version / 1000,
           version % 1000, version2 / 1000, version2 % 1000);
#endif
    pflush("Device count = %d\n", device_count);
  }
  if (*nGPU != 0) {
    if (*nGPU < 0 || *nGPU > device_count) {
      pflush("The requested GPU number, %d, is out of range; there are only %d devices\n",
             *nGPU, device_count);
      return 0;
    }
    current_device = *nGPU - 1;
    device_count = *nGPU;
  }
  for (; current_device < device_count; current_device++) {
    if (hipGetDeviceProperties( &device_properties, current_device)
        != hipSuccess) {
      pflerr("Error returned from trying to get properties of GPU device %d",
               current_device);
      return 0;
    }
    totalCores = totalCudaCores(device_properties.major, device_properties.minor,
                                device_properties.multiProcessorCount);
    if (*debug)
      pflush("Device %d (%s): cores %d  cr %d  major %d minor %d  mem %.0f\n",
             current_device, device_properties.name, 
             totalCores, device_properties.clockRate,
             device_properties.major, device_properties.minor,
             (float)device_properties.totalGlobalMem);
    gflops = totalCores * (float)device_properties.clockRate;

    // This is the best place to catch that the GPU is too old for the version
#ifdef HAS_LAYERS
    if (device_properties.major < 2) {
      if (*debug)
        pflush("The compute level of this GPU is only %d.%d and it cannot be used\n"
               "   with an IMOD package built with CUDA 4 or higher\n", 
               device_properties.major, device_properties.minor);
      gflops = -999.;
    }
#endif

    // Exclude emulation mode (?) which shows up on the Mac
    if( gflops > max_gflops && device_properties.major != 9999) {
      max_gflops = gflops;
      sMaxGflopsDevice = current_device;
      best_properties = device_properties;
    }
  }
    
  if (sMaxGflopsDevice >= 0) {
    *memory = best_properties.totalGlobalMem;
    for (ind = 0; ind < 3; ind++) {
#ifdef HAS_LAYERS
      maxTexLayer[ind] = best_properties.maxTexture2DLayered[ind];
#else
      maxTexLayer[ind] = 0;
#endif
      maxTex3D[ind] = best_properties.maxTexture3D[ind];
      if (ind < 2)
        maxTex2D[ind] = best_properties.maxTexture2D[ind];
    }
    return 1;
  }
  return 0;
}

/*
 * Allocate all needed arrays on the GPU.  Allocate a reconstructed slice or
 * reprojected line array of width x nyout, an array for nplanes of input data 
 * each with nyProj lines of length nxProjPad, and local alignment arrays if
 * numWarps > 0.  If numDelz > 0, this indicates reprojection with local
 * alignments and causes local projection factor arrays to be allocated for
 * nplanes lines, allocation of a CUDA array for those factors too, and 
 * allocation of an array of numDelz x nplanes for warpDelz values.  If 
 * nfilt > 0, also allocate arrays for line filtering with nfilt sets of
 * filters.  If nreproj > 0, allocate separate arrays for reprojecting one
 * slice while still doing regular backprojection arrays.
 */
int gpuallocarrays(int *width, int *nyout, int *nxProjPad, int *nyProj,
                   int *nplanes, int *nviews, int *numWarps, int *numDelz,
                   int *nfilt, int *nreproj, int *firstNpl, int *lastNpl, int *use3D)
{
  size_t pitch1, pitch2, pitch3, memTot;
  int nlines;
  hipError_t err;

  sUse3dTexture = *use3D;
  sSliceWidth = *width;
  sSliceThick = *nyout;    // Only good for backprojection!
  sNumViews = *nviews;
  sNumProjPlanes = *nplanes;
  sNxPlane = *nxProjPad;
  sNyPlane = *nyProj;
  sNumFilts = *nfilt;

  if (sMaxGflopsDevice < 0)
    return 1;
  if (!sDeviceSelected && hipSetDevice(sMaxGflopsDevice) != hipSuccess) {
    allocerr("Error selecting GPU device", nplanes, firstNpl, lastNpl, 1);
    return 1;
  }
  sDeviceSelected = 1;

  if (sNumViews > DELTA_OFS) {
    allocerr("Too many views for the constant memory available on the GPU\n",
             nplanes, firstNpl, lastNpl, 0);
    return 1;
  }

  if (sUse3dTexture < 0 && hipArrayLayered == 0) {
    pflerr("Bad parameter: texture type -1 was specified, but no layered textures\n"
           "   are available with this version of IMOD\n");
    return 1;
  }

  // Allocate memory for slice or reprojected lines on device
  size_t sizetmp = sSliceWidth * sizeof(float);
  if (hipMallocPitch((void **)&devSlice, &sSlicePitch, sizetmp, sSliceThick) != 
      hipSuccess) {
    allocerr("Failed to allocate slice array on GPU device", nplanes, 
             firstNpl, lastNpl, 1);
    return 1;
  }
  //pflush("reproj array size %d %d\n", sSliceWidth, sSliceThick);

  // Allocate memory for projection lines or slices to reproject
  hipChannelFormatDesc projDesc = hipCreateChannelDesc
    (32, 0, 0, 0, hipChannelFormatKindFloat);
  if (!sUse3dTexture)
    err = hipMallocArray(&devProj, &projDesc, sNxPlane, sNyPlane * sNumProjPlanes);
  else
    err = hipMalloc3DArray(&devProj, &projDesc, make_hipExtent 
                            (sNxPlane, sNyPlane, sNumProjPlanes)
#ifdef HAS_LAYERS
                            , sUse3dTexture > 0 ? 0 : hipArrayLayered
#endif
                            );
  if (err != hipSuccess) {
    pflush("malloc %d %d %d %d\n", sNxPlane, sNyPlane, sNumProjPlanes,
           sNyPlane * sNumProjPlanes);
    allocerr("Failed to allocate projection array on GPU device", nplanes, 
             firstNpl, lastNpl, 1);
    return 1;
  }
  memTot = sizetmp * sSliceThick + 4 * sNxPlane * sNyPlane * sNumProjPlanes;
  //pflush("input slice array size %d %d %d\n", sNxPlane, sNyPlane, sNumProjPlanes);

  // set texture parameters

  
  // Bind the array to the texture
  if (sUse3dTexture > 0) {
    projtex3D.addressMode[0] = hipAddressModeClamp;
    projtex3D.addressMode[1] = hipAddressModeClamp;
    projtex3D.filterMode = hipFilterModeLinear;
    projtex3D.normalized = false;
    err = hipBindTextureToArray(projtex3D, devProj, projDesc);
  } else if (sUse3dTexture < 0) {
#ifdef HAS_LAYERS
    projtexLayer.addressMode[0] = hipAddressModeClamp;
    projtexLayer.addressMode[1] = hipAddressModeClamp;
    projtexLayer.filterMode = hipFilterModeLinear;
    projtexLayer.normalized = false;
    err = hipBindTextureToArray(projtexLayer, devProj, projDesc);
#endif
  } else {
    projtex2D.addressMode[0] = hipAddressModeClamp;
    projtex2D.addressMode[1] = hipAddressModeClamp;
    projtex2D.filterMode = hipFilterModeLinear;
    projtex2D.normalized = false;
    err = hipBindTextureToArray(projtex2D, devProj, projDesc);
  }
  if (err != hipSuccess) {
    allocerr("Failed to bind projection array to texture", nplanes, firstNpl, lastNpl, 1);
    return 1;
  }

  if (sNumProjPlanes > 1) {
    sPlaneLoaded = (int *)malloc(sNumProjPlanes * sizeof(int));
    if (!sPlaneLoaded) {
      allocerr("Failed to malloc little array sPlaneLoaded\n", nplanes,
               firstNpl, lastNpl, 0);
      return 1;
    }
  }

  // Get arrays for reprojection of one slice
  if (*nreproj) {
    if (hipMallocArray(&devRpSlice, &projDesc, sSliceWidth, sSliceThick) !=
        hipSuccess) {
      allocerr("Failed to allocate slice array for reprojection on GPU device",
               nplanes, firstNpl, lastNpl, 1);
      return 1;
    }
    if (hipBindTextureToArray(rpSlicetex, devRpSlice, projDesc) != hipSuccess) {
      allocerr("Failed to bind reprojection slice array to texture", nplanes, 
               firstNpl, lastNpl, 1);
      return 1;
    }
    rpSlicetex.addressMode[0] = hipAddressModeClamp;
    rpSlicetex.addressMode[1] = hipAddressModeClamp;
    rpSlicetex.filterMode = hipFilterModeLinear;
    rpSlicetex.normalized = false;
  
    if (hipMallocPitch((void **)&devReproj, &sReprojPitch, 
                        (size_t)(sNxPlane * sizeof(float)), *nreproj) != hipSuccess) {
      allocerr("Failed to allocate reprojected line array on GPU device", 
               nplanes, firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += 4 * sSliceWidth * sSliceThick + sNxPlane * *nreproj;
  }

  // Get arrays for local proj factors
  if (*numWarps > 0) {
    nlines = sNyPlane;

    // Adjust and allocate for reprojection
    if (*numDelz) {
      nlines = sNumProjPlanes;
      sizetmp = sNxPlane * sizeof(float);

      if (hipMallocArray(&devLocalPfac, &projDesc, sNxPlane, 4 * nlines) != hipSuccess)
        {
          allocerr("Failed to allocate local factor texture array on GPU device",
                   nplanes, firstNpl, lastNpl, 1);
          return 1;
        }
      //pflush("local factor texture  %d %d\n", sNxPlane, 4 * nlines);
      
      pfactex.filterMode = hipFilterModePoint;
      pfactex.normalized = false;
      if (hipBindTextureToArray(pfactex, devLocalPfac, projDesc) != hipSuccess) {
        allocerr("Failed to bind local factor arrays to texture", nplanes, 
                 firstNpl, lastNpl, 1);
        return 1;
      }
      if (hipMallocArray(&devDelz, &projDesc, *numDelz, nlines) != hipSuccess) {
        allocerr("Failed to allocate warpDelz texture array on GPU device",
                 nplanes, firstNpl, lastNpl, 1);
        return 1;
      }
      //pflush("warpdelz texture  %d %d\n", *numDelz, nlines);
      delztex.filterMode = hipFilterModePoint;
      delztex.normalized = false;
      if (hipBindTextureToArray(delztex, devDelz, projDesc) != hipSuccess) {
        allocerr("Failed to bind warpDelz array to texture", nplanes, 
                 firstNpl, lastNpl, 1);
        return 1;
      }
      memTot += 4 * nlines * (4 * sNxPlane + *numDelz);
    }

    // Allocate the arrays always used for local data
    if (hipMallocPitch((void **)&devXprojFix, &pitch1, sizetmp, nlines) != hipSuccess ||
        hipMallocPitch((void **)&devXprojZ, &pitch2, sizetmp, nlines) != hipSuccess ||
        hipMallocPitch((void **)&devYprojFix, &pitch3, sizetmp, nlines) != hipSuccess ||
        hipMallocPitch((void **)&devYprojZ, &sLocalPitch, sizetmp, nlines) != 
        hipSuccess  || hipMallocArray(&devLocalData, &projDesc, *numWarps * sNumViews, 
                                        12) != hipSuccess) {
      allocerr("Failed to allocate local factor arrays on GPU device", nplanes,
               firstNpl, lastNpl, 1);
      return 1;
    }
    /* pflush("xdevYprojFix pitches  %d %d    localdata %d\n", sNxPlane, nlines,
     *numWarps * sNumViews); */
    if (pitch2 != pitch1 || pitch3 != pitch1 || sLocalPitch != pitch1) {
      allocerr("Array pitches for local GPU arrays do NOT match\n", nplanes,
               firstNpl, lastNpl, 0);
      return 1;
    }

    localtex.filterMode = hipFilterModePoint;
    localtex.normalized = false;
    if (hipBindTextureToArray(localtex, devLocalData, projDesc) != hipSuccess) {
      allocerr("Failed to bind local factor arrays to texture", nplanes, 
               firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += 4 * sizetmp * nlines + 48 * *numWarps * sNumViews;
  }

  // Get arrays for radial filtering
  if (sNumFilts > 0 || sNumProjPlanes > 1) {
    sizetmp = sNxPlane * sNyPlane * sizeof(float);
    if (hipMalloc((void **)&devFFT, sizetmp)  != hipSuccess ||
        (sNumFilts > 0 && hipMalloc((void **)&devRadialFilt, sizetmp * sNumFilts) 
         != hipSuccess)) {
      allocerr("Failed to allocate GPU arrays for radial filtering", nplanes,
               firstNpl, lastNpl, 1);
      return 1;
    }
    memTot += (1 + sNumFilts) * sizetmp;
  }

  pflush("Allocated %4d MB for arrays (including %d input planes) on the GPU\n"
         , (memTot + 512*1024)/(1024*1024), sNumProjPlanes);
  return 0;
}

// Routine to free all allocated resources
void gpudone()
{
  hipFree(devSlice);
  hipFreeArray(devProj);
  hipFree(devXprojFix);
  hipFree(devXprojZ);
  hipFree(devYprojFix);
  hipFree(devYprojZ);
  hipFreeArray(devLocalData);
  hipFreeArray(devLocalPfac);
  hipFreeArray(devDelz);
  hipFree(devFFT);
  hipFree(devRadialFilt);
  hipFree(devReproj);
  hipFreeArray(devRpSlice);
  if (sForwardPlan)
    hipfftDestroy(sForwardPlan);
  if (sInversePlan)
    hipfftDestroy(sInversePlan);
  devSlice = NULL;
  devProj = NULL;
  devXprojFix = NULL;
  devXprojZ = NULL;
  devYprojFix = NULL;
  devYprojZ = NULL;
  devLocalData = NULL;
  devLocalPfac = NULL;
  devDelz = NULL;
  devFFT = NULL;
  devRadialFilt = NULL;
  devReproj = NULL;
  devRpSlice = NULL;
  sForwardPlan = 0;
  sInversePlan = 0;
}

/*
 * ROUTINES FOR LOADING/MAINTAINING STACK OF PLANES ON GPU
 */ 

// Function to shift existing data in preparation for loading new data starting
// in position loadStart (numbered from 1) and with starting slice number
// lsliceStart
int gpushiftproj(int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int shift, shiftStart, numToShift, todo, dstY, srcY;
  size_t sizetmp = sNxPlane * sizeof(float);
  hipMemcpy3DParms cpyParms = {0};
  hipMemcpy3DParms tmpParms = {0};
  if (startm1 > 0) {
    if (checkProjLoad(numPlanes, lsliceStart, startm1))
      return 1;

    // Copy data down without overlap if it goes into occupied planes
    if (startm1 < sNumLoadedPlanes) {
      shift = sNumLoadedPlanes - startm1;
      numToShift = startm1;
      shiftStart = 0;
      if (sUse3dTexture) {
          tmpParms.dstPos = make_hipPos(0, 0, 0);
          tmpParms.dstPtr = make_hipPitchedPtr(devFFT, sNxPlane * sizeof(float), 
                                                sNxPlane, sNyPlane);
          tmpParms.srcArray = devProj;
          tmpParms.extent = make_hipExtent(sNxPlane, sNyPlane, 1);
          tmpParms.kind = hipMemcpyDeviceToDevice;
          cpyParms.srcPos = make_hipPos(0, 0, 0);
          cpyParms.srcPtr = make_hipPitchedPtr(devFFT, sNxPlane * sizeof(float),
                                                sNxPlane, sNyPlane);
          cpyParms.dstArray = devProj;
          cpyParms.extent = make_hipExtent(sNxPlane, sNyPlane, 1);
          cpyParms.kind = hipMemcpyDeviceToDevice;
      }

      // Loop on the planes or sets of planes to copy
      while (numToShift > 0) {
        if (!sUse3dTexture) {
          todo = shift;
          if (todo > numToShift)
            todo = numToShift;
          dstY = shiftStart * sNyPlane;
          srcY = dstY + shift * sNyPlane;
          //pflush("Copying down %d\n", todo);
          if (hipMemcpy2DArrayToArray(devProj, 0, dstY, devProj, 0, srcY,
                                       sizetmp, todo * sNyPlane,
                                       hipMemcpyDeviceToDevice) != hipSuccess){
            pflerr("Error copying segment of projection array down");
            sNumLoadedPlanes = 0;
            return 1;
          }
        } else {

          // Sadly this can only copy one plane at a time at least for the layered array
          todo = 1;
          tmpParms.srcPos = make_hipPos(0, 0, shiftStart + shift);
          if (hipMemcpy3D(&tmpParms) != hipSuccess) {
            pflerr("Error copying plane %d of projection array to devFFT", 
                   shiftStart + shift);
            sNumLoadedPlanes = 0;
            return 1;
          }
          cpyParms.dstPos = make_hipPos(0, 0, shiftStart);
          if (hipMemcpy3D(&cpyParms) != hipSuccess) {
            pflerr("Error copying devFFT to plane %d of projection array", 
                   shiftStart);
            sNumLoadedPlanes = 0;
            return 1;
          }
        }
        numToShift -= todo;
        shiftStart += todo;
      }
    }
  }
  sNumLoadedPlanes = startm1;
  sLsliceFirst = *lsliceStart - startm1;

  /*pflush("Initializing array num %d  first %d  loaded %d\n", sNumProjPlanes, 
    sLsliceFirst, sNumLoadedPlanes); */
  // Initialize array for keeping track of copied planes, and enable copying
  for (todo = 0; todo < sNumProjPlanes; todo++)
    sPlaneLoaded[todo] = todo < sNumLoadedPlanes ? 1 : 0;
  sCopyFilteredOK = 1;
  return 0;
}

// Function to load numPlanes planes of input data, starting in position
// loadStart (numbered from 1) and with starting slice number lsliceStart
int gpuloadproj(float *lines, int *numPlanes, int *lsliceStart, int *loadStart)
{
  int startm1 = *loadStart - 1;
  int todo, dstY, numCopy = 0;
  hipMemcpy3DParms cpyParms = {0};
  hipError_t err;

  if (startm1 > 0 && checkProjLoad(numPlanes, lsliceStart, startm1)) {
    sCopyFilteredOK = 0;
    return 1;
  }

  // Check for valid load
  if (startm1 + *numPlanes > sNumProjPlanes) {
    pflush("Trying to load past end of projection array\n");
    sCopyFilteredOK = 0;
    sNumLoadedPlanes = 0;
    return 1;
  }
  
  // Find the number to copy by the last plane not already loaded
  if (sCopyFilteredOK) {
    for (todo = startm1; todo < startm1 + *numPlanes; todo++)
      if (!sPlaneLoaded[todo])
        numCopy = todo + 1 - startm1;
  }
  sCopyFilteredOK = 0;

  // Finally do the load
  //if (numCopy) pflush("Loading %d planes\n", numCopy);
  if (numCopy) {
    if (sUse3dTexture) {
    
      cpyParms.srcPos = make_hipPos(0, 0, 0);
      cpyParms.dstPos = make_hipPos(0, 0, startm1);
      cpyParms.srcPtr = make_hipPitchedPtr(lines, sNxPlane * sizeof(float), sNxPlane, 
                                            sNyPlane);
      cpyParms.dstArray = devProj;
      cpyParms.extent = make_hipExtent(sNxPlane, sNyPlane, numCopy);
      cpyParms.kind = hipMemcpyHostToDevice;
      err = hipMemcpy3D(&cpyParms);
    } else {
      dstY = startm1 * sNyPlane;
      todo = numCopy * sNyPlane * sNxPlane * 4;
      err = hipMemcpyToArray(devProj, 0, dstY, lines, todo, hipMemcpyHostToDevice);
    }
    if (err != hipSuccess) {
      pflerr("Failed to copy projection array to device");
      sNumLoadedPlanes = 0;
      return 1;
    }
  }
  sNumLoadedPlanes = startm1 + *numPlanes;
  sLsliceFirst = *lsliceStart - startm1;
  return 0;
}

// Function to do initial check on parameters in load/shift calls
static int checkProjLoad(int *numPlanes, int *lsliceStart, int startm1)
{
  if (!sNumLoadedPlanes) {
    pflush("Trying to load into higher planes when none are loaded\n");
    return 1;
  }
  if (sLsliceFirst + sNumLoadedPlanes != *lsliceStart) {
    pflush("Starting slice %d does not match first slice %d + num loaded %d"
            "\n", *lsliceStart, sLsliceFirst, sNumLoadedPlanes);
    sNumLoadedPlanes = 0;
    return 1;
  }
  if (startm1 > sNumLoadedPlanes) {
    pflush("Starting plane %d is past number loaded %d\n", startm1+1, 
           sNumLoadedPlanes);
    sNumLoadedPlanes = 0;
    return 1;
  }
  return 0;
}

/*
 * ROUTINES FOR RADIAL FILTERING OF INPUT LINES
 */

// Kernel to multiply the FFT by the filter
__global__ void filterFFT(float *FFT, float *filter, int nxProjPad, int nviews, 
                          float scale)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < nviews && j < nxProjPad)
    FFT[i * nxProjPad + j] *= filter[i * nxProjPad + j] * scale;
}

// Function to load the filter lines into the array and generate plans
int gpuloadfilter(float *lines)
{
  size_t sizetmp = sNxPlane * sNumViews * sNumFilts * sizeof(float);
  if (hipMemcpy(devRadialFilt, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy radial filters to GPU array");
    gpudone();
    return 1;
  }
  if (hipfftPlan1d(&sForwardPlan, sNxPlane - 2, HIPFFT_R2C, sNumViews) != 
      HIPFFT_SUCCESS || hipfftPlan1d(&sInversePlan, sNxPlane - 2, HIPFFT_C2R, 
                                   sNumViews) != HIPFFT_SUCCESS) {
    pflush("Failed to generate a plan for CUFFT\n");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to filter the set of input lines
int gpufilterlines(float *lines, int *lslice, int *filterSet)
{
  int ind, blockX = 16;
  size_t sizetmp = sNxPlane * sNumViews * sizeof(float);
  float scale = 1.f / (sNxPlane - 2);
  hipMemcpy3DParms cpyParms = {0};
  hipError_t err;
  if (hipMemcpy(devFFT, lines, sizetmp, hipMemcpyHostToDevice) !=
      hipSuccess) {
    pflerr("Failed to copy lines to GPU array for radial filtering");
    return 1;
  }
  if (hipfftExecR2C(sForwardPlan, devFFT, (hipfftComplex *)devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in forward FFT on GPU\n");
    return 1;
  }
  
  // Filter!!!
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sNxPlane + blockSize.x - 1) / blockSize.x, 
                (sNumViews + blockSize.y - 1) / blockSize.y, 1);

  filterFFT<<<gridSize, blockSize>>>
    (devFFT, devRadialFilt + (*filterSet - 1) * sNxPlane * sNumViews, sNxPlane, 
     sNumViews, scale);
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflerr("Error executing threads for filtering"); 
    return 1;
  }
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after filtering");
    return 1;
  }

  if (hipfftExecC2R(sInversePlan, (hipfftComplex *)devFFT, devFFT) != 
      HIPFFT_SUCCESS) {
    pflush("Failure in inverse FFT on GPU\n");
    return 1;
  }
  if (hipMemcpy(lines, devFFT, sizetmp, hipMemcpyDeviceToHost) !=
      hipSuccess) {
    pflerr("Failed to copy radial filtered lines back from GPU array");
    return 1;
  }
  
  // If copying is OK and it is a slice in needed range, copy it to proj
  if (sCopyFilteredOK) {
    ind = *lslice - sLsliceFirst;
    if (ind >= 0 && ind < sNumProjPlanes) {
      //pflush("Copying %d to plane %d\n", *lslice,ind);
      if (sUse3dTexture) {
        cpyParms.srcPos = make_hipPos(0, 0, 0);
        cpyParms.dstPos = make_hipPos(0, 0, ind);
        cpyParms.srcPtr = make_hipPitchedPtr(devFFT, sNxPlane * sizeof(float), sNxPlane,
                                              sNyPlane);
        cpyParms.dstArray = devProj;
        cpyParms.extent = make_hipExtent(sNxPlane, sNyPlane, 1);
        cpyParms.kind = hipMemcpyDeviceToDevice;
        if (hipMemcpy3D(&cpyParms) == hipSuccess)
          sPlaneLoaded[ind] = 1;
      } else {
        if (hipMemcpyToArray(devProj, 0, ind * sNumViews, devFFT, sizetmp,
                            hipMemcpyDeviceToDevice) == hipSuccess)
          sPlaneLoaded[ind] = 1;
      }
    }
  }
  return 0;
}

/*
 * ROUTINES FOR SIMPLE BACK-PROJECTION (NO X-AXIS TILT, ETC)
 */

// Kernel for simple back-projection with testing at ends of lines
__global__ void bpNoXtTest(float *slice, int pitch, int jbase, int iwide,
                             int nxProj, int ithick, int nviews, 
                             float xcenIn, float xcenOut, float ycenOut, 
                             float edgefill)
{
  float cosBeta, sinBeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cosBeta = tables[iv+COSOFS];
      sinBeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycenOut) * sinBeta + xcenIn;
      xp =  zpart + (j + 1 - xcenOut) * cosBeta - 0.5f;
      //if (i == 150) printf("%d %d %d  %d  %d  %.2f  %.2f\n", iv, j, i, jlft, jrt, zpart + (1 - xcenOut) * cosBeta - 0.5f, zpart + (nxProj - xcenOut) * cosBeta - 0.5f);
      if (xp >= 0.5 && xp <= nxProj - 0.5) {
        kproj = iv + 0.5f;
        sum += tex2D(projtex2D, xp, kproj);
      } else {
        sum += edgefill;
      }
    }
    slice[i * pitch + j] = sum;
  }
}

// Kernel for simple back-projection with no testing
__global__ void bpNoXtFast(float *slice, int pitch, int jbase, int iwide,
                             int ithick, int nviews, 
                             float xcenIn, float xcenOut, float ycenOut)
{
  float cosBeta, sinBeta, zpart, kproj, xp;
  float sum = 0.;
  int iv;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    for (iv = 0; iv < nviews; iv++) {
      cosBeta = tables[iv+COSOFS];
      sinBeta = tables[iv+SINOFS];
      zpart = (i + 1 - ycenOut) * sinBeta + xcenIn;
      kproj = iv + 0.5f;
      xp =  zpart + (j + 1 - xcenOut) * cosBeta - 0.5f;
      sum += tex2D(projtex2D, xp, kproj);
    }
    slice[i * pitch + j] = sum;
  }
}

// Function to run simple backprojection
int gpubpnox(float *slice, float *lines, float *sinBeta, float *cosBeta,
             int *nxProj, float *xcenIn, float *xcenOut, float *ycenOut,
             float *edgefill)
{
  size_t sizetmp = sizeof(float) * sNxPlane * sNumViews;
  float cosinv[DELTA_OFS];
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp;
  int i, blockX = 16;

  if (sUse3dTexture) {
    pflush("GPU problem: Program called gpubpnox after setting up to use texture type "
           "%d\n", sUse3dTexture);
    return 1;
  }
  if (loadBetaInvertCos(cosBeta, sinBeta, cosinv, sNumViews))
    return 1;

  // Copy projections
  if (hipMemcpyToArray(devProj, 0, 0, lines, sizetmp, hipMemcpyHostToDevice)
      != hipSuccess) {
    pflerr("Failed to copy projection array to device");
    return 1;
  }

  // Find limits of region that needs no testing
  jlftmax = 1;
  jrtmin = sSliceWidth;
  for (iv = 0; iv < sNumViews; iv++) {
    for (i = 0; i <= sSliceThick - 1; i += sSliceThick - 1) {
      zpart = (i + 1 - *ycenOut) * sinBeta[iv] + *xcenIn;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcenOut;
      xrttmp = (*nxProj - zpart) * cosinv[iv] + *xcenOut;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcenOut) * cbet - 0.5f, zpart + (*nxProj - *xcenOut) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (sSliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0) {
    gridLeft = gridSize.x;
    gridRight = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, 0, sSliceWidth, *nxProj, sSliceThick, 
       sNumViews, *xcenIn, *xcenOut, *ycenOut, *edgefill);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpNoXtFast<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, blockX * gridLeft, sSliceWidth,
       sSliceThick, sNumViews, *xcenIn, *xcenOut, *ycenOut);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpNoXtTest<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, blockX * (gridLeft + gridFast), sSliceWidth, 
       *nxProj, sSliceThick, sNumViews, *xcenIn, *xcenOut, *ycenOut, *edgefill);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(devSlice, sSlicePitch, slice, sSliceWidth, 
                               sSliceThick));
    
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH X AXIS TILT AND/OR Z FACTORS
 */

// Kernel for BP with X-axis tilt/Z-factors and testing at ends of lines
#define BPXTTEST_START \
    for (iv = 0; iv < nviews; iv++) {    \
      cosBeta = tables[iv+COSOFS];         \
      sinBeta = tables[iv+SINOFS];         \
      sinAlpha = tables[iv+SALOFS];         \
      cosAlpha = tables[iv+CALOFS];         \
      zpart = yy * sinAlpha * sinBeta +          \
        zz * (cosAlpha * sinBeta + tables[iv+XZFOFS]) + xcenIn;         \
      yproj = yy * cosAlpha - zz * (sinAlpha - tables[iv+YZFOFS]) + centerSlice;   \
      xp =  zpart + xx * cosBeta - 0.5f;         \
      if (yproj >= 1. - ytol && yproj <= nyProj + ytol && xp >= 0.5 &&          \
          xp < nxProj - 0.5) {         \
        yproj = fmax(1.f, fmin((float)nyProj, yproj));         \
        jproj = min((int)yproj, nyProj - 1);         \
        fj = yproj - jproj;

#define BPXTTEST_END           \
      } else {         \
        sum += edgefill;         \
      }         \
    }

__global__ void bpXtiltTest(float *slice, int pitch, int jbase, int iwide,
                            int nxProj, int nyProj, int ithick, int nviews, 
                            float xcenIn, float xcenOut, float ycenOut, float yy,
                            float centerSlice, int lsliceBase, float edgefill, int use3D)
{
  float cosBeta, sinBeta, zpart, kproj, xp, zz, cosAlpha, sinAlpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
#ifdef HAS_LAYERS
  int jslice;
#endif
  float ytol = 3.05f;
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (j < iwide && i < ithick) {
    zz = (i + 1 - ycenOut);
    xx = (j + 1 - xcenOut);
#ifdef HAS_LAYERS
    if (use3D == 0) {
#endif
      BPXTTEST_START;
      kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
      sum += (1.f - fj) * tex2D(projtex2D, xp, kproj) + 
        fj * tex2D(projtex2D, xp, kproj + nviews);
      BPXTTEST_END;
#ifdef HAS_LAYERS
    } else {
      BPXTTEST_START;
      jslice = jproj - lsliceBase;
      kproj = iv + 0.5f;
      sum += (1.f - fj) * tex2DLayered(projtexLayer, xp, kproj, jslice) + 
        fj * tex2DLayered(projtexLayer, xp, kproj, jslice + 1);
      BPXTTEST_END;
    }
#endif
    slice[i * pitch + j] = sum;
  }
}

// Kernel for BP with X-axis tilt/Z-factors and no testing 
#define BPXTFAST_ALL \
      cosBeta = tables[iv+COSOFS];         \
      sinBeta = tables[iv+SINOFS];         \
      sinAlpha = tables[iv+SALOFS];         \
      cosAlpha = tables[iv+CALOFS];         \
      zpart = yy * sinAlpha * sinBeta +          \
        zz * (cosAlpha * sinBeta + tables[iv+XZFOFS]) + xcenIn;         \
      yproj = yy * cosAlpha - zz * (sinAlpha - tables[iv+YZFOFS]) + centerSlice;     \
      jproj = (int)yproj;         \
      fj = yproj - jproj;         \
      xp =  zpart + xx * cosBeta - 0.5f;

__global__ void bpXtiltFast(float *slice, int pitch, int jbase, int iwide, int ithick,
                            int nviews, float xcenIn, float xcenOut, float ycenOut,
                            float yy, float centerSlice, int lsliceBase, int use3D)
{
  float cosBeta, sinBeta, zpart, kproj, xp, zz, cosAlpha, sinAlpha, fj, yproj, xx;
  float sum = 0.;
  int iv, jproj;
#ifdef HAS_LAYERS
  int jslice;
#endif
  int j = blockIdx.x * blockDim.x + threadIdx.x + jbase;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick) {
    zz = (i + 1 - ycenOut);
    xx = (j + 1 - xcenOut);
#ifdef HAS_LAYERS
    if (use3D == 0) {
#endif
      for (iv = 0; iv < nviews; iv++) {
        BPXTFAST_ALL;
        kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
        sum += (1.f - fj) * tex2D(projtex2D, xp, kproj) + 
          fj * tex2D(projtex2D, xp, kproj + nviews);
      }
#ifdef HAS_LAYERS
    } else {
      for (iv = 0; iv < nviews; iv++) {
        BPXTFAST_ALL;
        jslice = jproj - lsliceBase;
        kproj = iv + 0.5f;
        sum += (1.f - fj) * tex2DLayered(projtexLayer, xp, kproj, jslice) + 
          fj * tex2DLayered(projtexLayer, xp, kproj, jslice + 1);
      }
    }
#endif
    slice[i * pitch + j] = sum;
  }
}

// Function to run back-projection with X-axis tilt/Z-factors
int gpubpxtilt(float *slice, float *sinBeta, float *cosBeta, 
               float *sinAlpha, float *cosAlpha, float *xzfac, float *yzfac,
               int *nxProj, int *nyProj, float *xcenIn, float *xcenOut, float *ycenOut,
               int *lslice, float *centerSlice, float *edgefill)
{
  int iv, jlft, jrt, jlftmax, jrtmin, gridLeft, gridFast, gridRight;
  float zpart, xlft, xrt, xlfttmp, xrttmp, yy, zz, yproj;
  int i, ytest, blockX = 16;
  float cosinv[DELTA_OFS];

  if (sUse3dTexture > 0) {
    pflush("GPU problem: Program called gpubpxtilt after setting up to use 3D textures"
           "\n"); 
    return 1;
  }

  if (loadBetaInvertCos(cosBeta, sinBeta, cosinv, sNumViews))
    return 1;

  // Copy alphas and z factors
  iv = sNumViews * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cosAlpha, iv, CALOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), sinAlpha, iv, SALOFS*4,
                            hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), xzfac, iv, XZFOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(tables), yzfac, iv, YZFOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }

  // Find limits of region that needs no testing.  Test every angle top & bot
  jlftmax = 1;
  jrtmin = sSliceWidth;
  yy = *lslice - *centerSlice;
  ytest = 0;
  for (iv = 0; iv < sNumViews; iv++) {
    for (i = 0; i <= sSliceThick - 1; i += sSliceThick - 1) {
      zz = (i + 1 - *ycenOut);
      zpart = yy * sinAlpha[iv] * sinBeta[iv] + zz * (cosAlpha[iv] * sinBeta[iv] +
                                                  xzfac[iv]) + *xcenIn;
      yproj = yy * cosAlpha[iv] - zz * (sinAlpha[iv] - yzfac[iv]) + *centerSlice;
      if (yproj < 1 || yproj > *nyProj - 1)
        ytest = 1;
      xlfttmp = (1. - zpart) * cosinv[iv] + *xcenOut;
      xrttmp = (*nxProj - zpart) * cosinv[iv] + *xcenOut;
      xlft = fmin(xlfttmp, xrttmp);
      xrt = fmax(xlfttmp, xrttmp);
      jlft = (int)ceilf(xlft);
      jrt = (int)ceilf(xrt) - 1;
      jlftmax = max(jlftmax, jlft);
      jrtmin = min(jrtmin, jrt);
      //printf("%d %f %d %.2f %d  %d  %.2f  %.2f\n", iv, cbet, i, zpart, jlft, jrt, zpart + (1 - *xcenOut) * cbet - 0.5f, zpart + (*nxProj - *xcenOut) * cbet - 0.5f);
    }
  }

  // Figure out grid sizes for left test, fast, and right test regions
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (sSliceThick + blockSize.y - 1) / blockSize.y, 1);

  gridLeft = (jlftmax - 1 + blockX - 1) / blockX;
  gridFast = jrtmin / blockX - gridLeft;
  if (gridFast <= 0 || ytest) {
    gridLeft = gridSize.x;
    gridRight = 0;
    gridFast = 0;
  } else
    gridRight = gridSize.x - (gridFast + gridLeft);

  if (gridLeft > 0) {
    gridSize.x = gridLeft;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, 0, sSliceWidth, *nxProj, *nyProj, sSliceThick, 
       sNumViews, *xcenIn, *xcenOut, *ycenOut, yy, *centerSlice, sLsliceFirst, *edgefill,
       sUse3dTexture);
    if (testReportErr("in left test region of backprojection"))
      return 1;
  }

  if (gridFast > 0) {
    gridSize.x = gridFast;
    bpXtiltFast<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, blockX * gridLeft, sSliceWidth, sSliceThick, sNumViews,
       *xcenIn, *xcenOut, *ycenOut, yy, *centerSlice, sLsliceFirst, sUse3dTexture);
    if (testReportErr("in no-test region of backprojection"))
      return 1;
  }

  if (gridRight > 0) {
    gridSize.x = gridRight;
    bpXtiltTest<<<gridSize, blockSize>>>
      (devSlice, sSlicePitch / 4, blockX * (gridLeft + gridFast), sSliceWidth, *nxProj,
       *nyProj, sSliceThick, sNumViews, *xcenIn, *xcenOut, *ycenOut, yy, *centerSlice, 
       sLsliceFirst, *edgefill, sUse3dTexture);
    if (testReportErr("in right test region of backprojection"))
      return 1;
  }

  return (synchronizeCopySlice(devSlice, sSlicePitch, slice, sSliceWidth,
                               sSliceThick));
}

/*
 * ROUTINES FOR BACK-PROJECTION WITH LOCAL ALIGNMENTS
 */

// Kernel for back-projection using local projection factors, testing as needed
#define BPLOCAL_START         \
      for (iv = 0; iv < nviews; iv++) {         \
        ind = iv * sLocalPitch + j;         \
        xp = xprojf[ind] + zz * xprojz[ind] - 0.5f;         \
        yproj = yprojf[ind] + zz * yprojz[ind];         \
        if (yproj >= lsliceBase - ytol && yproj <= lsliceLast + ytol &&          \
            xp >= 0.5f && xp < nxProj - 0.5f) {         \
          yproj = fmax((float)lsliceBase, fmin((float)lsliceLast, yproj));         \

#define BPLOCAL_END         \
      } else {         \
        sum += edgeFill;         \
      }         \
    }

__global__ void bpLocalTest(float *slice, int slPitch, float *xprojf, 
                            float *xprojz, float *yprojf, float *yprojz, 
                            int sLocalPitch, int iwide,
                            int nxProj, int lsliceLast, int ithick, int nviews,
                            float ycenOut, int lsliceBase, float edgeFill, int use3D)
{
  float kproj, xp, zz, fj, yproj, baseAdj;
  float sum = 0.;
  float ytol = 3.05f;
  int iv, jproj, ind;
#ifdef HAS_LAYERS
  int jslice;
#endif
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < ithick && j < iwide) {
    zz = (i + 1 - ycenOut);
    if (use3D > 0) {
      baseAdj = (float)lsliceBase - 0.5f;
      BPLOCAL_START;
      kproj = iv + 0.5f;
      sum += tex3D(projtex3D, xp, kproj, yproj - baseAdj);
      BPLOCAL_END;
#ifdef HAS_LAYERS
    } else if (use3D < 0) {
      BPLOCAL_START;
      kproj = iv + 0.5f;
      jproj = min((int)yproj, lsliceLast - 1);
      fj = yproj - jproj;
      jslice = jproj - lsliceBase;
      sum += (1.f - fj) * tex2DLayered(projtexLayer, xp, kproj, jslice) + 
        fj * tex2DLayered(projtexLayer, xp, kproj, jslice + 1);
      BPLOCAL_END;
    } else {
#endif
      BPLOCAL_START;
      jproj = min((int)yproj, lsliceLast - 1);
      fj = yproj - jproj;
      kproj = (jproj - lsliceBase) * nviews + iv + 0.5f;
      sum += (1.f - fj) * tex2D(projtex2D, xp, kproj) + 
        fj * tex2D(projtex2D, xp, kproj + nviews);
      BPLOCAL_END;
      }       
    slice[i * slPitch + j] = sum;
  }
}

// Kernel for computing the local projection factors from warping data
__global__ void localProjFactors
(float *xprjf, float *xprjz, float *yprjf, float *yprjz, int pitch, int iv, 
 int nviews, int iwide, int minX, int lslice, int nlines, int nxWarp, int nyWarp,
 int ixStartWarp, int iyStartWarp, int iDelXwarp, int iDelYwarp, float xcenOut,
 float xcenIn, float xcenPaxisOfs, float centerSlice)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int line = blockIdx.y * blockDim.y + threadIdx.y;
  int ind1, ind2, ind3, ind4, ixc, ixt, ixpos, iyt, iypos;
  float fnd1, fnd2, fnd3, fnd4, yzf1, yzf2, yzf3, yzf4, jpos;
  float f1, f2, f3, f4, xx, yy, fx, fy;
  float calf, salf, a11, a12, a21, a22, xadd, yadd, xalladd, yalladd;
  float calf2, salf2, a112, a122, a212, a222, xadd2, yadd2;
  float calf3, salf3, a113, a123, a213, a223, xadd3, yadd3;
  float calf4, salf4, a114, a124, a214, a224, xadd4, yadd4;
  float f1x, f2x, f3x, f4x, f1xy, f2xy, f3xy, f4xy;
  float f1y, f2y, f3y, f4y, f1yy, f2yy, f3yy, f4yy;
  float xp1f, xp1z, yp1f, xp2f, xp2z, yp2f, xp3f, xp3z, yp3f, xp4f, xp4z, yp4f;
  float cosBeta, sinBeta, cosBeta2, sinBeta2, cosBeta3, sinBeta3, cosBeta4, sinBeta4;

  if (j >= iwide || line >= nlines)
    return;
  if (iv < 0)
    iv = line;
  else
    lslice += line;

  // Need to add 1 to j when it is used as a position
  jpos = j + minX + 1;
  ixc = (int)floor(jpos - xcenOut + xcenPaxisOfs + 0.5f);
  ixt = min(max(ixc - ixStartWarp, 0), (nxWarp - 1) * iDelXwarp);
  ixpos = min(ixt / iDelXwarp + 1, nxWarp - 1);
  fx = ((float)(ixt - (ixpos - 1) * iDelXwarp)) / iDelXwarp;
  iyt = min(max(lslice - iyStartWarp, 0), (nyWarp - 1) * iDelYwarp);
  iypos = min(iyt / iDelYwarp + 1, nyWarp - 1);
  fy = ((float)(iyt - (iypos - 1) * iDelYwarp)) / iDelYwarp;

  ind1 = (nxWarp * (iypos - 1) + ixpos - 1) * nviews + iv;
  ind2 = ind1 + nviews;
  ind3 = ind1 + nxWarp * nviews;
  ind4 = ind3 + nviews;
  f1 = (1. - fy) * (1. - fx);
  f2 = (1. - fy) * fx;
  f3 = fy * (1. - fx);
  f4 = fy * fx;
  fnd1 = ind1;
  fnd2 = ind2;
  fnd3 = ind3;
  fnd4 = ind4;
  
  cosBeta = tex2D(localtex, fnd1, CBIND);
  sinBeta = tex2D(localtex, fnd1, SBIND);
  calf = tex2D(localtex, fnd1, CAIND);
  salf = tex2D(localtex, fnd1, SAIND);
  a11 = tex2D(localtex, fnd1, F11IND);
  a12 = tex2D(localtex, fnd1, F12IND);
  a21 = tex2D(localtex, fnd1, F21IND);
  a22 = tex2D(localtex, fnd1, F22IND);
  xadd = tex2D(localtex, fnd1, F13IND) + xcenIn - xcenIn * a11 - centerSlice * a12;
  yadd = tex2D(localtex, fnd1, F23IND) + centerSlice - xcenIn * a21 - centerSlice * a22;

  cosBeta2 = tex2D(localtex, fnd2, CBIND);
  sinBeta2 = tex2D(localtex, fnd2, SBIND);
  calf2 = tex2D(localtex, fnd2, CAIND);
  salf2 = tex2D(localtex, fnd2, SAIND);
  a112 = tex2D(localtex, fnd2, F11IND);
  a122 = tex2D(localtex, fnd2, F12IND);
  a212 = tex2D(localtex, fnd2, F21IND);
  a222 = tex2D(localtex, fnd2, F22IND);
  xadd2 = tex2D(localtex, fnd2, F13IND) + xcenIn - xcenIn * a112 - centerSlice * a122;
  yadd2 = tex2D(localtex, fnd2, F23IND) + centerSlice - xcenIn * a212 - 
    centerSlice * a222;

  cosBeta3 = tex2D(localtex, fnd3, CBIND);
  sinBeta3 = tex2D(localtex, fnd3, SBIND);
  calf3 = tex2D(localtex, fnd3, CAIND);
  salf3 = tex2D(localtex, fnd3, SAIND);
  a113 = tex2D(localtex, fnd3, F11IND);
  a123 = tex2D(localtex, fnd3, F12IND);
  a213 = tex2D(localtex, fnd3, F21IND);
  a223 = tex2D(localtex, fnd3, F22IND);
  xadd3 = tex2D(localtex, fnd3, F13IND) + xcenIn - xcenIn * a113 - centerSlice * a123;
  yadd3 = tex2D(localtex, fnd3, F23IND) + centerSlice - xcenIn * a213 - 
    centerSlice * a223;

  cosBeta4 = tex2D(localtex, fnd4, CBIND);
  sinBeta4 = tex2D(localtex, fnd4, SBIND);
  calf4 = tex2D(localtex, fnd4, CAIND);
  salf4 = tex2D(localtex, fnd4, SAIND);
  a114 = tex2D(localtex, fnd4, F11IND);
  a124 = tex2D(localtex, fnd4, F12IND);
  a214 = tex2D(localtex, fnd4, F21IND);
  a224 = tex2D(localtex, fnd4, F22IND);
  xadd4 = tex2D(localtex, fnd4, F13IND) + xcenIn - xcenIn * a114 - centerSlice * a124;
  yadd4 = tex2D(localtex, fnd4, F23IND) + centerSlice - xcenIn * a214 - 
    centerSlice * a224;
       
  f1x = f1 * a11;
  f2x = f2 * a112;
  f3x = f3 * a113;
  f4x = f4 * a114;
  f1xy = f1 * a12;
  f2xy = f2 * a122;
  f3xy = f3 * a123;
  f4xy = f4 * a124;

  f1y = f1 * a21;
  f2y = f2 * a212;
  f3y = f3 * a213;
  f4y = f4 * a214;
  f1yy = f1 * a22;
  f2yy = f2 * a222;
  f3yy = f3 * a223;
  f4yy = f4 * a224;

  xalladd = f1 * xadd + f2 * xadd2 + f3 * xadd3 + f4 * xadd4;
  yalladd = f1 * yadd + f2 * yadd2 + f3 * yadd3 + f4 * yadd4;
       
  // Each projection position is a sum of a fixed factor ("..f")
  // and a factor that multiplies z ("..z")
   
  xx = jpos - xcenOut;
  yy = lslice - centerSlice;
  xp1f = xx * cosBeta + yy * salf * sinBeta + xcenPaxisOfs;
  xp1z = calf * sinBeta + tex2D(localtex, fnd1, XZFIND);
  xp2f = xx * cosBeta2 + yy * salf2 * sinBeta2 + xcenPaxisOfs;
  xp2z = calf2 * sinBeta2 + tex2D(localtex, fnd2, XZFIND);
  xp3f = xx * cosBeta3 + yy * salf3 * sinBeta3 + xcenPaxisOfs;
  xp3z = calf3 * sinBeta3 + tex2D(localtex, fnd3, XZFIND);
  xp4f = xx * cosBeta4 + yy * salf4 * sinBeta4 + xcenPaxisOfs;
  xp4z = calf4 * sinBeta4 + tex2D(localtex, fnd4, XZFIND);

  yp1f = yy * calf + centerSlice;
  yp2f = yy * calf2 + centerSlice;
  yp3f = yy * calf3 + centerSlice;
  yp4f = yy * calf4 + centerSlice;

  // store the fixed and z - dependent component of the
  // projection coordinates
  yzf1 = tex2D(localtex, fnd1, YZFIND);
  yzf2 = tex2D(localtex, fnd2, YZFIND);
  yzf3 = tex2D(localtex, fnd3, YZFIND);
  yzf4 = tex2D(localtex, fnd4, YZFIND);
  ind1 = pitch * line + j;
  xprjf[ind1] = f1x * xp1f + f2x * xp2f + f3x * xp3f + f4x * xp4f + 
    f1xy * yp1f + f2xy * yp2f + f3xy * yp3f + f4xy * yp4f + xalladd;
  xprjz[ind1] = f1x * xp1z + f2x * xp2z + f3x * xp3z + f4x * xp4z - 
    (f1xy * (salf - yzf1) + f2xy * (salf2 - yzf2) + f3xy * (salf3 - yzf3) + 
     f4xy * (salf4 - yzf4));
  yprjf[ind1] = f1y * xp1f + f2y * xp2f + f3y * xp3f + f4y * xp4f + 
    f1yy * yp1f + f2yy * yp2f + f3yy * yp3f + f4yy * yp4f + yalladd;
  yprjz[ind1] = f1y * xp1z + f2y * xp2z + f3y * xp3z + f4y * xp4z - 
    (f1yy * (salf - yzf1) + f2yy * (salf2 - yzf2) + f3yy * (salf3 - yzf3) + 
     f4yy * (salf4 - yzf4));
}

// Function to load the local alignment data
int gpuloadlocals(float *packed, int *numWarps)
{
  size_t sizetmp = sizeof(float) * *numWarps * sNumViews * 12;
  if (hipMemcpyToArray(devLocalData, 0, 0, packed, sizetmp,
                        hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy local data to GPU array");
    gpudone();
    return 1;
  }
  return 0;
}

// Function to run back-projection with local alignments, first computing the
// the projection factors for all positions and views, then running the 
// back projection kernel
int gpubplocal(float *slice, int *lslice, int *nxWarp, int *nyWarp,
               int *ixStartWarp, int *iyStartWarp, int *iDelXwarp, int *iDelYwarp,
               int *nxProj, float *xcenOut, float *xcenIn, float *axisXoffset,
               float *ycenOut, float *centerSlice, float *edgefill)
{
  int blockX = 16;

  // Compute the local projection factors
  dim3 blockFac(blockX, 16, 1);
  dim3 gridFac((sSliceWidth + blockFac.x - 1) / blockFac.x, 
                (sNumViews + blockFac.y - 1) / blockFac.y, 1);
  localProjFactors<<<gridFac, blockFac>>>
    (devXprojFix, devXprojZ, devYprojFix, devYprojZ, sLocalPitch / 4, -1, sNumViews, 
     sSliceWidth, 0, *lslice, sNumViews, *nxWarp, *nyWarp, *ixStartWarp, *iyStartWarp, 
     *iDelXwarp, *iDelYwarp, *xcenOut, *xcenIn, *xcenIn + *axisXoffset, *centerSlice);
  if (testReportErr("computing localProjFactors"))
      return 1;

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after computing local factors");
    return 1;
  }

  // Do the backprojection
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (sSliceThick + blockSize.y - 1) / blockSize.y, 1);

  bpLocalTest<<<gridSize, blockSize>>>
    (devSlice, sSlicePitch / 4, devXprojFix, devXprojZ, devYprojFix, devYprojZ, 
     sLocalPitch / 4, sSliceWidth, *nxProj, sLsliceFirst + sNumLoadedPlanes - 1, 
     sSliceThick, sNumViews, *ycenOut, sLsliceFirst, *edgefill, sUse3dTexture);
  if (testReportErr("for local backprojection"))
      return 1;

  return (synchronizeCopySlice(devSlice, sSlicePitch, slice, sSliceWidth, 
                               sSliceThick));
}

/*
 * ROUTINES FOR REPROJECTION
 */

// Kernel to do simple reprojection (no X axis tilt or Z factors)
__global__ void reprojNox(float *lines, int pitch, int iwide, int ithick, 
                          int lsliceStart, int lsliceEnd, int lsliceBase, 
                          float xxlim, float xcenAdj, float xcenPaxisOfs,
                          float xProjOffset, float ycenAdj, float sinBeta,
                          float cbetinv, float delz, int numz, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz;
  float zz, sum, frac, zslice, xproj, xx;
  line = i + lsliceStart;
  if (j >= iwide || line > lsliceEnd)
    return;
  sum = 0.;
  xproj = j + 1 + xProjOffset;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (zz  * sinBeta + xcenPaxisOfs)) * cbetinv + xcenAdj;
    if (xx < 0.5f || xx > xxlim) {
      sum += frac * pmean;
    } else {
      zslice += (line - lsliceBase) * ithick;
      sum += frac * tex2D(projtex2D, xx, zslice);
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to do reprojection with X axis tilt and/or Z factors
__global__ void reprojXtilt
(float *lines, int pitch, int iwide, int ithick, int lsliceStart, int lsliceEnd, 
 int lsliceBase, int lsliceLast, float xxlim, float xcenAdj, float xcenPaxisOfs, 
 float xProjOffset, float centerSlice, float yProjOffset, float ycenAdj, float cbetinv, 
 float calfinv, float salfmyz, float salfsbet, float calsbetpxz, float delz, int numz, 
 float pmean, int use3D)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kz, iys;
#ifdef HAS_LAYERS
  int lslice;
#endif
  float zz, sum, frac, zslice, yproj, yy, yslice, xproj, xx, fy, ysliceUse, baseAdj;
  float ytol = 3.05f;
  line = i + lsliceStart;
  if (j >= iwide || line > lsliceEnd)
    return;
  sum = 0.;
  xproj = j + 1 + xProjOffset;
  yproj = line + yProjOffset;
  baseAdj = (float)lsliceBase - 0.5f;
  for (kz = 0; kz < numz; kz++) {
    zz = 1 + kz * delz;
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    zslice = zz - 0.5f;
    zz -= ycenAdj;
    yy = (yproj + zz * salfmyz - centerSlice) * calfinv;
    yslice = yy + centerSlice - yProjOffset;

    // the usual -0.5 is incorporated into xcenAdj
    xx = (xproj - (yy * salfsbet + zz * calsbetpxz + xcenPaxisOfs)) * cbetinv + xcenAdj;
    if (xx < 0.5f || xx > xxlim || yslice < lsliceBase - ytol ||
        yslice > lsliceLast + ytol) {
      sum += frac * pmean;
    } else if (use3D > 0) {
      ysliceUse = fmin((float)lsliceLast, fmax((float)lsliceBase, yslice)) - baseAdj;
      sum += frac * tex3D(projtex3D, xx, zslice, ysliceUse);
    } else {
      iys = (int)yslice;
      if (iys < lsliceBase) {
        iys = lsliceBase;
        fy = 0.;
      } else if (iys >= lsliceLast) {
        iys = lsliceLast - 1;
        fy = 1.;
      } else {
        fy = yslice - iys;
      }
#ifdef HAS_LAYERS
      if (use3D < 0) {
        lslice = iys - lsliceBase;
        sum += frac * ((1. - fy) * tex2DLayered(projtexLayer, xx, zslice, lslice) + 
                       fy * tex2DLayered(projtexLayer, xx, zslice, lslice + 1));
      } else {
#endif
        zslice += (iys - lsliceBase) * ithick;
        sum += frac * ((1. - fy) * tex2D(projtex2D, xx, zslice) + 
                       fy * tex2D(projtex2D, xx, zslice + ithick));
#ifdef HAS_LAYERS
      }
#endif
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to do simple reprojection at high angles (no X axis tilt or Z factors) 
__global__ void reprojNoxHigh
(float *lines, int pitch, int iwide, int ithick, int lsliceStart, int lsliceEnd, 
 int lsliceBase, float zzlim, float xcenAdj, float xcenPaxisOfs, float xProjOffset, 
 float ycenAdj, float cosBeta, float denomInv, float delx, int numx, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kx;
  float zz, sum, frac, zslice, xproj, xx;
  line = i + lsliceStart;
  if (j >= iwide || line > lsliceEnd)
    return;
  sum = 0.;
  xproj = j + 1 + xProjOffset;
  for (kx = 0; kx < numx; kx++) {
    xx = 1.f + kx * delx;
    frac = 1.f;
    if (xx > iwide) {
      frac = 1.f - (xx - (int)xx);
      xx = iwide;
    }
    
    zz = (xproj - xcenPaxisOfs - (xx - xcenAdj) * cosBeta) * denomInv;
    zslice = zz + ycenAdj;

    if (zslice < 0.5f || zslice > zzlim) {
      sum += frac * pmean;
    } else {
      zslice += (line - lsliceBase) * ithick;
      sum += frac * tex2D(projtex2D, xx - 0.5f, zslice);
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to do reprojection at high angles with X axis tilt and/or Z factors
__global__ void reprojXtiltHigh
(float *lines, int pitch, int iwide, int ithick, int lsliceStart, int lsliceEnd, 
 int lsliceBase, int lsliceLast, float zzlim, float xcenAdj, float xcenPaxisOfs, 
 float xProjOffset, float centerSlice, float yProjOffset, float ycenAdj, float cosBeta, 
 float calfinv, float salfmyz, float salfsbetdcal, float denomInv, float delx, int numx, 
 float pmean, int use3D)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, kx, iys;
#ifdef HAS_LAYERS
  int lslice;
#endif
  float zz, sum, frac, zslice, yproj, yy, yslice, xproj, xx, fy, ysliceUse, baseAdj;
  float ytol = 3.05f;
  line = i + lsliceStart;
  if (j >= iwide || line > lsliceEnd)
    return;
  sum = 0.;
  xproj = j + 1 + xProjOffset;
  yproj = line + yProjOffset;
  baseAdj = (float)lsliceBase - 0.5f;
  for (kx = 0; kx < numx; kx++) {
    xx = 1.f + kx * delx;
    frac = 1.f;
    if (xx > iwide) {
      frac = 1.f - (xx - (int)xx);
      xx = iwide;
    }
    
    // Here xcenAdj does not have the -0.5 and ycenAdj does
    zz = (xproj - (yproj - centerSlice) * salfsbetdcal - xcenPaxisOfs - 
          (xx - xcenAdj) * cosBeta) * denomInv;
    yy = (yproj + zz * salfmyz - centerSlice) * calfinv;
    yslice = yy + centerSlice - yProjOffset;
    zslice = zz + ycenAdj;

    if (zslice < 0.5f || zslice > zzlim || yslice < lsliceBase - ytol ||
        yslice > lsliceLast + ytol) {
      sum += frac * pmean;
    } else {
      xx -= 0.5f;
      if (use3D > 0) {
        ysliceUse = fmin((float)lsliceLast, fmax((float)lsliceBase, yslice)) - baseAdj;
        sum += frac * tex3D(projtex3D, xx, zslice, ysliceUse);
      } else {
        iys = (int)yslice;
        if (iys < lsliceBase) {
          iys = lsliceBase;
          fy = 0.;
        } else if (iys >= lsliceLast) {
          iys = lsliceLast - 1;
          fy = 1.;
        } else {
          fy = yslice - iys;
        }
#ifdef HAS_LAYERS
        if (use3D < 0) {
          lslice = iys - lsliceBase;
          sum += frac * ((1. - fy) * tex2DLayered(projtexLayer, xx, zslice, lslice) + 
                         fy * tex2DLayered(projtexLayer, xx, zslice, lslice + 1));
        } else {
#endif
          zslice += (iys - lsliceBase) * ithick;
          sum += frac * ((1. - fy) * tex2D(projtex2D, xx, zslice) + 
                         fy * tex2D(projtex2D, xx, zslice + ithick));
#ifdef HAS_LAYERS
        }
#endif
      }
    }
  }
  lines[pitch * i + j] = sum;
}

// Function to run reprojection for all cases except local alignments
int gpureproject(float *lines, float *sinBeta, float *cosBeta, float *sinAlpha, 
                 float *cosAlpha, float *xzfac, float *yzfac, float *delz,
                 int *lsliceStart, int *lsliceEnd, int *ithick,
                 float *xcenOut, float *xcenPaxisOfs, int *minXreproj, 
                 float *xProjOffset, float *ycenOut, int *minYreproj,
                 float *yProjOffset, float *centerSlice, int *ifAlpha, float *pmean)
{ 
  int blockX = 16;
  int numz, numx, numLines = *lsliceEnd + 1 - *lsliceStart;
  int lastSlice = sLsliceFirst + sNumLoadedPlanes - 1;
  float znum, xcenAdj, salfsbet, calsbetpxz, ycenAdj, salfmyz, cbetinv,calfinv;
  float delx, xnum, salsbetdcal, denomInv;

  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (numLines + blockSize.y - 1) / blockSize.y, 1);

  if (*ifAlpha == 0 && sUse3dTexture != 0) {
    pflush("GPU problem: Program called gpureproject after setting up to use texture "
           "type %d\n", sUse3dTexture);
    return 1;
  }

  // Common items
  xcenAdj = *xcenOut - (*minXreproj-1) - 0.5;
  ycenAdj = *ycenOut + 1 - *minYreproj;
  salfmyz = *sinAlpha - *yzfac;
  calfinv = 1. / *cosAlpha;
  calsbetpxz = *cosAlpha * *sinBeta + *xzfac;
  
  if (fabs(*sinBeta * *ithick) <= fabs(*cosBeta * sSliceWidth)) {

    // Regular low-angle lines
    znum = 1. + (*ithick - 1) / *delz;
    numz = (int)znum;
    if (znum - numz > 0.1)
      numz++;
    salfsbet = *sinAlpha * *sinBeta;
    cbetinv = 1. / *cosBeta;

    if (*ifAlpha) {
      reprojXtilt<<<gridSize, blockSize>>>
        (devSlice, sSlicePitch / 4, sSliceWidth, *ithick, *lsliceStart, *lsliceEnd, 
         sLsliceFirst, lastSlice, sNxPlane - 0.5, xcenAdj, *xcenPaxisOfs,
         *xProjOffset, *centerSlice, *yProjOffset, ycenAdj, cbetinv, calfinv, salfmyz,
         salfsbet, calsbetpxz, *delz, numz, *pmean, sUse3dTexture);
    } else {
      reprojNox<<<gridSize, blockSize>>>
        (devSlice, sSlicePitch / 4, sSliceWidth, *ithick, *lsliceStart, *lsliceEnd, 
         sLsliceFirst, sNxPlane - 0.5, xcenAdj, *xcenPaxisOfs, *xProjOffset,
         ycenAdj, *sinBeta, cbetinv, *delz, numz, *pmean);
    }
  } else {

    // High angle vertical lines
    delx = (float)fabs(*sinBeta);
    xnum = 1. + (sSliceWidth - 1) / delx;
    numx = (int)xnum;
    if (xnum - numx > 0.1)
      numx++;
    salsbetdcal = *sinAlpha * *sinBeta / *cosAlpha;
    denomInv = 1. / (salfmyz * salsbetdcal + calsbetpxz);
    if (*ifAlpha) {
      reprojXtiltHigh<<<gridSize, blockSize>>>
        (devSlice, sSlicePitch / 4, sSliceWidth, *ithick, *lsliceStart, *lsliceEnd, 
         sLsliceFirst, lastSlice, *ithick - 0.5, xcenAdj + 0.5, *xcenPaxisOfs,
         *xProjOffset, *centerSlice, *yProjOffset, ycenAdj - 0.5, *cosBeta, calfinv, 
         salfmyz, salsbetdcal, denomInv, delx, numx, *pmean, sUse3dTexture);
    } else {
      reprojNoxHigh<<<gridSize, blockSize>>>
        (devSlice, sSlicePitch / 4, sSliceWidth, *ithick, *lsliceStart, *lsliceEnd, 
         sLsliceFirst, *ithick - 0.5, xcenAdj + 0.5, *xcenPaxisOfs, *xProjOffset,
         ycenAdj - 0.5, *cosBeta, denomInv, delx, numx, *pmean);
    }
  }
  if (testReportErr("for reprojection"))
    return 1;
  return (synchronizeCopySlice(devSlice, sSlicePitch, lines, sSliceWidth,
                               numLines));
}

/*
 * ROUTINES TO REPROJECT A SINGLE SLICE
 */

// Kernel to reproject one slice
__global__ void reprojOneSlice(float *lines, int pitch, int iwide, int ithick, 
                               float ycen, int numproj, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int kz;
  float zz, sum, frac, xcenAdj, xx;
  sum = 0.;
  if (j >= iwide || i >= numproj )
    return;
  for (kz = 0; kz < rpNumz[i]; kz++) {
    zz = 1 + kz * tables[COSOFS + i];
    frac = 1.;
    if (zz > ithick) {
      frac = 1. - (zz - (int)zz);
      zz = ithick;
    }
    xcenAdj = iwide / 2;

    // Invert what is multipled by sine because these sines were never inverted
    // inside tilt.f, unlike the signs for regular reproj
    // The usual 0.5 is incorporated into xcenAdj
    xx = (j + 1 - ((ycen - zz)  * tables[SINOFS+i] + xcenAdj + 0.5f)) * 
      tables[INVOFS+i] + xcenAdj;
    if (xx < 0.5f || xx > iwide - 0.5f) {
      sum += frac * pmean;
    } else {
      sum += frac * tex2D(rpSlicetex, xx, zz - 0.5f);
    }
  }
  lines[pitch * i + j] = sum;
}

// Kernel to reproject one slice at high angles
__global__ void reprojOneHighSlice(float *lines, int pitch, int iwide, int ithick, 
                                   float ycen, int numproj, float pmean)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int kz;
  float zz, sum, frac, xcenAdj, xx, delta;
  if (j >= iwide || i >= numproj )
    return;
  sum = 0.f;
  if (rpNumz[i] >= 0) {
    xcenAdj = iwide / 2;
    delta = fabsf(tables[COSOFS + i]);
    for (kz = 0; kz < rpNumz[i]; kz++) {
      zz = 1 + kz * delta;
      frac = 1.f;
      if (zz > ithick) {
        frac = 1.f - (zz - (int)zz);
        zz = ithick;
      }
      
    // Invert what is multipled by sine because these sines were never inverted
    // inside tilt.f, unlike the signs for regular reproj
    // The usual -0.5 is incorporated into xcenAdj
      xx = (j + 1 - ((ycen - zz)  * tables[SINOFS+i] + xcenAdj + 0.5f)) * 
        tables[INVOFS+i] + xcenAdj;
      if (xx < 0.5f || xx > iwide - 0.5f) {
        sum += frac * pmean;
      } else {
        sum += frac * tex2D(rpSlicetex, xx, zz - 0.5f);
      }
    }
  } else {

    // Going across in X.  Here ycen incorporates the -0.5 and xcenAdj does not
    xcenAdj = iwide / 2 + 0.5f;
    ycen -= 0.5;
    delta = fabsf(tables[SINOFS + i]);
    for (kz = 0; kz < -rpNumz[i]; kz++) {
      xx = 1 + kz * delta;
      frac = 1.f;
      if (xx > iwide) {
        frac = 1.f - (xx - (int)xx);
        xx = iwide;
      }
      
      zz = ((xx - xcenAdj) * tables[COSOFS + i] - j - 1 + xcenAdj) * tables[SINVOFS + i] +
        ycen;
      if (zz < 0.5f || zz > ithick - 0.5f) {
        sum += frac * pmean;
      } else {
        sum += frac * tex2D(rpSlicetex, xx - 0.5f, zz);
      }
    }
  }
  lines[pitch * i + j] = sum;
}

// Function to reproject one slice
int gpureprojoneslice(float *slice, float *lines, float *sinBeta, float *cosBeta,
                      float *ycen, int *numproj, float *pmean)
{
  float znum, cosinv[DELTA_OFS], sininv[DELTA_OFS];
  int numz[DELTA_OFS];
  int blockX = 16;
  int iv, high = 0;

  // Get limited inverse cosines and number of points to do in Z
  loadBetaInvertCos(cosBeta, sinBeta, cosinv, *numproj);

  for (iv = 0; iv < *numproj; iv++) {
    if (fabs(sinBeta[iv] * sSliceThick) <= fabs(cosBeta[iv] * sSliceWidth)) {
      znum = 1. + (sSliceThick - 1) * fabs(cosinv[iv]);
      numz[iv] = (int)znum;
      if (znum - numz[iv] > 0.1)
        numz[iv]++;
      sininv[iv] = 0.;
    } else {

      // For high angle slice, get the number of columns in X, save as a negative
      high = 1;
      sininv[iv] = 1. / sinBeta[iv];
      znum = 1. + (sSliceWidth - 1) * fabs(sininv[iv]);
      numz[iv] = (int)znum;
      if (znum - numz[iv] > 0.1)
        numz[iv]++;
      numz[iv] = -numz[iv];
    }
  }

  // Load constant data
  iv = *numproj * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cosinv, iv, INVOFS*4, hipMemcpyHostToDevice)
      || hipMemcpyToSymbol(HIP_SYMBOL(rpNumz), numz, iv, 0, hipMemcpyHostToDevice) ||
      (high && hipMemcpyToSymbol(HIP_SYMBOL(tables), sininv, iv, SINVOFS*4, 
                                  hipMemcpyHostToDevice))) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }
  
  // Copy slice
  iv = sizeof(float) * sSliceWidth * sSliceThick;
  if (hipMemcpyToArray(devRpSlice, 0, 0, slice, iv, hipMemcpyHostToDevice)
      != hipSuccess) {
    pflerr("Failed to copy slice array to device");
    return 1;
  }
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (*numproj + blockSize.y - 1) / blockSize.y, 1);
  if (high)
    reprojOneHighSlice<<<gridSize, blockSize>>>
      (devReproj, sReprojPitch / 4, sSliceWidth, sSliceThick, *ycen, *numproj, *pmean);
  else
    reprojOneSlice<<<gridSize, blockSize>>>
      (devReproj, sReprojPitch / 4, sSliceWidth, sSliceThick, *ycen, *numproj, *pmean);

  if (testReportErr("for reprojection"))
    return 1;

  return (synchronizeCopySlice(devReproj, sReprojPitch, lines, sNxPlane, *numproj));
}

/*
 * ROUTINES FOR REPROJECTION WITH LOCAL ALIGNMENTS
 */

/*
  Finds loaded point that projects to xproj, yproj at centered Z value
  zz, using stored values for [xy]zfac[fv].  Takes starting value in xx,yy
  and returns found value.
  Xproj, yproj are coordinates in original aligned stack.
  XX coordinate is in terms of the loaded data in X
  YY coordinate is in yterms of slices of reconstruction
*/
__device__ void loadedProjectingPoint
(float xproj, float yproj, float zz, float ofsxpz, float ofsypf, float ofsypz, 
 int nxload, int lsliceBase, int lsliceLast, float *xx, float *yy)
{
  int iter, ix, iy, ifout;
  float xp11, yp11, xp12, yp12, xp21, yp21, xerr, yerr, dypx, dxpy,dxpx;
  float dypy, den, fx, fy, findx1, findx2, findy1, findy2;

  for (iter = 0; iter < 5; iter++) {
    ix = (int)floor(*xx);
    iy = (int)floor(*yy);
    ifout = 0;
    if (ix < 1 || ix >= nxload || iy < lsliceBase || iy >= lsliceLast) {
      ifout = 1;
      ix = min(nxload - 1, max(1, ix));
      iy = min(lsliceLast - 1, max(lsliceBase, iy));
    }

    findx1 = ix - 1;
    findx2 = findx1 + 1.;
    findy1 = iy - lsliceBase;
    findy2 = findy1 + 1;
    //*yy = tex2D(pfactex, findx1, findy1 + ofsypf); return;
    xp11 = tex2D(pfactex, findx1, findy1) + 
      tex2D(pfactex, findx1, findy1 + ofsxpz) * zz;
    yp11 = tex2D(pfactex, findx1, findy1 + ofsypf) + 
      tex2D(pfactex, findx1, findy1 + ofsypz) * zz;
    xp21 = tex2D(pfactex, findx2, findy1) + 
      tex2D(pfactex, findx2, findy1 + ofsxpz) * zz;
    yp21 = tex2D(pfactex, findx2, findy1 + ofsypf) + 
      tex2D(pfactex, findx2, findy1 + ofsypz) * zz;
    xp12 = tex2D(pfactex, findx1, findy2) + 
      tex2D(pfactex, findx1, findy2 + ofsxpz) * zz;
    yp12 = tex2D(pfactex, findx1, findy2 + ofsypf) + 
      tex2D(pfactex, findx1, findy2 + ofsypz) * zz;
 
    xerr = xproj - xp11;
    yerr = yproj - yp11;
    dxpx = xp21 - xp11;
    dxpy = xp12 - xp11;
    dypx = yp21 - yp11;
    dypy = yp12 - yp11;
    den = dxpx * dypy - dxpy * dypx;
    fx = (xerr * dypy - yerr * dxpy) / den;
    fy = (dxpx * yerr - dypx * xerr) / den;
    *xx = ix + fx;
    *yy = iy + fy;
    if (fx > -0.1 & fx < 1.1 && fy > -0.1 && fy < 1.1) 
      return;
    if (ifout && (iter > 0 ||  *xx < 0. || *xx > nxload + 1 || 
                  *yy < lsliceBase - 1. || *yy > lsliceLast + 1.))
      return;
  }
}

// Kernel for reprojection with local alignments
__global__ void reprojLocal
(float *lines, int pitch, int nWarpDelz, float dxWarpDelz, int nxload,
 int iwide, int ithick, int lsliceStart, int lsliceEnd, int lsliceBase, int lsliceLast,
 float xprojMin, float xprojMax, float xcenAdj, float xcenPaxisOfs,
 float xProjOffset, float centerSlice, float yProjOffset, float ycenAdj, float cosBeta,
 float sinBeta, float cbetinv, float calfinv, float salfmyz, float salfsbet,
 float calsbetpxz, float pmean, int use3D)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;
  int line, lastZdone, iy;
#ifdef HAS_LAYERS
  int lslice;
#endif
  float zz, sum, frac, zslice, yproj, yy, xproj, xx, fy, zind, fline, ofsypz;
  float xxtex, ofsxpz, ofsypf, baseAdj;
  float ytol = 3.05f;
  float zzlim, lbaseMtol, llastPtol, dxWarpInv;
  //  int skip =390;

  line = i + lsliceStart;
  sum = 0.;
  if (j >= iwide || line > lsliceEnd)
    return;

  ofsxpz = lsliceLast + 1 - lsliceBase;
  ofsypf = ofsxpz + ofsxpz;
  ofsypz = ofsypf + ofsxpz;
  fline = i;
  yproj = line + yProjOffset;
  baseAdj = (float)lsliceBase - 0.5f;

  /* Get x projection coord, starting centered Z coordinate, and
     approximate x and y coordinates 
     X coordinate needs to be a loaded X index
     Y coordinate is in slices of reconstruction */

  // ycenAdj needs to be ycen - (minYreproj - 1)
  // xcenAdj = xcen - (minXload - 1)
  xproj = j + 1 + xProjOffset;
  zz = 1. - ycenAdj;
  yy = (yproj + zz * salfmyz - centerSlice) * calfinv + centerSlice;
  xx = (xproj - (yy*salfsbet + zz * calsbetpxz + xcenPaxisOfs)) * cbetinv +
    xcenAdj;
  yy -= yProjOffset;
  //lines[pitch * i + j] = yy; return;

  // Precalculate some items, doesn't help
  zzlim = ithick + 1 - ycenAdj;
  lbaseMtol = lsliceBase - ytol;
  llastPtol = lsliceLast + ytol;
  dxWarpInv = 1. / dxWarpDelz;

  // Move on ray up in Z
  lastZdone = 0;
              
  while (zz < zzlim && !lastZdone) {

    // xprojMin/Max already adjusted by 5
    if (xproj < xprojMin || xproj > xprojMax) {
      sum = sum + pmean;
      //if (zz + ycenAdj > ithick - skip) {lines[pitch * i + j] = 0; return;}
    } else {
      loadedProjectingPoint(xproj, yproj, zz, ofsxpz, ofsypf, ofsypz,
                            nxload, lsliceBase, lsliceLast, &xx, &yy);
      //if (zz + ycenAdj > ithick - skip) {lines[pitch * i + j] = yy; return;}

      // If X or Y is out of bounds, fill with mean
      if (yy < lbaseMtol || yy > llastPtol || xx < 1. || xx >= nxload) {
        sum = sum + pmean;
      } else {

        // otherwise, get x, y, z indexes, clamp y to limits, allow
        // a fractional Z pixel at top of volume
        xxtex = xx - 0.5f;
        yy = max((float)lsliceBase, min(lsliceLast - 0.01, yy));
        zslice = zz + ycenAdj;
        frac = 1.;
        if (zslice > ithick) {
          frac = 1. - (zslice - (int)zslice);
          zslice = ithick - 0.5f;
          lastZdone = 1;
        } else
          zslice -= 0.5f;
                     
        // Do the interpolation
        if (use3D > 0) {
          sum += frac * tex3D(projtex3D, xxtex, zslice, yy - baseAdj);
#ifdef HAS_LAYERS
        } else if (use3D < 0) {
          iy = yy;
          fy = yy - iy;
          lslice = iy - lsliceBase;
          sum += frac * ((1. - fy) * tex2DLayered(projtexLayer, xxtex, zslice, lslice) +
                         fy * tex2DLayered(projtexLayer, xxtex, zslice, lslice + 1));
#endif
        } else {
          iy = yy;
          fy = yy - iy;
          zslice += (iy - lsliceBase) * ithick;
          sum += frac * ((1. - fy) * tex2D(projtex2D, xxtex, zslice) +
                         fy * tex2D(projtex2D, xxtex, zslice + ithick));
        }

        // ELIMINATED JUMPING, IT TAKES 50% LONGER
      }
    }
                 
    // Adjust Z by local factor, move X approximately for next pixel
    zind = max(0., min(nWarpDelz - 1., xx * dxWarpInv));
    zz = zz + tex2D(delztex, zind, fline);
    xx = xx + sinBeta;
  }
  lines[pitch * i + j] = sum;
}

// Function to do reprojection with local alignments
int gpureprojlocal
(float *lines, float *sinBeta, float *cosBeta, float *sinAlpha, float *cosAlpha,
 float *xzfac, float *yzfac, int *nxWarp, int *nyWarp, int *ixStartWarp, 
 int *iyStartWarp, int *iDelXwarp, int *iDelYwarp, float *warpDelz, int *nWarpDelz, 
 float *dxWarpDelz, float *xprojMin, float *xprojMax, int *lsliceStart, int *lsliceEnd,
 int *ithick, int *iview, float *xcenOut, float *xcenIn, float *axisXoffset, 
 int *minXload, float *xProjOffset, float *ycenAdj, float *yProjOffset,
 float *centerSlice, float *pmean)
{
  int blockX = 16;
  int numLines = *lsliceEnd + 1 - *lsliceStart;
  int lastSlice = sLsliceFirst + sNumLoadedPlanes - 1;
  int nbd, nbp;
  float xcenAdj, salfsbet, calsbetpxz, salfmyz, cbetinv,calfinv;

  xcenAdj = *xcenOut - (*minXload-1);
  salfsbet = *sinAlpha * *sinBeta;
  calsbetpxz = *cosAlpha * *sinBeta + *xzfac;
  salfmyz = *sinAlpha - *yzfac;
  cbetinv = 1. / *cosBeta;
  calfinv = 1. / *cosAlpha;
  nbd = (int)floor(*yProjOffset + 0.5);

  // Compute the local projection factors
  dim3 blockFac(blockX, 16, 1);
  dim3 gridFac((sNxPlane + blockFac.x - 1) / blockFac.x, 
                (sNumLoadedPlanes + blockFac.y - 1) / blockFac.y, 1);
  localProjFactors<<<gridFac, blockFac>>>
    (devXprojFix, devXprojZ, devYprojFix, devYprojZ, sLocalPitch / 4, *iview - 1, 
     sNumViews, sNxPlane, *minXload - 1, sLsliceFirst + nbd, sNumLoadedPlanes, *nxWarp,
     *nyWarp, *ixStartWarp, *iyStartWarp, *iDelXwarp, *iDelYwarp, *xcenOut, *xcenIn, 
     *xcenIn+*axisXoffset, *centerSlice);
  if (testReportErr("computing localProjFactors"))
      return 1;
  /* return (synchronizeCopySlice(devYprojFix, sLocalPitch, lines, sSliceWidth,
     numLines)); */

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after computing local factors");
    return 1;
  }

  // Load the texture arrays
  nbd = sizeof(float) * *nWarpDelz * numLines;
  nbp = sizeof(float) * sNxPlane;
  if (hipMemcpyToArray(devDelz, 0, 0, warpDelz, nbd, hipMemcpyHostToDevice)
      != hipSuccess ||
      hipMemcpy2DToArray(devLocalPfac, 0, 0, devXprojFix, sLocalPitch, nbp, 
                          sNumLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(devLocalPfac, 0, sNumLoadedPlanes, devXprojZ, sLocalPitch,
                          nbp, sNumLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(devLocalPfac, 0, 2*sNumLoadedPlanes, devYprojFix, sLocalPitch,
                          nbp, sNumLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess ||
      hipMemcpy2DToArray(devLocalPfac, 0, 3*sNumLoadedPlanes, devYprojZ, sLocalPitch,
                          nbp, sNumLoadedPlanes, hipMemcpyDeviceToDevice) 
      != hipSuccess) {
    pflerr("Failed to copy local proj factors to texture array");
    return 1;
  }

  // Do the reprojection
  dim3 blockSize(blockX, 16, 1);
  dim3 gridSize((sSliceWidth + blockSize.x - 1) / blockSize.x, 
                (numLines + blockSize.y - 1) / blockSize.y, 1);
  reprojLocal<<<gridSize, blockSize>>>
    (devSlice, sSlicePitch / 4, *nWarpDelz, *dxWarpDelz, sNxPlane, sSliceWidth,
     *ithick, *lsliceStart, *lsliceEnd, sLsliceFirst, lastSlice, *xprojMin, *xprojMax,
     xcenAdj, *xcenIn + *axisXoffset, *xProjOffset, *centerSlice, *yProjOffset, *ycenAdj,
     *cosBeta, *sinBeta, cbetinv, calfinv, salfmyz, salfsbet, calsbetpxz, *pmean, 
     sUse3dTexture);
  if (testReportErr("for local reprojection"))
      return 1;
  return (synchronizeCopySlice(devSlice, sSlicePitch, lines, sSliceWidth,
                               numLines));
}

/*
 * UTILITY ROUTINES
 */
   
// Load cosine and sine beta into constant array and compute inverse cosine
static int loadBetaInvertCos(float *cosBeta, float *sinBeta, float *cosinv, 
                             int num)
{
  int i, iv;
  float yy;

  // Invert cosines with limit
  for (i = 0; i < num; i++) {
    yy = cosBeta[i];
    if (fabs(yy) < 0.001f)
      yy = yy >= 0 ? 0.001f : -0.001f;
    cosinv[i] = 1.f / yy;
  }

  // Copy sines/cosines
  iv = num * sizeof(float);
  if (hipMemcpyToSymbol(HIP_SYMBOL(tables), cosBeta, iv, 0, hipMemcpyHostToDevice) ||
      hipMemcpyToSymbol(HIP_SYMBOL(tables), sinBeta, iv, SINOFS*4,
                            hipMemcpyHostToDevice)) {
    pflerr("Failed to copy constant data to GPU");
    return 1;
  }
  return 0;
}

// Synchronize the threads and copy computed data back to caller's array
static int synchronizeCopySlice(float *devslc, int pitch, float *slice,
                                int width, int numLines)
{
  int sizetmp;
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after backprojection");
    return 1;
  }

  // Get slice back
  sizetmp = sizeof(float) * width;
  if (hipMemcpy2D(slice, sizetmp, devslc, pitch, sizetmp, numLines, 
                   hipMemcpyDeviceToHost) != hipSuccess) {
    pflerr("Error copying slice back to host");
    return 1;
  }
  return 0;
}

// Test for and report error after executing threads           
static int testReportErr(const char *mess)
{
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    pflush("Error executing threads %s: %s\n", mess,
           hipGetErrorString(err));
    return 1;
  }
  return 0;
}

// Print a message with flushes to get it out before fortran output
static void pflush(const char *format, ...)
{
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf("%s", errorMess);
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

// In case of error, find the error string and print it with message
static void pflerr(const char *format, ...)
{
  hipError_t err;
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  printf("%s", errorMess);
  err = hipGetLastError();
  pflush(": %s\n", hipGetErrorString(err));
  fflush(stdout);  
  fflush(stdout);
  va_end(args);
}

// Print appropriate error from allocation and free all arrays
static void allocerr(const char *mess, int *nplanes, int *firstNpl,
                     int *lastNpl, int ifcuda)
{
  const char *whichText[3] = {"first", "last", "only"};
  int which = 2;
  gpudone();
  if (*firstNpl != *lastNpl) {
    if (*nplanes == *firstNpl)
      which = 0;
    else if (*nplanes == *lastNpl)
      which = 1;
    else
      return;
  }
  if (ifcuda)
    pflerr("On %s try (for %d planes), %s", whichText[which], *nplanes, mess);
  else
    pflush("On %s try (for %d planes), %s", whichText[which], *nplanes, mess);
}


