#include "hip/hip_runtime.h"

/*
 *  gpuframe.cu -- Kernel and supporting code for frame summing on GPU
 *
 *  Author: David Mastronarde   email: mast@colorado.edu
 *
 *  Copyright (C) 2016 by the Regents of the University of 
 *  Colorado.  See dist/COPYRIGHT for full copyright notice.
 *
 *  $Id$
 */
#include <stdio.h>
#include <stdarg.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "b3dutil.h"
#include "gpuframe.h"
#include "framealign.h"
#include "frameutil.h"

#define PI 3.141593
#define FULL_INDENT 8

// Surfaces save ~1% in pre-processing and 7-10% in reduction, as well as saving memory
#if CUDA_VERSION < 4000
#define hipDeviceSynchronize hipDeviceSynchronize
#define NO_SURFACES
#endif

static void destroyPlan(hipfftHandle &plan);
static void free2Darray(hipArray **array);
static hipError_t malloc2Darray(hipArray **arrayPtr, int type, int sizeX, int sizeY);
static hipError_t bindUnpadArray(hipArray *arrayPtr, int type);

static FrameGPU sFGPU;

// Use almost all of constant memory for the sines and cosines
#define MAX_KERNEL 100
#define INT_KERNEL_SCALE 16000.f
#define MAX_TABLE (65536 / 4 - MAX_KERNEL)
__constant__ float trigTable[MAX_TABLE];
__constant__ int intKernel[MAX_KERNEL];

/*
 * Static CUDA variables (to keep them out of the .h file)
 */
static hipfftHandle sFullForwardPlan = 0;
static hipfftHandle sSumInversePlan = 0;
static hipfftHandle sAlignForwardPlan = 0;
static hipfftHandle sAlignInversePlan = 0;
static texture<float, 1, hipReadModeElementType> sSumTex;
static texture<float, 1, hipReadModeElementType> sOddTex;
static texture<float, 1, hipReadModeElementType> sFullTex;
static texture<float, 1, hipReadModeElementType> sMaskTex;
static texture<float, 1, hipReadModeElementType> sDWFilterTex;
static texture<float, 2, hipReadModeElementType> sUnpadFloatTex;
static texture<unsigned char, 2, hipReadModeElementType> sUnpadByteTex;
static texture<short int, 2, hipReadModeElementType> sUnpadShortTex;
static texture<unsigned short int, 2, hipReadModeElementType> sUnpadUShortTex;
static texture<float, 2, hipReadModeElementType> sGainRefTex;
static texture<unsigned char, 2, hipReadModeElementType> sDefectTex;
static hipChannelFormatDesc sChanDesc;
static hipChannelFormatDesc sByteChanDesc;
static hipChannelFormatDesc sShortChanDesc;
static hipChannelFormatDesc sUShortChanDesc;
static hipArray *sGainRefArray = NULL;
static hipArray *sDefectMapArray = NULL;
static hipArray *sTempRawArray = NULL;
static hipArray *sTempFloatArray = NULL;
#ifndef NO_SURFACES
static surface<void, 2> sTempSurfRef;
#endif
std::vector<hipArray *>sSavedUnpadded;
static hipArray *sReducedInXarray = NULL;

#define START_TIMER  if (mTrackTime) mWallStart = wallTime();
#define ADD_TIME(a) if (mTrackTime) a += wallTime() - mWallStart;

#define SETUP_TEXTURE(tex)                    \
  tex.addressMode[0] = hipAddressModeClamp;    \
  tex.addressMode[1] = hipAddressModeClamp;    \
  tex.filterMode = hipFilterModePoint;         \
  tex.normalized = false;

/*
 * KERNELS
 */

// Shift an FFT array and add into a sum array, possibly with size reduction
__global__ void shiftAndAddToSum(float *sumArr, float *nonDWsum, int ixStart,
                                 int iyInStart, int iyOutStart, int numXdo, int numYdo,
                                 int nxInDim, int nxOutDim, int tableYoffset, 
                                 int numDoseFilt, float doseFiltDelta, float freqDelX, 
                                 float freqDelY)
{
  int indIn, indOut, trigXind, trigYind, ixFull, iyFull;
  float real, imag, xcos, xsin, ycos, ysin, phre, phim, sumReal, sumImag, xfreq, yfreq;
  float freq, newReal, newImag, atten = 1.f;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;

  // textures only make a tiny difference here
  if (ix < numXdo && iy < numYdo) {
    ixFull = ix + ixStart;
    iyFull = iy + iyInStart;
    if (numDoseFilt > 0) {

      // Determine attenuation factor based on frequency, or return if it will be 0
      // Multiple if statements testing against a maximum frequency do not save any time
      // Ie., testing "if (freq > (numDoseFilt - 1) * doseFiltDelta"
      xfreq = ixFull * freqDelX;
      yfreq = iyFull * freqDelY;
      if (yfreq > 0.5f)
        yfreq = 1.f - yfreq;
      freq = sqrt(xfreq * xfreq + yfreq * yfreq);
      atten = tex1Dfetch(sDWFilterTex, freq / doseFiltDelta + 0.5f);
    }
    indIn = 2 * (ixFull + nxInDim * iyFull);
    indOut = 2 * (ix + ixStart + nxOutDim * (iy + iyOutStart));
    real = tex1Dfetch(sFullTex, indIn);
    imag = tex1Dfetch(sFullTex, indIn + 1);
    //real = fullArr[indIn];
    //imag = fullArr[indIn + 1];
    trigXind = 2 * ix;
    trigYind = tableYoffset + 2 * iy;
    xcos = trigTable[trigXind];
    xsin = trigTable[trigXind + 1];
    ycos = trigTable[trigYind];
    ysin = trigTable[trigYind + 1];
    phre = xcos * ycos - xsin * ysin;
    phim = xsin * ycos + xcos * ysin;
    newReal = phre * real - phim * imag;
    newImag = phim * real + phre * imag;
    sumReal = tex1Dfetch(sSumTex, indOut);
    sumImag = tex1Dfetch(sSumTex, indOut + 1);
    sumArr[indOut] = sumReal + atten * newReal;
    sumArr[indOut + 1] = sumImag + atten * newImag;
    if (nonDWsum) {
      nonDWsum[indOut] += newReal;
      nonDWsum[indOut + 1] += newImag;
    }
  }
}

// Shift an array and store it back shifted, and add it to a sum, with no reduction
__global__ void shiftInPlaceAddToSum(float *fullArr, float *sumArr, int ixStart,
                                     int iyStart, int numXdo, int numYdo, int nxDim,
                                     int tableYoffset)
{
  int indIn, trigXind, trigYind;
  float real, imag, xcos, xsin, ycos, ysin, phre, phim, sumReal, sumImag;
  float shiftReal, shiftImag;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < numXdo && iy < numYdo) {
    indIn = 2 * (ix + ixStart + nxDim * (iy + iyStart));
    real = tex1Dfetch(sFullTex, indIn);
    imag = tex1Dfetch(sFullTex, indIn + 1);
    //real = fullArr[indIn];
    //imag = fullArr[indIn + 1];
    trigXind = 2 * ix;
    trigYind = tableYoffset + 2 * iy;
    xcos = trigTable[trigXind];
    xsin = trigTable[trigXind + 1];
    ycos = trigTable[trigYind];
    ysin = trigTable[trigYind + 1];
    phre = xcos * ycos - xsin * ysin;
    phim = xsin * ycos + xcos * ysin;
    sumReal = tex1Dfetch(sSumTex, indIn);
    sumImag = tex1Dfetch(sSumTex, indIn + 1);
    shiftReal = phre * real - phim * imag;
    shiftImag = phim * real + phre * imag;
    sumArr[indIn] = sumReal + shiftReal;
    sumArr[indIn + 1] = sumImag + shiftImag;
    //sumArr[indIn] += phre * real - phim * imag;
    //sumArr[indIn + 1] += phim * real + phre * imag;
    fullArr[indIn] = shiftReal;
    fullArr[indIn + 1] = shiftImag;
  }
}

// Add the odd sum to the even sum
__global__ void addOddToEvenSum(float *evenArr, int sumXplus, int sumYpad)
{

  // textures did not speed this up
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < sumXplus && iy < sumYpad) {
    int ind = ix + sumXplus * iy;
    evenArr[ind] = tex1Dfetch(sSumTex, ind) + tex1Dfetch(sOddTex, ind);
  }
}

// Subtract one array from another, multiple by a filter mask, and store in sumArr
__global__ void subtractFilterSum(float *subArr, float *sumArr, int sumXplus, int sumYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < sumXplus && iy < sumYpad) {
    int ind = ix + sumXplus * iy;
    sumArr[ind] = (tex1Dfetch(sSumTex, ind) - subArr[ind]) * tex1Dfetch(sMaskTex, ind);
  }
}

// Apply filter mask to an FTT
__global__ void filterAlignFFT(float *fft, int alignXplus, int alignYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < alignXplus && iy < alignYpad) {
    int ind = ix + alignXplus * iy;
    fft[ind] *= tex1Dfetch(sMaskTex, ind);
  }
}

// Compute conjugate product of two FFTs and store in prod array
__global__ void conjugateProduct(float *array, float *brray, float *prod, int nxFFT,
                                 int alignYpad)
{
  int jx, jp1;
  float a, b, c, d;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < nxFFT && iy < alignYpad) {
    jx = 2 * (ix + nxFFT * iy);
    jp1 = jx + 1;
    a = array[jx];
    b = array[jp1];
    c = brray[jx];
    d = brray[jp1];
    prod[jx] = a * c + b * d;
    prod[jp1] = b * c - a * d;
  }
}

// Extract the corners of a cross-correlation image into small array with origin at center
__global__ void wrapCorners(float *fromArr, float *toArr, int nxFrom, int nyFrom, 
                            int nxTo, int nyTo, int ixFrom, int iyFrom)
{
  int fromX, fromY;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < nxFrom && iy < nyFrom) {
    fromX = (ix + ixFrom) % nxFrom;
    fromY = (iy + iyFrom) % nyFrom;
    toArr[ix + iy * nxTo] = tex1Dfetch(sFullTex, fromX + fromY * nxFrom);
  }
}

// Sum different numbers of FFTs into groups
__global__ void sum2IntoGroup(float *arr1, float *arr2, float *groupArr, int alignXplus,
                              int alignYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < alignXplus && iy < alignYpad) {
    int ind = ix + alignXplus * iy;
    groupArr[ind] = arr1[ind] + arr2[ind];
  }
}

__global__ void sum3IntoGroup(float *arr1, float *arr2, float * arr3, float *groupArr,
                              int alignXplus, int alignYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < alignXplus && iy < alignYpad) {
    int ind = ix + alignXplus * iy;
    groupArr[ind] = arr1[ind] + arr2[ind] + arr3[ind];
  }
}

__global__ void sum4IntoGroup(float *arr1, float *arr2, float *arr3, float *arr4, 
                              float *groupArr, int alignXplus, int alignYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < alignXplus && iy < alignYpad) {
    int ind = ix + alignXplus * iy;
    groupArr[ind] = arr1[ind] + arr2[ind] + arr3[ind] + arr4[ind];
  }
}

__global__ void sum5IntoGroup(float *arr1, float *arr2, float *arr3, float *arr4, 
                              float *arr5, float *groupArr, int alignXplus, int alignYpad)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix < alignXplus && iy < alignYpad) {
    int ind = ix + alignXplus * iy;
    groupArr[ind] = arr1[ind] + arr2[ind] + arr3[ind] + arr4[ind] + arr5[ind];
  }
}

// Macro for preprocessing a specific type with its texture
// Multiply by gain if doing that
// If pixel is above truncation limit or marked as a defect, average surrounding 
// box of 9x9 pixels excluding central 9, picking only pixels that pass the same test
#define PREPROC_PIXEL(mtyp, tex)                                        \
  case mtyp:                                                            \
  pixVal = tex2D(tex, ix, iy);                                          \
  if (doGain)                                                           \
    pixVal *= tex2D(sGainRefTex, ix, iy);                               \
  if ((truncLimit > 0 && pixVal > truncLimit) ||                        \
      (doDefects > 0 && tex2D(sDefectTex, ix, iy) > 0)) {               \
    for (delY = -4; delY <= 4; delY++) {                                \
      for (delX = -4; delX <= 4; delX++) {                              \
        if (delX >= -1 && delX <= 1 && delY >= -1 && delY <= 1)         \
          continue;                                                     \
        inX = ix + delX;                                                \
        inY = iy + delY;                                                \
        pixVal = tex2D(tex, inX, inY);                                  \
        if (doGain)                                                     \
          pixVal *= tex2D(sGainRefTex, inX, inY);                       \
        if ((truncLimit > 0 && pixVal > truncLimit) ||                  \
            (doDefects > 0 && tex2D(sDefectTex, inX, inY) > 0))         \
          continue;                                                     \
        sum += pixVal;                                                  \
        nsum++;                                                         \
      }                                                                 \
    }                                                                   \
    pixVal = sum / nsum;                                                \
  }                                                                     \
  break;

// Kernel to pre-process and writ e to the surface reference
__global__ void preprocessFrame(float *linearArr, int type, int nx, int ny, int doGain, 
                                float truncLimit, int doDefects)
{
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  float pixVal, sum = 0.;
  int nsum = 0, inX, inY, delX, delY;
  if (ix >= nx || iy >= ny)
    return;
  switch (type) {
    PREPROC_PIXEL(MRC_MODE_BYTE, sUnpadByteTex);
    PREPROC_PIXEL(MRC_MODE_SHORT, sUnpadShortTex);
    PREPROC_PIXEL(MRC_MODE_USHORT, sUnpadUShortTex);
    PREPROC_PIXEL(MRC_MODE_FLOAT, sUnpadFloatTex);
  }
#ifdef NO_SURFACES
  linearArr[ix + iy * nx] = pixVal;
#else
  surf2Dwrite(pixVal, sTempSurfRef, ix * sizeof(float), iy);
#endif
}

// Pad a full image with noise that tapers down to the mean
__global__ void noiseTaperPad(int type, int nxBox, int nyBox, float *outArr,
                              int nxDimOut, int nx, int ny, int noiseLength, 
                              int noiseRows, int cornerSize, float dmean, int seed)
{
  int padInX, padInY, outInd, imageX, imageY, outInX = 0, xNear, outInY = 0, yNear;
  float xAtten, yAtten, atten;
  int nxNoise, nyNoise, ixNoise, iyNoise, ixHigh, iyHigh, inX, inY;
  int indInBox, pseudo, ixInBox, iyInBox;
  int xDirToData = 1, yDirToData = 1;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix >= nx || iy >= ny)
    return;
  padInX = (nx - nxBox) / 2;
  padInY = (ny - nyBox) / 2;
  outInd = ix + iy * nxDimOut;
  imageX = ix - padInX;
  imageY = iy - padInY;
  xDirToData = 1;
  
  // Determine if it is outside the image, the direction toward data, and nearest pixel 
  // in each direction, as well as attenuation for the taper
  if (imageX < 0) {
    outInX = 1;
    xNear = 0;
    xAtten = (float)ix / (padInX + 1.f);
  }
  if (imageX >= nxBox) {
    outInX = 1;
    xDirToData = -1;
    xNear = nxBox - 1;
    xAtten = (float)(nx - ix - 1) / (padInX + 1.f);
  }
  if (imageY < 0)  {
    outInY = 1;
    yNear = 0;
    yAtten = (float)iy / (padInY + 1.);
  }
  if (imageY >= nyBox) {
    outInY = 1;
    yDirToData = -1;
    yNear = nyBox - 1;
    yAtten = (float)(ny - iy - 1) / (padInY + 1.f);
  }

  // If inside, we're done, copy the pixel
  if (!outInX && !outInY) {
    switch (type) {
    case MRC_MODE_BYTE:
      outArr[outInd] = tex2D(sUnpadByteTex, imageX, imageY);
      break;
    case MRC_MODE_SHORT:
      outArr[outInd] = tex2D(sUnpadShortTex, imageX, imageY);
      break;
    case MRC_MODE_USHORT:
      outArr[outInd] = tex2D(sUnpadUShortTex, imageX, imageY);
      break;
    default:
      outArr[outInd] = tex2D(sUnpadFloatTex, imageX, imageY);
      break;
    }
  } else {

    // Otherwise is in a corner
    if (outInX && outInY) {
      nxNoise = cornerSize;
      nyNoise = cornerSize;
      ixNoise = xNear;
      iyNoise = yNear;
      atten = max(xAtten, yAtten);

      // Or in left or right border, position box along edge
    } else if (outInX) {
      ixNoise = xNear;
      nxNoise = noiseRows;
      iyNoise = imageY - noiseLength / 2;
      
      // Expand box in other direction if it is near corner
      if (iyNoise < 0) {
        nxNoise = min(cornerSize, nxNoise - iyNoise);
        iyNoise = 0;
      }
      iyHigh = imageY + noiseLength / 2;  // One past
      if (iyHigh > nyBox) {
        nxNoise = min(cornerSize, nxNoise + (iyHigh - nyBox));
        iyHigh = nyBox;
      }
      nyNoise = iyHigh - iyNoise;
      atten = xAtten;

      // Or in top or bottom border, do same tests
    } else {
      iyNoise = yNear;
      nyNoise = noiseRows;
      ixNoise = imageX - noiseLength / 2;
      if (ixNoise < 0) {
        nyNoise = min(cornerSize, nyNoise - ixNoise);
        ixNoise = 0;
      }
      ixHigh = imageX + noiseLength / 2;  // One past
      if (ixHigh > nxBox) {
        nyNoise = min(cornerSize, nyNoise + (ixHigh - nxBox));
        ixHigh = nxBox;
      }
      nxNoise = ixHigh - ixNoise;
      atten = yAtten;
    } 

    // Get the random position, convert to an index in box and coordinates in box
    pseudo = (seed + 197 * (ix + 157) * (iy + 179)) & 0xFFFFF;
    pseudo = (197 * (pseudo + 1)) & 0xFFFFF;
    indInBox = (pseudo >> 4) % (nxNoise * nyNoise);
    ixInBox = indInBox % nxNoise;
    iyInBox = indInBox / nxNoise;
    inX = ixNoise + ixInBox * xDirToData;
    inY = iyNoise + iyInBox * yDirToData;
    switch (type) {
    case MRC_MODE_BYTE:
      outArr[outInd] = atten * (tex2D(sUnpadByteTex, inX, inY) - dmean) + dmean;
      break;
    case MRC_MODE_SHORT:
      outArr[outInd] = atten * (tex2D(sUnpadShortTex, inX, inY) - dmean) + dmean;
      break;
    case MRC_MODE_USHORT:
      outArr[outInd] = atten * (tex2D(sUnpadUShortTex, inX, inY) - dmean) + dmean;
      break;
    default:
      outArr[outInd] = atten * (tex2D(sUnpadFloatTex, inX, inY) - dmean) + dmean;
      break;
    }
  }
}


/*
 * Image reduction: column reduction macros and kernel, followed by row reduction kernel
 * This is an odd set of macros to minimize the number of lines in the column reduction 
 * kernel.  It saves about 25% to specify the unroll
 */
#define REDUCE_COL_BYTE                                                 \
  for (inX = 0; inX < numLoop; inX++)                                   \
    isum += intKernel[inX] * tex2D(sUnpadByteTex, inX + addXoffset, iySrc); \
  sum = isum / INT_KERNEL_SCALE;                                        \
  break;                                                                \
 case MRC_MODE_SHORT:

#define REDUCE_COL_SHORT                                                \
  for (inX = 0; inX < numLoop; inX++)                                   \
    isum += intKernel[inX] * tex2D(sUnpadShortTex, inX + addXoffset, iySrc); \
  sum = isum / INT_KERNEL_SCALE;                                        \
  break;                                                                \
 case MRC_MODE_USHORT:

#define REDUCE_COL_USHORT                                               \
  for (inX = 0; inX < numLoop; inX++)                                   \
    isum += intKernel[inX] * tex2D(sUnpadUShortTex, inX + addXoffset, iySrc); \
  sum = isum / INT_KERNEL_SCALE;                                        \
  break;                                                                \
 case MRC_MODE_FLOAT:

#define REDUCE_COL_FLOAT                                                \
  for (inX = 0; inX < numLoop; inX++)                                   \
    sum += trigTable[inX] * tex2D(sUnpadFloatTex, inX + addXoffset, iySrc); \
  break;

// Kernel to reduce portion of full image in X from different types of data, outputting
// floats through a surface reference
__global__ void reduceColumns(int binning, float *linearArr, int type, int nxOut,
                              int nyOut, int iyStart, int delXstart, int delXend) 
{
  int ixSrc, iySrc, isum = 0;
  int addXoffset, inX, numLoop = delXend + 1 - delXstart;
  float sum = 0.;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix >= nxOut || iy >= nyOut)
    return;
  
  // ixSrc is a source coordinate in full image relative to the mXstart and delX needs
  // to be added to get to the absolute source coord
  // iySrc is a absolute source coordinate in full image, and iyStart is mYstart minus 
  // half the support (kernel radius)
  ixSrc = ix * binning;
  iySrc = iy + iyStart;
  
  addXoffset = ixSrc + delXstart;
  if (numLoop >= 40) {
    switch (type) {
    case MRC_MODE_BYTE:
#pragma unroll 40
      REDUCE_COL_BYTE;
#pragma unroll 40
      REDUCE_COL_SHORT;
#pragma unroll 40
      REDUCE_COL_USHORT;
#pragma unroll 40
      REDUCE_COL_FLOAT;
    }
  } else if (numLoop >= 32) {
    switch (type) {
    case MRC_MODE_BYTE:
#pragma unroll 32
      REDUCE_COL_BYTE;
#pragma unroll 32
      REDUCE_COL_SHORT;
#pragma unroll 32
      REDUCE_COL_USHORT;
#pragma unroll 32
      REDUCE_COL_FLOAT;
    }
  } else if (numLoop >= 24) {
    switch (type) {
    case MRC_MODE_BYTE:
#pragma unroll 24
      REDUCE_COL_BYTE;
#pragma unroll 24
      REDUCE_COL_SHORT;
#pragma unroll 24
      REDUCE_COL_USHORT;
#pragma unroll 24
      REDUCE_COL_FLOAT;
    }
  } else if (numLoop >= 16) {
    switch (type) {
    case MRC_MODE_BYTE:
#pragma unroll 16
      REDUCE_COL_BYTE;
#pragma unroll 16
      REDUCE_COL_SHORT;
#pragma unroll 16
      REDUCE_COL_USHORT;
#pragma unroll 16
      REDUCE_COL_FLOAT;
    }
  } else if (numLoop >= 8) {
    switch (type) {
    case MRC_MODE_BYTE:
#pragma unroll 8
      REDUCE_COL_BYTE;
#pragma unroll 8
      REDUCE_COL_SHORT;
#pragma unroll 8
      REDUCE_COL_USHORT;
#pragma unroll 8
      REDUCE_COL_FLOAT;
    }
  } else {
    switch (type) {
    case MRC_MODE_BYTE:
      REDUCE_COL_BYTE;
      REDUCE_COL_SHORT;
      REDUCE_COL_USHORT;
      REDUCE_COL_FLOAT;
    }
  }
  
#ifdef NO_SURFACES
  linearArr[ix + iy * nxOut] = sum;
#else
  surf2Dwrite(sum, sTempSurfRef, ix * sizeof(float), iy);
#endif
}

// Second kernel to reduce the X-reduced image in Y.  This is much quicker (36 ms vs 
// 221 ms for X reduction) and the unroll actuaaly cost a bit of time
__global__ void reduceRowsTaperPad(float *outArr, int binning, int inYstart,
                                   int ixLow, int ixHigh, int iyLow, int iyHigh,
                                   int nxDimOut, int alignXpad, int alignYpad, 
                                   int nxTaper, int nyTaper,
                                   float dmean, int delYstart, int delYend)
{
  int outInd, ixSrc, iySrc;
  int addYoffset, inY, numLoop = delYend + 1 - delYstart;
  float fracX, fracY, fmin, sum;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix >= alignXpad || iy >= alignYpad)
    return;
  outInd = ix + nxDimOut * iy;
  if (ix < ixLow || ix >= ixHigh || iy < iyLow || iy >= iyHigh) {
    outArr[outInd] = dmean;
    return;
  }

  // ixSrc is absolute coordinate in X-reduced image
  // iySrc is location in full image relative to mYstart there, adjusted downward by
  // start of X-reduced image in Y (mYstart - half-support), so when delY is added it
  // will be location in X-reduced image
  ixSrc = ix - ixLow;
  iySrc = (iy - iyLow) * binning - inYstart;

  addYoffset = iySrc + delYstart;
  sum = 0;
  for (inY = 0; inY < numLoop; inY++)
    sum += trigTable[inY] * tex2D(sUnpadFloatTex, ixSrc, inY + addYoffset);
  fracX = 1.;
  fracY = 1.;
  if (ix < nxTaper + ixLow)
    fracX = (ix + 1.f - ixLow) / (nxTaper + 1.);
  else if (ix >= ixHigh - nxTaper)
    fracX = (ixHigh - ix) / (nxTaper + 1.);
  if (iy < nyTaper + iyLow)
    fracY = (iy + 1.f - iyLow) / (nyTaper + 1.);
  else if (iy >= iyHigh - nyTaper)
    fracY = (iyHigh - iy) / (nyTaper + 1.);
  if (fracX < 1 || fracY < 1.) {
    fmin = min(fracX, fracY);
    outArr[outInd] = fmin * (sum - dmean) + dmean;
  } else {
    outArr[outInd] = sum;
  }
}

// Kernel to simply trim, pad, and taper inside when there is no reduction
__global__ void trimTaperPad(float *outArr, int type, int ixLow, int ixHigh, int iyLow,
                             int iyHigh, int nxDimOut, int alignXpad, int alignYpad,
                             int nxTaper, int nyTaper, float dmean)
{
  int outInd, ixSrc, iySrc;
  float fracX, fracY, fmin, sum;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix >= alignXpad || iy >= alignYpad)
    return;
  outInd = ix + nxDimOut * iy;
  if (ix < ixLow || ix >= ixHigh || iy < iyLow || iy >= iyHigh) {
    outArr[outInd] = dmean;
    return;
  }
  ixSrc = (ix + ixLow);
  iySrc = (iy + iyLow);

  // Does reduction on CPU reach outside the subarea being computed?
  switch (type) {
  case MRC_MODE_BYTE:
    sum = tex2D(sUnpadByteTex, ixSrc, iySrc);
    break;
  case MRC_MODE_SHORT:
    sum = tex2D(sUnpadShortTex, ixSrc, iySrc);
    break;
  case MRC_MODE_USHORT:
    sum = tex2D(sUnpadUShortTex, ixSrc, iySrc);
    break;
  default:
    sum = tex2D(sUnpadFloatTex, ixSrc, iySrc);
    break;
  }

  fracX = 1.;
  fracY = 1.;
  if (ix < nxTaper + ixLow)
    fracX = (ix + 1.f - ixLow) / (nxTaper + 1.);
  else if (ix >= ixHigh - nxTaper)
    fracX = (ixHigh - ix) / (nxTaper + 1.);
  if (iy < nyTaper + iyLow)
    fracY = (iy + 1.f - iyLow) / (nyTaper + 1.);
  else if (iy >= iyHigh - nyTaper)
    fracY = (iyHigh - iy) / (nyTaper + 1.);
  if (fracX < 1 || fracY < 1.) {
    fmin = min(fracX, fracY);
    outArr[outInd] = fmin * (sum - dmean) + dmean;
  } else {
    outArr[outInd] = sum;
  }
}


/////////////////////////////////////////////////////////////////////////////
// THE FrameGPU CLASS
/////////////////////////////////////////////////////////////////////////////

FrameGPU::FrameGPU()
{
  sChanDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  sByteChanDesc = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
  sShortChanDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);
  sUShortChanDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
  clearAllItems();
  // framealign calls zeroTimers which is implemented in .h file
}

/*
 * complete cleanup of both functionalities
 */
void FrameGPU::cleanup()
{
  B3DFREE(mXshiftTrig);
  B3DFREE(mYshiftTrig);
  cleanAlignItems();
  cleanSumItems();
  cleanPreProc();
  clearAllItems();
}

/*
 * Clean up and free resources for alignment
 */
void FrameGPU::cleanAlignItems()
{
  unbindVariableBindings();
  destroyPlan(sAlignForwardPlan);
  destroyPlan(sAlignInversePlan);
  freeCudaStack(mSavedBinPad);
  freeCudaStack(mSavedGroups);
  freeCudaArray(&mWorkBinPad);
  freeCudaArray(&mCorrBinPad);
  freeCudaArray(&mAlignSum);
  freeCudaArray(&mSubareaCorr);
  freeCudaArray(&mRealCorr);
  if (mFiltMask) {
    freeCudaArray(&mFiltMask);
    hipUnbindTexture(sMaskTex);
  }
  B3DFREE(mHostSubarea);
  free2Darray(&sReducedInXarray);
  freeCudaArray(&mReducedInXlinear);
  mRedColX = mRedColY = 0;
  mAlignXpad = mAlignYpad = 0;
  mAliFiltSize = 0;
  mUnpaddedBytes = 0;
  mDoAlignBinPad = 0;
}

/*
 * Clean up and free resources for summing
 */
void FrameGPU::cleanSumItems()
{
  unbindVariableBindings();
  freeCudaArray(&mWorkFullSize);
  freeCudaArray(&mEvenSum);
  freeCudaArray(&mOddSum);
  freeCudaArray(&mNonDWsum);
  if (mDoseWgtFilter)
    hipUnbindTexture(sDWFilterTex);
  for (int ind = 0; ind < (int)sSavedUnpadded.size(); ind++)
    free2Darray(&sSavedUnpadded[ind]);
  sSavedUnpadded.clear();
  mSavedFrameNums.clear();
  mUnpadEdgeMeans.clear();
  mNumOnUnpadStack = 0;
  freeCudaArray(&mDoseWgtFilter);
  destroyPlan(sFullForwardPlan);
  destroyPlan(sSumInversePlan);
  mFullXpad = mFullYpad = 0;
  mSumXpad = mSumYpad = 0;
  mDoNoiseTaper = 0;
}

/*
 * Free up the two bindings that can be set to different arrays
 */
void FrameGPU::unbindVariableBindings()
{
  if (mBoundToFull)
    hipUnbindTexture(sFullTex);
  if (mBoundSum >= 0)
    hipUnbindTexture(sSumTex);
  mBoundToFull = NULL;
  mBoundSum = -1;
}

/*
 * Cleans up pre-processing arrays
 */
void FrameGPU::cleanPreProc()
{
  if (sGainRefArray) {
    hipUnbindTexture(sGainRefTex);
    free2Darray(&sGainRefArray);
    sGainRefArray = NULL;
  }
  if (sDefectMapArray) {
    hipUnbindTexture(sDefectTex);
    free2Darray(&sDefectMapArray);
  }
  free2Darray(&sTempFloatArray);
  freeCudaArray(&mProcessedLinear);
  mNxGain = 0;
  mNyGain = 0;
  mCamSizeX = 0;
  mCamSizeY = 0;
  mTruncLimit = 0.;
  mTempFloatSizeX = 0;
  mTempFloatSizeY = 0;
}

/*
 * Initializes all pointers and tested member variables on cleanup or construction
 */
void FrameGPU::clearAllItems()
{
  mWorkFullSize = NULL;
  mEvenSum = NULL;
  mOddSum = NULL;
  mNonDWsum = NULL;
  mDoseWgtFilter = NULL;
  mXshiftTrig = NULL;
  mYshiftTrig = NULL;
  mWorkBinPad = NULL;
  mCorrBinPad = NULL;
  mFiltMask = NULL;
  mAlignSum = NULL;
  mSubareaCorr = NULL;
  mRealCorr = NULL;
  mBoundToFull = NULL;
  mHostSubarea = NULL;
  mReducedInXlinear = NULL;
  mProcessedLinear = NULL;
#ifdef NO_SURFACES
  mNoSurfaces = 1;
#else
  mNoSurfaces = 0;
#endif
  mAliFiltSize = 0;
  mFullXpad = mFullYpad = 0;
  mSumXpad = mSumYpad = 0;
  mAlignXpad = mAlignYpad = 0;
  mBoundSum = -1;
  mXtrigSize = 0;
  mYtrigSize = 0;
  mGroupSize = 1;
  mDWFilterDelta = 0.;
  mDWFilterSize = 0;
  mNxGain = 0;
  mNyGain = 0;
  mCamSizeX = 0;
  mCamSizeY = 0;
  mTruncLimit = 0.;
  mTempFloatSizeX = 0;
  mTempFloatSizeY = 0;
  free2Darray(&sTempRawArray);
  mNxRaw = 0;
  mNyRaw = 0;
  mNumOnUnpadStack = 0;
  mRedColX = mRedColY = 0;
}

// Free one regular array, set its pointer to NULL
void FrameGPU::freeCudaArray(float **array)
{
  if (*array)
    hipFree(*array);
  *array = NULL;
}

/* convenience function to free a stack of regular arrays */
void FrameGPU::freeCudaStack(std::vector<float *> &saved)
{
  for (int ind = 0; ind < saved.size(); ind++)
    freeCudaArray(&saved[ind]);
  saved.clear();
}

// Free a 2D array and set its pointer to null
static void free2Darray(hipArray **array)
{
  if (*array)
    hipFreeArray(*array);
  *array = NULL;
}

// Destroy an FFT plan and set to 0
static void destroyPlan(hipfftHandle &plan)
{
  if (plan)
    hipfftDestroy(plan);
  plan = 0;
}

/*
 * Test whether a GPU is available, either a GPU of the given number if nGPU is
 * > 0, or the one with the best processing rate if nGPU is 0, and return the
 * memory in bytes.  Return value is 1 for success, 0 for failure.
 */
int FrameGPU::gpuAvailable(int nGPU, float *memory, int debug)
{
  int current_device = 0;
  int device_count = 0;
  int totalCores;
  float gflops;
  struct hipDeviceProp_t device_properties, best_properties;

  // The Mac mini comes through with a clock rate of 0 so allow a 0 product
  float max_gflops = -1.;
  mDebug = debug;
  *memory = 0;
  hipGetDeviceCount( &device_count );
  if (debug) {
#if CUDA_VERSION >= 3000
    int version, version2;
    hipRuntimeGetVersion(&version2);
    hipDriverGetVersion(&version);
    utilPrint("CUDA version - driver: %d.%02d  runtime: %d.%02d\n", version / 1000,
              version % 1000, version2 / 1000, version2 % 1000);
#endif
    utilPrint("Device count = %d\n", device_count);
  }
  if (nGPU != 0) {
    if (nGPU < 0 || nGPU > device_count) {
      utilPrint("The requested GPU number, %d, is out of range; there are only %d "
                "devices\n", nGPU, device_count);
      return 0;
    }
    current_device = nGPU - 1;
    device_count = nGPU;
  }
  for (; current_device < device_count; current_device++) {
    if (hipGetDeviceProperties( &device_properties, current_device)
        != hipSuccess) {
      pflerr("Error returned from trying to get properties of GPU device %d",
             current_device);
      return 0;
    }
    totalCores = totalCudaCores(device_properties.major, device_properties.minor,
                                device_properties.multiProcessorCount);
    if (debug) {
      utilPrint("Device %d (%s): mp %d cores %d  cr %d  major %d minor %d  mem %.0f",
                current_device, device_properties.name,
                device_properties.multiProcessorCount, totalCores,
                device_properties.clockRate, device_properties.major,
                device_properties.minor, (float)device_properties.totalGlobalMem);
#if CUDA_VERSION >= 4000
      utilPrint("  tex1d %d", device_properties.maxTexture1DLinear);
#endif
      utilPrint("\n");
    }
    gflops = totalCores * (float)device_properties.clockRate;

    // Exclude emulation mode (?) which shows up on the Mac
    if( gflops > max_gflops && device_properties.major != 9999) {
      max_gflops = gflops;
      mMax_gflops_device = current_device;
      best_properties = device_properties;
    }
  }
    
  if (mMax_gflops_device >= 0) {
    *memory = best_properties.totalGlobalMem;
    if (hipSetDevice(mMax_gflops_device) != hipSuccess) {
      pflerr("Error selecting GPU device %d", mMax_gflops_device + 1);
      return 0;
    }
    mDeviceSelected = 1;
    return 1;
  }
  return 0;

}

/*
 * Set parameters and arrays for pre-processing if any.
 * This needs to be called (only) on first frame of a set since processing parameters
 * are not available in framealign::initialize
 */
int FrameGPU::setPreProcParams(float *gainRef, int nxGain, int nyGain, float truncLimit,
                               unsigned char *defectMap, int camSizeX, int camSizeY)
{
  int err = 0;

  // Set flags for actions to do based on what has been passed in
  mCorrectDefects = (defectMap != NULL && camSizeX > 0)  ? 1 : 0;
  mDoGainNorm = (gainRef != NULL && nxGain > 0) ? 1 : 0;
  mDoPreprocess = (mCorrectDefects || mDoGainNorm || truncLimit > 0) ? 1 : 0;

  // Cleanup if anything has changed
  if (!mDoPreprocess || nxGain != mNxGain || nyGain != mNyGain || 
      (mNxGain > 0 && !mDoGainNorm) || camSizeX != mCamSizeX || camSizeY != mCamSizeY || 
      (mCamSizeX > 0 && !mCorrectDefects))
    cleanPreProc();

  // Setup gain normalization
  if (mDoGainNorm && !mNxGain) {
    if (testErrCode(hipMallocArray(&sGainRefArray, &sChanDesc, nxGain, nyGain),
                    "allocate gain reference array on GPU", 0)) {
      err = 1;
    } else {
    
      // Bind texture
      SETUP_TEXTURE(sGainRefTex);
      if (testErrCode(hipBindTextureToArray(sGainRefTex, sGainRefArray, sChanDesc),
                      "bind gain reference array to texture", 0)) {
        err = 1;
      } else {
        mNxGain = nxGain;
        mNyGain = nyGain;

        // Copy to array
        if (testErrCode(hipMemcpyToArray(sGainRefArray, 0, 0, gainRef, nxGain * nyGain * 
                                          sizeof(float), hipMemcpyHostToDevice),
                        "copy gain reference to GPU", 0))
          err = 1;
      }
    }
    //dumpUnpadArray(sGainRefArray, nxGain, nyGain, MRC_MODE_FLOAT, "gain reference");
  }

  // Set up defect correction
  if (!err && mCorrectDefects && camSizeX != mCamSizeX) {
    if (testErrCode(hipMallocArray(&sDefectMapArray, &sByteChanDesc, mUnpaddedX, 
                                    mUnpaddedY) , "allocate defect map array on GPU", 0)){
      err = 1;
    } else {

      // Bind texture
      SETUP_TEXTURE(sDefectTex);
      if (testErrCode(hipBindTextureToArray(sDefectTex, sDefectMapArray, sByteChanDesc),
                      "bind defect map array to texture", 0)) {
        err = 1;
      } else {
        mCamSizeX = camSizeX;
        mCamSizeY = camSizeY;
        
        // Copy to array
        if (testErrCode(hipMemcpyToArray(sDefectMapArray, 0, 0, defectMap, mUnpaddedX * 
                                          mUnpaddedY, hipMemcpyHostToDevice),
                        "copy defect map to GPU", 0))
          err = 1;
      }
    }
  }

  // Need a 2D temp float array if processing at all
  if (!err && mDoPreprocess && 
      (!sTempFloatArray || (mNoSurfaces && !mProcessedLinear) || 
       mUnpaddedX != mTempFloatSizeX || mUnpaddedY != mTempFloatSizeY)) {
    free2Darray(&sTempFloatArray);
    freeCudaArray(&mProcessedLinear);
#ifdef NO_SURFACES
    if (hipMallocArray(&sTempFloatArray, &sChanDesc, mUnpaddedX, mUnpaddedY) != 
        hipSuccess || hipMalloc(&mProcessedLinear, mUnpaddedX * mUnpaddedY * 
                                  sizeof(float)) != hipSuccess) {
      pflerr("allocate temp float 2D array or linear array on GPU");
      err = 1;
#else
    if (testErrCode(hipMallocArray(&sTempFloatArray, &sChanDesc, mUnpaddedX, mUnpaddedY,
                                    hipArraySurfaceLoadStore),
                    "allocate temp float 2D array on GPU", 0)) {
      err = 1;
#endif
    } else {
      mTempFloatSizeX = mUnpaddedX;
      mTempFloatSizeY = mUnpaddedY;
    }
  }
  mTruncLimit = truncLimit;

  // An error at this stack can be handled ny simply canceling all the initial GPU 
  // operations
  if (err) {
    cleanPreProc();
    mDoNoiseTaper = mDoAlignBinPad = mDoPreprocess = mStackUnpadded = 0;
  }
  return err;
}

/*
 * Allocate a 2D array of the given type and size
 */
static hipError_t malloc2Darray(hipArray **arrayPtr, int type, int sizeX, int sizeY)
{
  switch (type) {
  case MRC_MODE_BYTE:
    return(hipMallocArray(arrayPtr, &sByteChanDesc, sizeX, sizeY));
  case MRC_MODE_SHORT:
    return(hipMallocArray(arrayPtr, &sShortChanDesc, sizeX, sizeY));
  case MRC_MODE_USHORT:
    return(hipMallocArray(arrayPtr, &sUShortChanDesc, sizeX, sizeY));
  case MRC_MODE_FLOAT:
    return(hipMallocArray(arrayPtr, &sChanDesc, sizeX, sizeY));
  }
  return hipErrorUnknown;
}

/*
 * Bind one of the unpadded 2D arrays to a texture of the right type
 */
static hipError_t bindUnpadArray(hipArray *arrayPtr, int type)
{
  switch (type) {
  case MRC_MODE_BYTE:
    SETUP_TEXTURE(sUnpadByteTex);
    return(hipBindTextureToArray(sUnpadByteTex, arrayPtr));
  case MRC_MODE_SHORT:
    SETUP_TEXTURE(sUnpadShortTex);
    return(hipBindTextureToArray(sUnpadShortTex, arrayPtr));
  case MRC_MODE_USHORT:
    SETUP_TEXTURE(sUnpadUShortTex);
    return(hipBindTextureToArray(sUnpadUShortTex, arrayPtr));
  case MRC_MODE_FLOAT:
    SETUP_TEXTURE(sUnpadFloatTex);
    return(hipBindTextureToArray(sUnpadFloatTex, arrayPtr));
  }
  return hipErrorUnknown;
}

/*
 * Unbind an unpadded 2D array of the given type
 */
void FrameGPU::unbindUnpadArray(int type2d)
{
  switch (type2d) {
  case MRC_MODE_BYTE:
    hipUnbindTexture(sUnpadByteTex);
    break;
  case MRC_MODE_SHORT:
    hipUnbindTexture(sUnpadShortTex);
    break;
  case MRC_MODE_USHORT:
    hipUnbindTexture(sUnpadUShortTex);
    break;
  case MRC_MODE_FLOAT:
    hipUnbindTexture(sUnpadFloatTex);
    break;
  }
}

/*
 * Set parameters for the reduction tapering and the length parameter for noise tapering
 * This can be called on all frames of a set and should always be called, regardless of
 * whether doing binpad
 */
void FrameGPU::setBinPadParams(int xstart, int xend, int ystart, int yend, int binning,
                               int nxTaper, int nyTaper, int type, int filtType,
                               int noiseLen)
{
  mXstart = xstart;
  mXend = xend;
  mYstart = ystart;
  mYend = yend;
  mAliBinning = binning;
  mNxTaper = nxTaper;
  mNyTaper = nyTaper;
  mStackType = type;
  mNoiseLength = noiseLen;
  mAntiFiltType = filtType;
}

/*
 * Allocate a temporary 2D array for raw data of the current "stack" type when there
 * is no stack or the stack is not big enough; and also an X-reduced array when
 * doing reduction - padding,
 * This has to get called by summing and aligning as it is needed independently by
 * those two operations
 */
int FrameGPU::manageRawTempArray(int aligning)
{
  int support, redColY;
  int redColX = (mXend + 1 - mXstart) / mAliBinning;

  // Need a "raw" temporary array if not stacking 
  if ((mDoAlignBinPad || mDoNoiseTaper) && (!mStackUnpadded || mStackIsLimited)) {
    if (!sTempRawArray || mNxRaw != mUnpaddedX || mNyRaw != mUnpaddedY) {
      free2Darray(&sTempRawArray);
      if (testErrCode(malloc2Darray(&sTempRawArray, mStackType, mUnpaddedX, mUnpaddedY), 
                      "allocate raw 2D temp array on GPU", 1))
        return 1;
      mNxRaw = mUnpaddedX;
      mNyRaw = mUnpaddedY;
    }
  } else {
    free2Darray(&sTempRawArray);
    mNxRaw = mNyRaw = 0;
  }

  // Manage column-reduced array(s) for reducing
  if (mDoAlignBinPad && mAliBinning > 1) {
    if (aligning) {
      selectZoomFilter(mAntiFiltType, 1. / mAliBinning, &support);
      redColY = (mYend + 1 - mYstart) + 2 * (support / 2 + 1);
      if (!sReducedInXarray || mRedColX != redColX || mRedColY != redColY ||
          (mNoSurfaces && !mReducedInXlinear)) {
        free2Darray(&sReducedInXarray);
        freeCudaArray(&mReducedInXlinear);
#ifdef NO_SURFACES
        if (testErrCode(hipMallocArray(&sReducedInXarray, &sChanDesc, redColX, redColY),
                        "allocate row reduction 2D array on GPU", 1))
          return 1;
        if (testErrCode(hipMalloc(&mReducedInXlinear, redColX * redColY * sizeof(float)),
                        "allocate row reduction output array on GPU", 1))
          return 1;
#else
        if (testErrCode(hipMallocArray(&sReducedInXarray, &sChanDesc, redColX, redColY,
                                        hipArraySurfaceLoadStore),
                        "allocate row reduction 2D array on GPU", 1))
          return 1;
#endif
        mRedColX = redColX;
        mRedColY = redColY;
      }
    }
  } else {
    free2Darray(&sReducedInXarray);
    mRedColX = mRedColY = 0;
    freeCudaArray(&mReducedInXlinear);
  }
  return 0;
}

/*
 * Set some basic parameters for the initial processing steps
 * This needs to be called before setup of align OR summing; summiong setup is
 * done in two places in framealign
 */
void FrameGPU::setUnpaddedSize(int unpadX, int unpadY, int flags, int debug)
{
  mUnpaddedX = unpadX;
  mUnpaddedY = unpadY;
  mDoNoiseTaper = (flags & GPU_DO_NOISE_TAPER) ? 1 : 0;
  mDoAlignBinPad = (flags & GPU_DO_BIN_PAD) ? 1 : 0;
  mStackUnpadded = (flags & STACK_FULL_ON_GPU) ? 1 : 0;
  mStackIsLimited = (flags & GPU_STACK_LIMITED) ? 1 : 0;
  mDebug = debug % 10;
  mTrackTime = (debug / 10) % 10;
}

/*
 * Set parameters and initialize for summing aligned images into one or two arrays,
 * allocating new arrays as needed
 */
int FrameGPU::setupSumming(int fullXpad, int fullYpad, int sumXpad,
                           int sumYpad, int evenOdd)
{
  //size_t workSize;
  int error;
  int nonDW = evenOdd & 2;
  bool sumChanged = sumXpad != mSumXpad || sumYpad != mSumYpad;
  evenOdd &= 1;
  if (!mDeviceSelected)
    return -1;

  // Do not clean up on any failures, so that deferred summing can recover the stack
  // proceed on CPU.  Initial framealign setupSumming does full cleanup

  // Manage work array if size has changed
  mFullBytes = (fullXpad + 2) * fullYpad * sizeof(float);
  if (fullXpad != mFullXpad || fullYpad != mFullYpad) {
    freeCudaArray(&mWorkFullSize);
    if (testErrCode(hipMalloc((void **)&mWorkFullSize, mFullBytes),
                    "allocate full work array on GPU", 0))
      return 1;
  }

  // Manage even sum array if size has changed
  mSumBytes = (sumXpad + 2) * sumYpad * sizeof(float);
  if (sumChanged) {
    freeCudaArray(&mEvenSum);
    if (testErrCode(hipMalloc((void **)&mEvenSum, mSumBytes),
                    "allocate array for main sum on GPU", 0))
      return 1;

    SETUP_TEXTURE(sSumTex);
    if (bindSumArray(0)) {
      return 1;
    }
  }

  // Manage odd sum array if size has changed or need for it has changed
  if (sumChanged || evenOdd != mDoEvenOdd) {
    freeCudaArray(&mOddSum);
    if (evenOdd && testErrCode(hipMalloc((void **)&mOddSum, mSumBytes),
                               "allocate array for odd sum on GPU", 0))
      return 1;
  }

  // Manage non-dose weight sum array if size or need for it has changed
  if (sumChanged || nonDW != mDoUnDWsum) {
    freeCudaArray(&mNonDWsum);
    if (nonDW && testErrCode(hipMalloc((void **)&mNonDWsum, mSumBytes),
                             "allocate array for unweighted sum on GPU", 0))
      return 1;
  }

  // Manage FFT plans;
  destroyPlan(sFullForwardPlan);
  destroyPlan(sSumInversePlan);
  //START_TIMER;
  /* Not available in CUDA 4:
  size_t workSize;
  hipfftEstimate2d(fullYpad, fullXpad, HIPFFT_R2C, &workSize);
  utilPrint("Work size estimate for forward  %u\n", workSize); */
  error = hipfftPlan2d(&sFullForwardPlan, fullYpad, fullXpad, HIPFFT_R2C);
  if (error != HIPFFT_SUCCESS) {
    utilPrint("Failed to make plan for full forward FFT (error %d)\n", error);
    return 1;
  }
  //if (mDebug)
  //PRINT2("plan time: ", wallTime() - mWallStart);
  /*hipfftGetSize2d(sFullForwardPlan, fullYpad, fullXpad, HIPFFT_R2C, &workSize);
    utilPrint("Work size for forward plan %u\n", workSize);*/

  if (manageShiftTrigs(sumXpad, sumYpad)) {
    return 1;
  }

  // Clear sum arrays
  if (hipMemset(mEvenSum, 0, mSumBytes) != hipSuccess || 
      (evenOdd && hipMemset(mOddSum, 0, mSumBytes) != hipSuccess) || 
      (nonDW && hipMemset(mNonDWsum, 0, mSumBytes) != hipSuccess)) {
    pflerr("Failed to zero out sum array on GPU");
    return 1;
  }
  
  mFullXpad = fullXpad;
  mFullYpad = fullYpad;
  mSumXpad = sumXpad;
  mSumYpad = sumYpad;
  mDoEvenOdd = evenOdd;
  mDoUnDWsum = nonDW;
  mNumFramesSummed = 0;
  mNumAlignedFrames = 0;
  mDWFilterSize = 0;
  return 0;
}

/*
 * Set parameters and initialize for aligning by a particular strategy, allocating the
 * appropriate arrays
 */
int FrameGPU::setupAligning(int alignXpad, int alignYpad, int sumXpad, int sumYpad,
                            float *alignMask, int aliFiltSize, int groupSize, 
                            int expectStackSize, int doAlignSum)
{
  int error;
  int xpad = sumXpad;
  int ypad = sumYpad;

  if (doAlignSum || !expectStackSize) {
    xpad = B3DMAX(sumXpad, alignXpad);
    ypad = B3DMAX(sumYpad, alignYpad);
  }
  if (xpad && manageShiftTrigs(xpad, ypad)) {
    cleanup();
    return 1;
  }

  // Manage align arrays etc if size has changed
  if (alignXpad != mAlignXpad || alignYpad != mAlignYpad || groupSize != mGroupSize ||
      expectStackSize != mExpectStackSize || doAlignSum != mDoAlignSum) {

    // Cleanup existing
    error = mDoAlignBinPad;
    cleanAlignItems();
    mDoAlignBinPad = error;
    
    // Allocate arrays that are needed
    mAlignBytes = (alignXpad + 2) * alignYpad * sizeof(float);
    if (hipMalloc((void **)&mWorkBinPad, mAlignBytes) != hipSuccess ||
        hipMalloc((void **)&mFiltMask, mAlignBytes) != hipSuccess ||
        ((doAlignSum || !expectStackSize) && 
         (hipMalloc((void **)&mAlignSum, mAlignBytes) != hipSuccess ||
          hipMalloc((void **)&mCorrBinPad, mAlignBytes) != hipSuccess)) ||
        hipMalloc((void **)&mRealCorr, alignXpad * alignYpad * sizeof(float)) !=
        hipSuccess) {
      pflerr("Failed to allocate arrays for aligning on GPU");
      cleanup();
      return 1;
    }

    // Make plans
    error = hipfftPlan2d(&sAlignForwardPlan, alignYpad, alignXpad, HIPFFT_R2C);
    if (error == HIPFFT_SUCCESS)
      error = hipfftPlan2d(&sAlignInversePlan, alignYpad, alignXpad, HIPFFT_C2R);
    if (error != HIPFFT_SUCCESS) {
      utilPrint("Failed to make plan for align FFTs (error %d)\n", error);
      cleanup();
      return 1;
    }
    
    // Copy mask array
    START_TIMER;
    if (testErrCode(hipMemcpy(mFiltMask, alignMask, mAlignBytes, hipMemcpyHostToDevice),
                    "copy filter mask to GPU array", 1))
      return 1;
    ADD_TIME(mWallCopy);
    
    // Bind filter mask
    sMaskTex.filterMode = hipFilterModePoint;
    sMaskTex.normalized = false;
    if (testErrCode(hipBindTexture(NULL, sMaskTex, mFiltMask, sChanDesc, mAlignBytes),
                    "bind filter mask array to texture", 1))
      return 1;
  }
    
  // Get arrays for extracting subarea from correlation, both on device and host make
  // it a multiple of a nice size
  if (!mSubareaCorr || !mHostSubarea || aliFiltSize != mAliFiltSize) {
    mBigSubareaSize = NICE_GPU_DIVISOR * (aliFiltSize / NICE_GPU_DIVISOR + 1);
    freeCudaArray(&mSubareaCorr);
    B3DFREE(mHostSubarea);
    if (testErrCode(hipMalloc((void **)&mSubareaCorr, 
                               mBigSubareaSize * mBigSubareaSize * sizeof(float)),
                    "allocate array for subarea of correlation on GPU", 1))
      return 1;
    mHostSubarea = B3DMALLOC(float, mBigSubareaSize * mBigSubareaSize);
    if (!mHostSubarea) {
      utilPrint("Failed to allocate memory for oversized subarea\n");
      cleanup();
      return 1;
    }
  }
  
  // Clear sum arrays
  if ((doAlignSum || !expectStackSize) && clearAlignSum()) {
    cleanup();
    return 1;
  }
  
  mAlignXpad = alignXpad;
  mAlignYpad = alignYpad;
  mExpectStackSize = expectStackSize;
  mDoAlignSum = doAlignSum;
  mAliFiltSize = aliFiltSize;
  mGroupSize = groupSize;
  mNumOnUnpadStack = 0;
  mNumFramesSummed = 0;
  mNumAlignedFrames = 0;
  return 0;
}

/*
 * Set the dose weighting filter to use on the next final summing operation.  This can
 * be called before the addToSums call even when not dose weighting, although the filter
 * size is initialized to 0 when setting up
 */
int FrameGPU::setupDoseWeighting(float *filter, int filtSize, float delta)
{
  std::vector<float> filtSubset;
  int numBytes;
  mDWFilterSize = filtSize;
  if (!filtSize) {
    mDWFilterDelta = 0.;
    return 0;
  }

  mDWFilterDelta = delta;
  numBytes = mDWFilterSize * sizeof(float);
  if (!mDoseWgtFilter) {
    if (testErrCode(hipMalloc((void **)&mDoseWgtFilter, numBytes),
                    "allocate array for dose weight filter on GPU", 0))
      return 1;

    sDWFilterTex.addressMode[0] = hipAddressModeClamp;
    sDWFilterTex.addressMode[1] = hipAddressModeClamp;
    sDWFilterTex.filterMode = hipFilterModeLinear;
    sDWFilterTex.normalized = false;
    if (testErrCode(hipBindTexture(NULL, sDWFilterTex, mDoseWgtFilter, sChanDesc,
                                    numBytes),
                    "bind dose weight filter array to texture", 0))
      return 1;
  }

  // Copy the filter
  if (testErrCode(hipMemcpy(mDoseWgtFilter, filter, numBytes, hipMemcpyHostToDevice),
                  "copy dose weight filter to GPU array", 0))
    return 1;
  return 0;
}

/*
 * Zero out the sum for adding up an alignment reference
 */
int FrameGPU::clearAlignSum()
{
  if (testErrCode(hipMemset(mAlignSum, 0, mAlignBytes),
                  "zero out align sum array on GPU", 0))
    return 1;
  return 0;
}

/*
 * Take the FFT of a full image and shift add it to the appropriate sum
 */
int FrameGPU::addToFullSum(float *fullArr, float shiftX, float shiftY)
{
  float *sumArr = mEvenSum;
  int err, dataSize, ind, needBound = 0;
  hipArray *dev2dArr;
  int type2d = mStackType;
  float edgeMean;
  int blockX = 16;
  static int seed = 123456;

  if (manageRawTempArray(0))
    return 1;
  dev2dArr = sTempRawArray;

  // Select array
  if (mDoEvenOdd && (mNumFramesSummed % 2) != 0) {
    sumArr = mOddSum;
    needBound = 1;
  }

  /* Data flow for doing noise pad on GPU:
         No stack                         Stack
     load to sTempRawArray       get from raw stack array
     
                         Need preproc
     proc to sTempFloatArray     proc to sTempFloatArray  
  */
  
  if (!fullArr && (!mNumOnUnpadStack || mSavedFrameNums[0] != mNumFramesSummed)) {
    utilPrint("A NULL array was passed to addToFullSum but frame %d instead of %d is "
              "first on GPU stack\n", mNumOnUnpadStack ? mSavedFrameNums[0] : -1,
              mNumFramesSummed);
    return 1;
  }
  if (mDoNoiseTaper) {

    // If stacking, get array from stack
    if (!fullArr) {
      dev2dArr = sSavedUnpadded[0];
      edgeMean = mUnpadEdgeMeans[0];
      
    } else {

      // Otherwise get an edge mean and copy to raw temp array
      // Should these edge means be indented some?
      edgeMean = frameEdgeMean(fullArr, type2d, mUnpaddedX, FULL_INDENT, 
                               mUnpaddedX - FULL_INDENT - 1, mUnpaddedY - FULL_INDENT,
                               mUnpaddedY - FULL_INDENT - 1);
      dataSizeForMode(mStackType, &dataSize, &ind);
      START_TIMER;
      if (testErrCode(hipMemcpyToArray(dev2dArr, 0, 0, fullArr, mUnpaddedX * mUnpaddedY *
                                        dataSize, hipMemcpyHostToDevice),
                      "copy unpadded image to GPU array for noise/pad", 0))
        return 1;
      ADD_TIME(mWallCopy);
    }

    // Preprocess and change to output array and type
    if (mDoPreprocess) {
      if (runPreprocess(dev2dArr, mStackType, mNumFramesSummed))
        return 1;
      dev2dArr = sTempFloatArray;
      type2d = MRC_MODE_FLOAT;
    }

    if (testErrCode(bindUnpadArray(dev2dArr, type2d), "bind unpadded array to texture", 
                    0))
      return 1;
      
    // Do the noise pad
    START_TIMER;
    seed = (197 * (seed + 1)) & 0xFFFFF;
    dim3 blockSize(blockX, 8, 1);
    dim3 gridSize((mFullXpad + blockSize.x - 1) / blockSize.x,
                  (mFullYpad + blockSize.y - 1) / blockSize.y, 1);
    noiseTaperPad<<<gridSize, blockSize>>>
      (type2d, mUnpaddedX, mUnpaddedY, mWorkFullSize, mFullXpad + 2, mFullXpad,
       mFullYpad, mNoiseLength, 8, 20, edgeMean, seed);
    if (testReportErr("to noise pad full image"))
      return 1;
    
    if (hipDeviceSynchronize() != hipSuccess) {
      pflerr("Error return from synchronizing after noise padding full image");
      return 1;
    }
    ADD_TIME(mWallNoise);
    unbindUnpadArray(type2d);
    
  } else {
    
    // Copy to device
    START_TIMER;
    if (testErrCode(hipMemcpy(mWorkFullSize, fullArr, mFullBytes, hipMemcpyHostToDevice)
                    , "copy full padded image to GPU array", 0))
      return 1;
    ADD_TIME(mWallCopy);
  }    
  //dumpImage(mWorkFullSize, mFullXpad + 2, mFullXpad, mFullYpad, 0, "noise pad");
  
  // take FFT
  START_TIMER;
  err = hipfftExecR2C(sFullForwardPlan, mWorkFullSize, (hipfftComplex *)mWorkFullSize);
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failure in forward full FFT on GPU (CUFFT error %d)\n", err);
    return 1;
  }
  if (mTrackTime && hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after forward full FFT on GPU");
    return 1;
  }
  ADD_TIME(mWallFFT);
  //dumpFFT(mWorkFullSize, mFullXpad, mFullYpad, "full fft", 1);

  //dumpFFT(sumArr, mSumXpad, mSumYpad, "reduced sum", 1);
  if (shiftAddCommon(mWorkFullSize, sumArr, needBound, mFullXpad, mFullYpad,
                     mSumXpad, mSumYpad, shiftX, shiftY, 0, true))
    return 1;
                         
  mNumFramesSummed++;

  // Roll stack and reduce number on it
  if (!fullArr) {
    dev2dArr = sSavedUnpadded[0];
    for (ind = 0; ind < mNumOnUnpadStack - 1; ind++) {
      sSavedUnpadded[ind] = sSavedUnpadded[ind + 1];
      mSavedFrameNums[ind] = mSavedFrameNums[ind + 1];
      mUnpadEdgeMeans[ind] = mUnpadEdgeMeans[ind + 1];
    }
    sSavedUnpadded[ind] = dev2dArr;
    mNumOnUnpadStack--;
  }
      
  return 0;
}

/*
 * Common routine for shifting an FFT and adding it to a sum array
 */
int FrameGPU::shiftAddCommon(float *fullArr, float *sumArr, int needBound, int fullXpad,
                             int fullYpad, int sumXpad, int sumYpad, float shiftX,
                             float shiftY, int shiftSource, bool applyDoseFilt)
{
  int blockX = 32;
  int nxFFT = sumXpad / 2 + 1;
  float redFac = (float) fullXpad / (float)sumXpad;
  float dxy = -(redFac - 1) / (2. * redFac);
  float dxOut = shiftX / redFac + dxy;
  float dyOut = shiftY / redFac + dxy;
  bool reducing = sumXpad < fullXpad;
  double freq, arg;
  int ix, iy, ixSuper, iySuper, ixStart, iyStart, tableYoffset, iyInStart, ind;
  int numYdo, numXdo, byteScale, trigSize, numXsuperBlocks, numYsuperBlocks;

  // Setup all sines and cosines in host memory arrays
  START_TIMER;
  for (ix = 0; ix < nxFFT; ix++) {
    ind = 2 * ix;
    freq = 0.5 * ix / (nxFFT - 1.);
    arg = -2. * PI * freq * dxOut;
    mXshiftTrig[ind] = (float)cos(arg);
    mXshiftTrig[ind + 1] = (float)sin(arg);
  }

  for (iy = 0; iy < sumYpad; iy++) {
    ind = 2 * iy;
    freq = iy / (float)sumYpad;
    if (freq > 0.5)
      freq = freq - 1.;
    arg = -2. * PI * freq * dyOut;
    mYshiftTrig[ind] = (float)cos(arg);
    mYshiftTrig[ind + 1] = (float)sin(arg);
  }    

  // Take care of binding
  if (bindSumArray(needBound))
    return 1;
  
  if (bindFullOrCorrArray(fullArr, (fullXpad + 2 ) * fullYpad * sizeof(float)))
    return 1;

  // Determine blocking of the shift/reduction
  trigSize = 2 * (nxFFT + 1 + sumYpad);
  numXsuperBlocks = (trigSize + MAX_TABLE - 1) / MAX_TABLE;
  numYsuperBlocks = numXsuperBlocks;
  if (reducing)
    numYsuperBlocks = 2 * ((numYsuperBlocks + 1) / 2);
  iyStart = 0;
  byteScale = 2 * sizeof(float);
  tableYoffset = ((nxFFT + 1) / numXsuperBlocks);

  // Loop on the blocks
  for (iySuper = 0; iySuper < numYsuperBlocks; iySuper++) {
    if (reducing && iySuper == numYsuperBlocks / 2 - 1)
      numYdo = B3DMIN(sumYpad / 2 - iyStart, sumYpad / numYsuperBlocks);
    else
      numYdo = B3DMIN(sumYpad - iyStart, sumYpad / numYsuperBlocks);
    
    if (testErrCode(hipMemcpyToSymbol(HIP_SYMBOL(trigTable), &mYshiftTrig[2 * iyStart],
                                       numYdo * byteScale, byteScale * tableYoffset,
                                       hipMemcpyHostToDevice), 
                    "copy constant data to GPU", 0))
      return 1;

    ixStart = 0;
    for (ixSuper = 0; ixSuper < numXsuperBlocks; ixSuper++) {
      numXdo = B3DMIN(nxFFT - ixStart, (nxFFT + 1) / numXsuperBlocks);
      if (testErrCode(hipMemcpyToSymbol(HIP_SYMBOL(trigTable), &mXshiftTrig[2 * ixStart],
                                         numXdo * byteScale, 0, hipMemcpyHostToDevice),
                      "copy constant data to GPU", 0))
        return 1;

      // Do one superblock
      dim3 blockSize(blockX, 8, 1);
      dim3 gridSize((nxFFT + blockSize.x - 1) / blockSize.x, 
                    (sumYpad + blockSize.y - 1) / blockSize.y, 1);
      iyInStart = iyStart;
      if (reducing && iySuper >= numYsuperBlocks / 2)
        iyInStart += fullYpad - sumYpad;

      if (shiftSource)
        shiftInPlaceAddToSum<<<gridSize, blockSize>>>
          (fullArr, sumArr, ixStart, iyStart, numXdo, numYdo, nxFFT, 2 * tableYoffset);
      else
        shiftAndAddToSum<<<gridSize, blockSize>>>
          (sumArr, mDoUnDWsum ? mNonDWsum : NULL, ixStart, iyInStart, iyStart, numXdo,
           numYdo, fullXpad / 2 + 1, nxFFT, 2 * tableYoffset, 
           applyDoseFilt ? mDWFilterSize : 0, mDWFilterDelta, 1.f / fullXpad,
           1.f / fullYpad); 
      if (testReportErr("to add to sum")) {
        return 2;
      }
      if (hipDeviceSynchronize() != hipSuccess) {
        pflerr("Error return from synchronizing after shift and add block %d %d", ixSuper,
               iySuper);
        return 1;
      }
      ixStart += numXdo;
    }
    iyStart += numYdo;
  }
  //dumpFFT(sumArr, sumXpad, sumYpad, "fft of sum", 0);
  ADD_TIME(mWallShift);
  return 0;
}

/*
 * Return summed FFTs, add them and return real sum
 */
int FrameGPU::returnSums(float *sumArr, float *evenArr, float *oddArr, int evenOddOnly)
{
  int blockX = 32;
  int err, error = 0;
  int sumXplus = mSumXpad + 2;
  // FFT scaling consists of the standard forward fft scaling, divided by the reduction 
  // factor as in fourierReduceImage
  // Image scaling then includes the standard FFT scaling for the inverse FFT
  float fftScale = 1. / (sqrt((double)mFullXpad * mFullYpad) * (float)mFullYpad/mSumYpad);
  float imScale = (fftScale / sqrt((double)mSumXpad * mSumYpad));

  // Copy the even/odd arrays back if they exist
  // Not sure this is right if emergency call is made and there are no odd sums
  START_TIMER;
  if (mDoEvenOdd && evenArr && oddArr) {
    if (hipMemcpy(evenArr, mEvenSum, mSumBytes, hipMemcpyDeviceToHost) !=hipSuccess ||
        hipMemcpy(oddArr, mOddSum, mSumBytes, hipMemcpyDeviceToHost) != hipSuccess) {
      pflerr("Failure to copy even/odd sums back from GPU");
      error = 1;
    }
    normalize(evenArr, fftScale, sumXplus * mSumYpad);
    normalize(oddArr, fftScale, sumXplus * mSumYpad);
  }
  ADD_TIME(mWallCopy);
  if (evenOddOnly)
    return error;

  // If there are even and odd, add them into even
  if (mDoEvenOdd) {
    if (bindSumArray(0)) {
      error += 2;
    } else {
      sOddTex.filterMode = hipFilterModePoint;
      sOddTex.normalized = false;
      if (hipBindTexture(NULL, sOddTex, mOddSum, sChanDesc, mSumBytes) != hipSuccess) {
        error += 2;
      }
    }

    if (error < 2) {
      START_TIMER;
      dim3 blockSize(blockX, 8, 1);
      dim3 gridSize((sumXplus + blockSize.x - 1) / blockSize.x, 
                    (mSumYpad + blockSize.y - 1) / blockSize.y, 1);
      addOddToEvenSum<<<gridSize, blockSize>>>
        (mEvenSum, sumXplus, mSumYpad);
      if (testReportErr("to add odd and even sums"))
        error += 2;
      if (error < 2 && hipDeviceSynchronize() != hipSuccess) {
        pflerr("Error return from synchronizing after adding odd and even sums");
        error += 2;
      }
      ADD_TIME(mWallAddEO);
    }
    hipUnbindTexture(sOddTex);
  }

  // Inverse FFT after destroying forward plan
  destroyPlan(sFullForwardPlan);
  err = hipfftPlan2d(&sSumInversePlan, mSumYpad, mSumXpad, HIPFFT_C2R);
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failed to make plan for inverse sum FFT (size %d %d, error %d)\n",
              mSumXpad, mSumYpad, err);
    error += 2;
  }

  START_TIMER;
  if (error < 2) {
    err = hipfftExecC2R(sSumInversePlan, (hipfftComplex *)mEvenSum, mEvenSum);
    if (err != HIPFFT_SUCCESS) {
      utilPrint("Failure in sum inverse FFT on GPU (error %d)\n", err);
      error += 2;
    }
  }
  if (error < 2 && mTrackTime && hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after inverse FFT on GPU");
    return error + 2;
  }
  ADD_TIME(mWallFFT);
  destroyPlan(sSumInversePlan);

  // Get result back
  START_TIMER;
  if (error < 2 && hipMemcpy(sumArr, mEvenSum, mSumBytes, hipMemcpyDeviceToHost) != 
      hipSuccess) {
    pflerr("Failure to copy real sum from GPU");
    error += 2;
  }
  ADD_TIME(mWallCopy);

  // If there is a failure in result and even only not copied yet, copy it
  if (error == 2 && !mDoEvenOdd) {
    if (hipMemcpy(evenArr, mEvenSum, mSumBytes, hipMemcpyDeviceToHost) != hipSuccess){
      error++;
    } else if (mOddSum) {
      memset(mOddSum, 0, mSumBytes);
      normalize(evenArr, fftScale, sumXplus * mSumYpad);
    }
  }
  if (error < 2)
    normalize(sumArr, imScale, sumXplus * mSumYpad);

  return error;
}

/*
 * Inverse transform and return the non-dose weighted sum
 */
int FrameGPU::returnUnweightedSum(float *sumArr)
{
  float fftScale = 1. / (sqrt((double)mFullXpad * mFullYpad) * (float)mFullYpad/mSumYpad);
  float imScale = (fftScale / sqrt((double)mSumXpad * mSumYpad));
  int err;
  if (!mDoUnDWsum)
    return 1;
  err = hipfftPlan2d(&sSumInversePlan, mSumYpad, mSumXpad, HIPFFT_C2R);
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failed to make plan for inverse nonDW FFT (size %d %d, error %d)\n",
              mSumXpad, mSumYpad, err);
    return 1;
  }

  START_TIMER;
  err = hipfftExecC2R(sSumInversePlan, (hipfftComplex *)mNonDWsum, mNonDWsum);
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failure in nonDW sum inverse FFT on GPU (error %d)\n", err);
    return 1;
  }
  if (mTrackTime && hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after inverse FFT on GPU");
    return 1;
  }
  ADD_TIME(mWallFFT);
  destroyPlan(sSumInversePlan);

  // Get result back. The supplied array must be big enough: (x + 2) * y
  START_TIMER;
  if (testErrCode(hipMemcpy(sumArr, mNonDWsum, mSumBytes, hipMemcpyDeviceToHost), 
                  "copy nonDW sum from GPU", 0))
    return 1;
  ADD_TIME(mWallCopy);
  normalize(sumArr, imScale, (mSumXpad + 2) * mSumYpad);
  return 0;
}

/*
 * Take in the next align image, copy to appropriate place, transform and filter
 * Errors in bin/pad and stacking operations return a 2 so that only initial operations
 * can be moved to CPU, otherwise return a 1
 */
int FrameGPU::processAlignImage(float *binArr, int stackInd, int groupInd, int stackOnGpu)
{  int err, type2d, dataSize, frmStkInd, blockX = 32;
  int ind, nxBox, nyBox, ixLow, ixHigh, iyLow, iyHigh, iyStart;
  float *devArr = mWorkBinPad;
  float *groupArr;
  float edgeMean;
  bool makingGroup = mGroupSize > 1 && groupInd >= 0;
  FloatVec weights;
  IntVec iWeights;
  int support, delXstart, delXend, delYstart, delYend;
  hipArray *dev2dArr;

  if (manageRawTempArray(1))
    return 2;

  // Set array to copy into, make new if needed and push on stack
  if (stackInd >= 0) {
    if (stackInd < mSavedBinPad.size()) {
      devArr = mSavedBinPad[stackInd];
    } else if (stackInd == mSavedBinPad.size()) {
      if (testErrCode(hipMalloc((void **)&devArr, mAlignBytes), 
                      "allocate new array for align image on GPU", 0))
        return 1;
      mSavedBinPad.push_back(devArr);
    } else {
      utilPrint("Stack index in call to processAlignImage is %d, stack size is only %d\n",
                stackInd, mSavedBinPad.size());
      return 1;
    }
  }

  // If grouping, set array for that too
  if (makingGroup) {
    if (groupInd > (int)mSavedGroups.size()) {
      utilPrint("Group index in call to processAlignImage is %d, group stack size is only"
                " %d\n", groupInd, mSavedGroups.size());
      return 1;
    }
    if (groupInd < mSavedGroups.size()) {
      groupArr = mSavedGroups[groupInd];
    } else {
      if (testErrCode(hipMalloc((void **)&groupArr, mAlignBytes),
                      "allocate new array for group align image on GPU", 0))
        return 1;
      mSavedGroups.push_back(groupArr);
    }
  }

  /* Data flow for doing bin/pad on GPU:
         No stack                         Stack
     load to sTempRawArray       load to raw stack array

                       Need preproc
     proc to sTempFloatArray     proc to sTempFloatArray  
   */

  if (mDoAlignBinPad) {

    // Set where to load this image to
    dev2dArr = sTempRawArray;
    type2d = mStackType;
    if (stackOnGpu) {

      // When stacking, we need to get the edge mean now while we have the data
      edgeMean = frameEdgeMean(binArr, type2d, mUnpaddedX, FULL_INDENT, 
                               mUnpaddedX - FULL_INDENT - 1, mUnpaddedY - FULL_INDENT,
                               mUnpaddedY - FULL_INDENT - 1);

      // Put in existing spot at end of stack or make a new spot on stack
      frmStkInd = mNumOnUnpadStack;
      if (frmStkInd < sSavedUnpadded.size()) {
        dev2dArr = sSavedUnpadded[frmStkInd];
        mUnpadEdgeMeans[frmStkInd] = edgeMean;
        mSavedFrameNums[frmStkInd] = mNumAlignedFrames;
        mNumOnUnpadStack++;
      } else {
        if (testErrCode(malloc2Darray(&dev2dArr, mStackType, mUnpaddedX, mUnpaddedY),
                        "allocate 2D array for full stack on GPU", 0))
          return 2;
        sSavedUnpadded.push_back(dev2dArr);
        mUnpadEdgeMeans.push_back(edgeMean);
        mSavedFrameNums.push_back(mNumAlignedFrames);
        mNumOnUnpadStack++;
      }
    }

    // copy to array
    dataSizeForMode(mStackType, &dataSize, &err);
    START_TIMER;
    if (testErrCode(hipMemcpyToArray(dev2dArr, 0, 0, binArr, mUnpaddedX * mUnpaddedY *
                                      dataSize, hipMemcpyHostToDevice),
                    "copy unpadded image to GPU array for bin/pad", 0))
      return 2;
    ADD_TIME(mWallCopy);
    //dumpUnpadArray(dev2dArr, mUnpaddedX, mUnpaddedY, type2d, "input array");
    
    if (mDoPreprocess) {
      if (runPreprocess(dev2dArr, type2d, mNumAlignedFrames))
        return 2;
      
      dev2dArr = sTempFloatArray;
      type2d = MRC_MODE_FLOAT;
    }
    //dumpUnpadArray(dev2dArr, mUnpaddedX, mUnpaddedY, type2d, "preproc array");

    if (testErrCode(bindUnpadArray(dev2dArr, type2d), 
                    "bind input array to texture for reduction", 0))
      return 2;

    nxBox = (mXend + 1 - mXstart) / mAliBinning;
    nyBox = (mYend + 1 - mYstart) / mAliBinning;
    ixLow = mAlignXpad / 2 - nxBox / 2;
    ixHigh = ixLow + nxBox;
    iyLow = mAlignYpad / 2 - nyBox / 2;
    iyHigh = iyLow + nyBox;
    
    edgeMean = frameEdgeMean(binArr, mStackType, mUnpaddedX, mXstart, mXend, mYstart,
                             mYend);

    if (mAliBinning > 1) {

      // Set up the reduction filter for X and copy to constant memory
      selectZoomFilter(mAntiFiltType, 1. / mAliBinning, &support);
      makeReductionWeights(mXstart, support, weights, delXstart, delXend);
      if (type2d == MRC_MODE_FLOAT) {
        if (testErrCode(hipMemcpyToSymbol(HIP_SYMBOL(trigTable), &weights[0], weights.size() * 
                                           sizeof(float), 0, hipMemcpyHostToDevice),
                        "copy constant X weight data to GPU", 0))
          return 2;
      } else {
        for (ind = 0; ind < (int)weights.size(); ind++)
          iWeights.push_back(B3DNINT(INT_KERNEL_SCALE * weights[ind]));
        if (testErrCode(hipMemcpyToSymbol(HIP_SYMBOL(intKernel), &iWeights[0], iWeights.size() * 
                                           sizeof(int), 0, hipMemcpyHostToDevice),
                        "copy integer constant weight data to GPU", 0))
          return 2;
      }
      iyStart = mYstart - (support / 2 + 1);

#ifndef NO_SURFACES
      if (testErrCode(cudaBindSurfaceToArray(sTempSurfRef, sReducedInXarray),
                      "bind X-reduced array to surface for reduction", 0))
        return 2;
#endif

      START_TIMER;
      dim3 blockSize(blockX, 8, 1);
      dim3 gridSize((mRedColX + blockSize.x - 1) / blockSize.x, 
                    (mRedColY + blockSize.y - 1) / blockSize.y, 1);
      reduceColumns<<<gridSize, blockSize>>>
        (mAliBinning, mReducedInXlinear, type2d, mRedColX, mRedColY, iyStart, delXstart,
         delXend);
      if (testReportErr("to do reduction in X"))
        return 2;

      // Do not synchronize here  - saves 3% of time
      /*if (hipDeviceSynchronize() != hipSuccess) {
        pflerr("Error return from synchronizing after reducing/padding image");
        return 2;
        } */ 
      unbindUnpadArray(type2d);
      //dumpUnpadArray(sReducedInXarray, mRedColX, mRedColY, MRC_MODE_FLOAT, 
      //"x-reduced array");

      if (mNoSurfaces && testErrCode
          (hipMemcpyToArray(sReducedInXarray, 0, 0, mReducedInXlinear, mRedColX * 
                             mRedColY * sizeof(float), hipMemcpyDeviceToDevice),
           "copy X-reduced output array to 2D array", 0))
        return 1;
    
      if (testErrCode(bindUnpadArray(sReducedInXarray, MRC_MODE_FLOAT), 
                      "bind X-reduced array to texture for Y reduction", 0))
        return 2;

      weights.clear();
      makeReductionWeights(mYstart, support, weights, delYstart, delYend);
      if (testErrCode(hipMemcpyToSymbol(HIP_SYMBOL(trigTable), &weights[0], weights.size() * 
                                         sizeof(float), 0, hipMemcpyHostToDevice),
                      "copy constant Y weight data to GPU", 0))
        return 2;

      //dim3 blockSize(blockX, 8, 1);
      dim3 gridSize2((mUnpaddedX + blockSize.x - 1) / blockSize.x, 
                     (mUnpaddedY + blockSize.y - 1) / blockSize.y, 1);
      reduceRowsTaperPad<<<gridSize2, blockSize>>>
        (devArr, mAliBinning, iyStart, ixLow, ixHigh, iyLow, 
         iyHigh, mAlignXpad + 2, mAlignXpad, mAlignYpad, mNxTaper, mNyTaper, edgeMean, 
         delYstart, delYend);
      if (testReportErr("to reduce, taper and pad image"))
        return 2;
    } else {

      // binning 1 is a simple copy of trimmed area with padding and taper inside
      START_TIMER;
      dim3 blockSize(blockX, 8, 1);
      dim3 gridSize((mUnpaddedX + blockSize.x - 1) / blockSize.x, 
                    (mUnpaddedY + blockSize.y - 1) / blockSize.y, 1);
      trimTaperPad<<<gridSize, blockSize>>>
        (devArr, type2d, ixLow, ixHigh, iyLow, iyHigh, mAlignXpad + 2, mAlignXpad,
         mAlignYpad, mNxTaper, mNyTaper, edgeMean);
      if (testReportErr("to trim, taper and pad image"))
        return 2;
    }
    
    if (hipDeviceSynchronize() != hipSuccess) {
      pflerr("Error return from synchronizing after reducing/padding image");
      return 2;
    }
    ADD_TIME(mWallRedPad);
    unbindUnpadArray(type2d);

  } else {

    // NO BIN/PAD HERE, copy taper-padded image to device
    START_TIMER;
    if (testErrCode(hipMemcpy(devArr, binArr, mAlignBytes, hipMemcpyHostToDevice),
                    "copy align image to GPU array", 0))
      return 1;
    ADD_TIME(mWallCopy);
  }
  //dumpImage(devArr, mAlignXpad + 2, mAlignXpad, mAlignYpad, 0, "reduction", mNumAlignedFrames);
  
  // If grouping and not refining, just sum real-space stack into group array
  if (makingGroup && !mDoAlignSum) {
    if (sumIntoGroup(stackInd, groupInd))
      return 1;
    devArr = groupArr;
  }
  
  if (mGroupSize == 1 || mDoAlignSum || groupInd >= 0) {
    
    // Take the FFT
    START_TIMER;
    err = hipfftExecR2C(sAlignForwardPlan, devArr, (hipfftComplex *)devArr);
    if (err != HIPFFT_SUCCESS) {
      utilPrint("Failure in forward FFT of align image on GPU (error %d)\n", err);
      return 1;
    }
    if (mTrackTime && hipDeviceSynchronize() != hipSuccess) {
      pflerr("Error return from synchronizing after forward FFT of align image on GPU");
      return 1;
    }
    ADD_TIME(mWallFFT);
    
    // Filter the array
    START_TIMER;
    dim3 blockSize(blockX, 8, 1);
    dim3 gridSize((mAlignXpad + 2 + blockSize.x - 1) / blockSize.x, 
                  (mAlignYpad + blockSize.y - 1) / blockSize.y, 1);
    filterAlignFFT<<<gridSize, blockSize>>>
      (devArr, mAlignXpad + 2, mAlignYpad);
    if (testReportErr("to filter align image FFT"))
      return 1;
    
    if (hipDeviceSynchronize() != hipSuccess) {
      pflerr("Error return from synchronizing after filtering align image FFT");
      return 1;
    }
    ADD_TIME(mWallFilt);
  }

  // Or if grouping and refining, now sum the filtered FFTs
  if (makingGroup && mDoAlignSum && sumIntoGroup(stackInd, groupInd))
    return 1;
  
  mNumAlignedFrames++;
  return 0;
}

/* 
 * Do the preprocessing: gain normalization, truncation, defect removal
 */
 int FrameGPU::runPreprocess(void *dev2dArr, int type2d, int frame)
{
  int blockX = 16;

  //dumpUnpadArray(dev2dArr, mUnpaddedX, mUnpaddedY, type2d, "raw image");
  
  // Bind input array to texture and output array to surface
  if (testErrCode(bindUnpadArray((hipArray *)dev2dArr, type2d), 
                  "bind array to texture for preprocessing", 0))
    return 1;
#ifndef NO_SURFACES
  if (testErrCode(cudaBindSurfaceToArray(sTempSurfRef, sTempFloatArray),
                  "bind output array to surface for preprocessing", 0))
    return 1;
#endif
  
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((mUnpaddedX + 2 + blockSize.x - 1) / blockSize.x, 
                (mUnpaddedY + blockSize.y - 1) / blockSize.y, 1);
  preprocessFrame<<<gridSize, blockSize>>>
    (mProcessedLinear, type2d, mUnpaddedX, mUnpaddedY, mDoGainNorm, mTruncLimit,
        mCorrectDefects);
  if (testReportErr("to preprocess image"))
    return 1;
      
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after preprocessing image");
    return 1;
  }
  if (mNoSurfaces && testErrCode
      (hipMemcpyToArray(sTempFloatArray, 0, 0, mProcessedLinear, mUnpaddedX *
                         mUnpaddedY * sizeof(float), hipMemcpyDeviceToDevice),
       "copy pre-processed output array to 2D array", 0))
    return 1;
  ADD_TIME(mWallPreproc);

  // Unbind input texture: there is no surface unbinding as the next bind unbinds it
  unbindUnpadArray(type2d);
  //dumpUnpadArray(sTempFloatArray, mUnpaddedX, mUnpaddedY, MRC_MODE_FLOAT,
  //             "gain normed", frame);
  return 0;
}

/*
 * Get weights for image reduction in one dimension and offsets that apply to given 
 * starting coordinate in the image
 */
void FrameGPU::makeReductionWeights(int startCoord, int support, FloatVec &weights,
                                    int &delStart, int &delEnd)
{
  // Replicate variables and equations in zoomFiltInterp routine when passed startCoord
  float cen = startCoord + 0.5 * mAliBinning;
  int ind, i0 = floor(cen + 0.5 - support);
  int i1 = ceil(cen + 0.5 + support);
  float filtVal;
  delStart = -999;

  // Loop on support and keep track of first and last non-zero value
  // The function returns normalized values
  for (ind = i0; ind <= i1; ind++) {
    filtVal = zoomFiltValue(ind + 0.5 - cen);
    if (filtVal) {
      weights.push_back(filtVal);
      delEnd = ind;
      if (delStart == -999)
        delStart = ind;
    }
  }
}

/* 
 * Sum from 2 to 5 images or FFTs into a group array
 */
int FrameGPU::sumIntoGroup(int stackInd, int groupInd)
{
  int blockX = 32;
  float *groupArr;
  if (stackInd + 1 - mGroupSize < 0 || stackInd >= (int)mSavedBinPad.size() || 
      groupInd < 0 || groupInd >= (int)mSavedGroups.size()) {
    utilPrint("Index in call to sumIntoGroup is out of range (%d and %d - sizes %d and "
              "%d\n", stackInd, groupInd, mSavedBinPad.size(), mSavedGroups.size());
    return 1;
  }
  groupArr = mSavedGroups[groupInd];

  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((mAlignXpad + 2 + blockSize.x - 1) / blockSize.x, 
                (mAlignYpad + blockSize.y - 1) / blockSize.y, 1);
  if (mGroupSize == 2)
    sum2IntoGroup<<<gridSize, blockSize>>>
      (mSavedBinPad[stackInd - 1], mSavedBinPad[stackInd], 
       groupArr, mAlignXpad + 2, mAlignYpad);
  else if (mGroupSize == 3)
    sum3IntoGroup<<<gridSize, blockSize>>>
      (mSavedBinPad[stackInd - 2], mSavedBinPad[stackInd - 1], mSavedBinPad[stackInd], 
       groupArr, mAlignXpad + 2, mAlignYpad);
  else if (mGroupSize == 4)
    sum4IntoGroup<<<gridSize, blockSize>>>
      (mSavedBinPad[stackInd - 3], mSavedBinPad[stackInd - 2], mSavedBinPad[stackInd - 1],
       mSavedBinPad[stackInd], groupArr, mAlignXpad + 2, mAlignYpad);
  else
    sum5IntoGroup<<<gridSize, blockSize>>>
      (mSavedBinPad[stackInd - 4], mSavedBinPad[stackInd - 3], mSavedBinPad[stackInd - 2],
       mSavedBinPad[stackInd - 1], mSavedBinPad[stackInd], groupArr, mAlignXpad + 2,
       mAlignYpad);
  if (testReportErr("to sum group on GPU"))
    return 1;
  
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after summing group on GPU");
    return 1;
  }
  ADD_TIME(mWallGroup);
  return 0;
}

/*
 * Correlate two images that have been stored somewhere
 * aliInd is a stack index or -1 to align mWorkBinPad; refInd is a stack index or -1 to
 * align to mAlignSum or -2 to align to mWorkBinPad
 */
int FrameGPU::crossCorrelate(int aliInd, int refInd, float *subarea, int subXoffset,
                             int subYoffset)
{
  float *aliArr = mWorkBinPad;
  float *refArr = mAlignSum;
  float *corrArr = mCorrBinPad;
  int ixFrom0[4], ixTo0[4], iyFrom0[4], iyTo0[4], ixFrom1[4], ixTo1[4], iyFrom1[4];
  int iyTo1[4], roundXoffset, roundYoffset, ix0, iy0, bigSubBytes, err;
  int blockX = 32;
  std::vector<float *> *savedArr = mGroupSize > 1 ? &mSavedGroups : &mSavedBinPad;

  // Check the indexes and set the source and destination arrays 
  if (aliInd >= (int)savedArr->size() || refInd >= (int)savedArr->size()) {
    utilPrint("Index for align image (%d) or reference (%d) is too big (stack size %d)",
              aliInd, refInd, savedArr->size());
    return 1;
  }
  if (aliInd >= 0)
    aliArr = (*savedArr)[aliInd];
  if (refInd >= 0)
    refArr = (*savedArr)[refInd];
  if (refInd < -1)
    refArr = mWorkBinPad;
  if (aliInd >= 0 && refInd >= 0)
    corrArr = mWorkBinPad;

  // Take the conjugate product
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((mAlignXpad + 2 + blockSize.x - 1) / blockSize.x, 
                (mAlignYpad + blockSize.y - 1) / blockSize.y, 1);
  conjugateProduct<<<gridSize, blockSize>>>
    (aliArr, refArr, corrArr, mAlignXpad / 2 + 1, mAlignYpad);
  if (testReportErr("to take conjugate product"))
    return 1;

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after taking conjugate product");
    return 1;
  }
  ADD_TIME(mWallConj);

  START_TIMER;
  err = hipfftExecC2R(sAlignInversePlan, (hipfftComplex *)corrArr, mRealCorr);
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failure in inverse FFT of align image on GPU (error %d)\n", err);
    return 1;
  }
  if (mTrackTime && hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after inverse FFT of align image on GPU");
    return 1;
  }
  ADD_TIME(mWallFFT);

  // extract the subarea
  // This is an intrinsically slow operation, but copying lines direct from device
  // to host was almost twice as slow
  roundXoffset = NICE_GPU_DIVISOR * B3DNINT((float)subXoffset / NICE_GPU_DIVISOR);
  roundYoffset = NICE_GPU_DIVISOR * B3DNINT((float)subYoffset / NICE_GPU_DIVISOR);
  utilCoordsForWrap(mAlignXpad, mAlignYpad, mBigSubareaSize, mBigSubareaSize,
                    roundXoffset, roundYoffset, ixFrom0, ixTo0, iyFrom0, iyTo0,
                    ixFrom1, ixTo1, iyFrom1, iyTo1);
  
  if (bindFullOrCorrArray(mRealCorr, mAlignBytes))
    return 1;
  
  dim3 gridSize2((mBigSubareaSize + blockSize.x - 1) / blockSize.x, 
                 (mBigSubareaSize + blockSize.y - 1) / blockSize.y, 1);
  wrapCorners<<<gridSize2, blockSize>>>
    (corrArr, mSubareaCorr, mAlignXpad, mAlignYpad, mBigSubareaSize, mBigSubareaSize, 
     ixFrom0[2], iyFrom0[2]);
  if (testReportErr("to wrap subarea corr in GPU"))
    return 1;
  
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after wrapping subarea corr in GPU");
    return 1;
  }
  ADD_TIME(mWallExtract);
  //dumpImage(mSubareaCorr, mBigSubareaSize, mBigSubareaSize, mBigSubareaSize, 0,
  //"big subarea dev");
  
  // Unload the subarea
  mWallStart= wallTime();
  bigSubBytes = mBigSubareaSize * mBigSubareaSize * sizeof(float);
  if (hipMemcpy(mHostSubarea, mSubareaCorr, bigSubBytes, hipMemcpyDeviceToHost) != 
      hipSuccess) {
    pflerr("Copying subarea correlation to host");
    return 1;
  }
  
  // Extract proper area into subarea array
  ix0 = (mBigSubareaSize - mAliFiltSize) / 2 + (subXoffset - roundXoffset);
  iy0 = (mBigSubareaSize - mAliFiltSize) / 2 + (subYoffset - roundYoffset);
  sliceTaperInPad(mHostSubarea, SLICE_MODE_FLOAT, mBigSubareaSize, ix0, 
                  ix0 + mAliFiltSize - 1, iy0, iy0 + mAliFiltSize - 1, subarea,
                  mAliFiltSize + 2, mAliFiltSize, mAliFiltSize, 0, 0);
  ADD_TIME(mWallCopy);
  return 0;
}

/*
 * Shift one alignment image and add it into align sum, possibly in place
 */
int FrameGPU::shiftAddToAlignSum(int stackInd, float shiftX, float shiftY,
                                 int shiftSource)
{
  float *binArr = mWorkBinPad;
  if (stackInd >= (int)mSavedBinPad.size()) {
    utilPrint("Stack index in call to shiftAddToalignSum is %d, stack size is "
              "only %d\n", stackInd, mSavedBinPad.size());
    return 1;
  }
  if (stackInd >= 0)
    binArr = mSavedBinPad[stackInd];
  //dumpFFT(binArr, mAlignXpad, mAlignYpad, "fft of shift/add", 1);
  int err = shiftAddCommon(binArr, mAlignSum, 2, mAlignXpad, mAlignYpad, mAlignXpad, 
                           mAlignYpad, shiftX, shiftY, shiftSource, false);
  //dumpFFT(binArr, mAlignXpad, mAlignYpad, "fft after shifing", 1);
  //dumpFFT(mAlignSum, mAlignXpad, mAlignYpad, "Align sum", 1);
  return err;
}

/*
 * Upload a new filter mask to the the device for high-frequency filtering in refinement
 */
int FrameGPU::newFilterMask(float *alignMask)
{
  START_TIMER;
  if (testErrCode(hipMemcpy(mFiltMask, alignMask, mAlignBytes, hipMemcpyHostToDevice),
                  "copy filter mask to GPU array", 0))
    return 1;
  ADD_TIME(mWallCopy);
  return 0;
}

/*
 * Subtract one image on stack from the align sum and apply the filter mask to the
 * leave-one-out sum
 */
int FrameGPU::subtractAndFilterAlignSum(int stackInd, int groupRefine)
{
  int blockX = 32;
  std::vector<float *> *savedArr = groupRefine ? &mSavedGroups : &mSavedBinPad;

  if (stackInd >= (int)savedArr->size() || stackInd < 0) {
    utilPrint("Stack index in call to addOrSubtractAlignSum is %d, stack size is "
              "only %d\n", stackInd, savedArr->size());
    return 1;
  }

  START_TIMER;
  bindSumArray(2);
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((mAlignXpad + 2 + blockSize.x - 1) / blockSize.x, 
                (mAlignYpad + blockSize.y - 1) / blockSize.y, 1);
  subtractFilterSum<<<gridSize, blockSize>>>
    ((*savedArr)[stackInd], mWorkBinPad, mAlignXpad + 2, mAlignYpad);
  if (testReportErr("to subtract from aligned sum"))
    return 1;

  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after subtracting from align sum");
    return 1;
  }
  ADD_TIME(mWallSubtract);
  return 0;
}

/*
 * Roll the stack of alignment images
 */
void FrameGPU::rollAlignStack()
{
  if (mSavedBinPad.size() == 0)
    return;
  utilRollSavedFrames(mSavedBinPad, mSavedBinPad.size());
}

/*
 * Roll the stack of group images
 */
void FrameGPU::rollGroupStack()
{
  if (mGroupSize > 1 && mSavedGroups.size() > 0)
    utilRollSavedFrames(mSavedGroups, mSavedGroups.size());
}

/*
 * Return the number of arrays saved in savedBinPad and savedGroups so that allocations
 * can be made for fetching them back after an error
 */
void FrameGPU::numberOfAlignFFTs(int *numBinPad, int *numGroups)
{
  *numBinPad = (int)mSavedBinPad.size();
  *numGroups = (int)mSavedGroups.size();
}

/*
 * Return all the data from GPU being used for alignment to try to recover from error
 */
int FrameGPU::returnAlignFFTs(float **saved, float **groups, float *alignSum,
                              float *workArr)
{
  int ind, loop, alignPix = (mAlignXpad + 2) * mAlignYpad;
  float *array;
  float fftScale = 1. / sqrt((double)mAlignXpad * mAlignYpad);
  std::vector<float *> *devSaved = &mSavedBinPad;
  float **hostSaved = saved;
  for (loop = 0; loop < 2; loop ++) {
    for (ind = 0; ind < devSaved->size(); ind++) {
      if (!hostSaved)
        return 1;
      array = hostSaved[ind];
      if (hipMemcpy(array, devSaved->at(ind), mAlignBytes, hipMemcpyDeviceToHost) !=
          hipSuccess) {
        pflerr("Error copying align FFT back from GPU");
        return 1;
      }

      // Normalize unless it is a real-space image kept to make group sum in mSavedBinPad
      if (!(loop == 0 && mGroupSize > 1 && !mDoAlignSum))
        normalize(array, fftScale, alignPix);
    }
    devSaved = &mSavedGroups;
    hostSaved = groups;
  }
  if (alignSum) {
    if (hipMemcpy(alignSum, mAlignSum, mAlignBytes, hipMemcpyDeviceToHost) != 
        hipSuccess) {
      pflerr("Error copying align sum FFT back from GPU");
      return 1;
    }
    normalize(array, fftScale, alignPix);
  }
  if (workArr) {
    if (hipMemcpy(workArr, mWorkBinPad, mAlignBytes, hipMemcpyDeviceToHost) != 
        hipSuccess) {
      pflerr("Error copying align work FFT back from GPU");
      return 1;
    }
    normalize(workArr, fftScale, alignPix);
  }
  return 0;
}

/*
 * Return a stacked full frame from the first position on that stack, and its frame number
 */
int FrameGPU::returnStackedFrame(float *array, int *frameNum)
{
  hipArray *dev2dArr;
  int err, dataSize;
  dataSizeForMode(mStackType, &dataSize, &err);
  if (!mNumOnUnpadStack) {
    utilPrint("Program error: returnStackedFrame called with no frames left on stack\n");
    return 1;
  }
  dev2dArr = sSavedUnpadded[0];
  *frameNum = mSavedFrameNums[0];
  if (hipMemcpyFromArray(array, dev2dArr, 0, 0, dataSize * mUnpaddedX * mUnpaddedY,
                      hipMemcpyDeviceToHost) != hipSuccess) {
    pflerr("Error copying stacked full frame back from GPU");
    return 1;
  }
  free2Darray(&dev2dArr);

  // Remove from stack
  sSavedUnpadded.erase(sSavedUnpadded.begin());
  mSavedFrameNums.erase(mSavedFrameNums.begin());
  mUnpadEdgeMeans.erase(mUnpadEdgeMeans.begin());
  mNumOnUnpadStack--;
  return 0;
}

/*
 * Bind whichever sum array is needed
 */
int FrameGPU::bindSumArray(int needBound) 
{
  float *array = mEvenSum;
  int sizeArr = mSumBytes;
  if (needBound == mBoundSum)
    return 0;
  if (needBound == 1) {
    array = mOddSum;
  } else if (needBound == 2) {
    array = mAlignSum;
    sizeArr = mAlignBytes;
  }

  if (mBoundSum >= 0)
    hipUnbindTexture(sSumTex);
  if (testErrCode(hipBindTexture(NULL, sSumTex, array, sChanDesc, sizeArr),
                  "bind array for sum to texture", 0))
    return 1;
  mBoundSum = needBound;
  return 0;
}

/*
 * And bind whichever full or correlation array is needed to this texture
 */
int FrameGPU::bindFullOrCorrArray(float *fullArr, size_t sizeTmp)
{
  if (mBoundToFull != fullArr) {
    sFullTex.filterMode = hipFilterModePoint;
    sFullTex.normalized = false;
    if (mBoundToFull)
      hipUnbindTexture(sFullTex);
    mBoundToFull = NULL;
    if (testErrCode(hipBindTexture(NULL, sFullTex, fullArr, sChanDesc, sizeTmp), 
                    "bind source array for summing to texture", 0))
      return 1;
    mBoundToFull = fullArr;
  }
  return 0;
}

/*
 * Normalize an array after FFT
 */
void FrameGPU::normalize(float *data, float scale, int numPix)
{
  for (int ind = 0; ind < numPix; ind++)
    data[ind] *= scale;
}

/*
 * Make sure the arrays for storing shift sines and cosines for copy to constant
 * memory are big enough
 */
int FrameGPU::manageShiftTrigs(int xpad, int ypad)
{
  if (xpad + 2 > mXtrigSize) {
    B3DFREE(mXshiftTrig);
    mXshiftTrig = B3DMALLOC(float, xpad + 2);
    mXtrigSize = xpad + 2;
  }

  if (ypad * 2 > mYtrigSize) {
    B3DFREE(mYshiftTrig);
    mYshiftTrig = B3DMALLOC(float, ypad * 2);
    mYtrigSize = ypad * 2;
  }
  if (!mXshiftTrig || !mYshiftTrig) {
    utilPrint("Failed to allocate arrays for sines/cosines\n");
    return 1;
  }
  return 0;
}

void FrameGPU::printTimers()
{
  utilPrint("GPU: copy %.4f  pre %.4f  FFT %.4f  shift %.4f  e/o add %.4f  filt %.4f\n "
            "   red %.4f  noise %.4f  conj %.4f  extr %.4f  subtr %.4f "
            " group %.4f\n", mWallCopy, mWallPreproc,
            mWallFFT, mWallShift, mWallAddEO, mWallFilt,  mWallRedPad,
            mWallNoise, mWallConj, mWallExtract, mWallSubtract, mWallGroup);
}

// Test for and report error after executing threads           
int FrameGPU::testReportErr(const char *mess)
{
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    utilPrint("Error executing threads %s: %s\n", mess,
              hipGetErrorString(err));
    return 1;
  }
  return 0;
}

// Test error code from cuda calls and output messages
int FrameGPU::testErrCode(int errCode, const char *mess, int cleanAll)
{
  if (errCode != hipSuccess) {
    pflerr("Failed to %s", mess);
    if (cleanAll)
      cleanup();
    return 1;
  }
  return 0;
}

// In case of error, find the error string and print it with message
void FrameGPU::pflerr(const char *format, ...)
{
  hipError_t err;
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  err = hipGetLastError();
  utilPrint("%s: %s\n", errorMess, hipGetErrorString(err));
  va_end(args);
}

/*
 * Routines to output images afer unloading from GPU
 */
void FrameGPU::dumpFFT(float *fft, int nxPad, int nyPad, const char *descrip, int doReal)
{
  int sizeTmp = (nxPad + 2) * nyPad;
  float *temp = B3DMALLOC(float, sizeTmp);
  if (!temp)
    return;
  if (hipMemcpy(temp, fft, sizeTmp * sizeof(float), hipMemcpyDeviceToHost) ==
      hipSuccess)
    utilDumpFFT(temp, nxPad, nyPad, descrip, doReal);
  free(temp);
}

void FrameGPU::dumpImage(float *image, int nxDim, int nxPad, int nyPad, int isCorr,
                         const char *descrip, int frame)
{
  int sizeTmp = nxDim * nyPad;
  float *temp = B3DMALLOC(float, sizeTmp);
  if (!temp)
    return;
  if (hipMemcpy(temp, image, sizeTmp * sizeof(float), hipMemcpyDeviceToHost) == 
      hipSuccess)
    utilDumpImage(temp, nxDim, nxPad, nyPad, isCorr, descrip);
  free(temp);
}

void FrameGPU::dumpUnpadArray(void *array, int sizeX, int sizeY, int type,
                              const char *descrip, int frame)
{
  int err, dataSize, sizeTmp;
  hipArray *dev2dArr = (hipArray *)array;
  dataSizeForMode(type, &dataSize, &err);
  sizeTmp = sizeX * sizeY * dataSize;
  unsigned char *temp = B3DMALLOC(unsigned char, sizeTmp);
  if (!temp)
    return;
  float *ftemp = B3DMALLOC(float, sizeTmp);
  if (!ftemp) {
    free(temp);
    return;
  }
  if (hipMemcpyFromArray(temp, dev2dArr, 0, 0, sizeTmp, hipMemcpyDeviceToHost) == 
      hipSuccess) {
    sliceTaperOutPad(temp, type, sizeX, sizeY, ftemp, sizeX,sizeX, sizeY , 0, 0.);
    utilDumpImage(ftemp, sizeX, sizeX, sizeY, 0, descrip, frame);
  }

  free(temp);
  free(ftemp);
}

/*
 * Possible addition to libcfshr/filtxcorr, takes an edge mean for any data type
 */
float FrameGPU::frameEdgeMean(void *array, int type, int nxdim, int ixlo, int ixhi,
                              int iylo, int iyhi)
{
  double sum = 0.;
  float dmean;
  float *fdata = (float *)array;
  unsigned char *bdata = (unsigned char *)array;
  short *sdata = (short *)array;
  unsigned short *usdata = (unsigned short *)array;
  int ix, iy;

  switch (type) {
  case MRC_MODE_BYTE:
    for (ix = ixlo; ix <= ixhi; ix++) {
      sum += bdata[ix + iylo * nxdim];
      sum += bdata[ix + iyhi * nxdim];
    }
    for (iy = iylo + 1; iy < iyhi; iy++) {
      sum += bdata[ixlo + iy * nxdim];
      sum += bdata[ixhi + iy * nxdim];
    }
    break;
  case MRC_MODE_SHORT:
    for (ix = ixlo; ix <= ixhi; ix++)
      sum += sdata[ix + iylo * nxdim] + sdata[ix + iyhi * nxdim];
    for (iy = iylo + 1; iy < iyhi; iy++)
      sum += sdata[ixlo + iy * nxdim] + sdata[ixhi + iy * nxdim];
    break;
  case MRC_MODE_USHORT:
    for (ix = ixlo; ix <= ixhi; ix++)
      sum += usdata[ix + iylo * nxdim] + usdata[ix + iyhi * nxdim];
    for (iy = iylo + 1; iy < iyhi; iy++)
      sum += usdata[ixlo + iy * nxdim] + usdata[ixhi + iy * nxdim];
    break;
  case MRC_MODE_FLOAT:
    for (ix = ixlo; ix <= ixhi; ix++)
      sum += fdata[ix + iylo * nxdim] + fdata[ix + iyhi * nxdim];
    for (iy = iylo + 1; iy < iyhi; iy++)
      sum += fdata[ixlo + iy * nxdim] + fdata[ixhi + iy * nxdim];
    break;
  }
  dmean = sum / (2 * (ixhi - ixlo + iyhi - iylo));
  return dmean;
}


////////////////////////////////////////////////////////////////////////////
// WRAPPER FUNCTIONS
///////////////////////////////////////////////////////////////////////////

DLL_EX_IM int fgpuGpuAvailable(int nGPU, float *memory, int debug)
{
  return sFGPU.gpuAvailable(nGPU, memory, debug);
}

DLL_EX_IM void fgpuSetUnpaddedSize(int unpadX, int unpadY, int flags, int debug)
{
  sFGPU.setUnpaddedSize(unpadX, unpadY, flags, debug);
}

DLL_EX_IM int fgpuSetPreProcParams(float *gainRef, int nxGain, int nyGain,
                                   float truncLimit, unsigned char *defectMap,
                                   int camSizeX, int camSizeY)
{
  return sFGPU.setPreProcParams(gainRef, nxGain, nyGain, truncLimit, defectMap,
                               camSizeX, camSizeY);
}

DLL_EX_IM void fgpuSetBinPadParams(int xstart, int xend, int ystart, int yend,
                                   int binning, int nxTaper, int nyTaper, int type,
                                   int filtType, int noiseLen)
{
  sFGPU.setBinPadParams(xstart, xend, ystart, yend, binning, nxTaper, nyTaper,
                        type, filtType, noiseLen);
}

DLL_EX_IM int fgpuSetupSumming(int fullXpad, int fullYpad, int sumXpad, int sumYpad,
                               int evenOdd)
{
  return sFGPU.setupSumming(fullXpad, fullYpad, sumXpad, sumYpad, evenOdd);
}

DLL_EX_IM int fgpuSetupAligning(int alignXpad, int alignYpad, int sumXpad, int sumYpad,
                                float *alignMask, int aliFiltSize, int groupSize,
                                int expectStackSize, int doAlignSum)
{
  return sFGPU.setupAligning(alignXpad, alignYpad, sumXpad, sumYpad, alignMask,
                             aliFiltSize, groupSize, expectStackSize, doAlignSum);
}

DLL_EX_IM int fgpuSetupDoseWeighting(float *filter, int filtSize, float delta)
{
  return sFGPU.setupDoseWeighting(filter, filtSize, delta);
}

DLL_EX_IM int fgpuAddToFullSum(float *fullArr, float shiftX, float shiftY)
{
  return sFGPU.addToFullSum(fullArr, shiftX, shiftY);
}

DLL_EX_IM int fgpuReturnSums(float *sumArr, float *evenArr, float *oddArr,
                             int evenOddOnly)
{
  return sFGPU.returnSums(sumArr, evenArr, oddArr, evenOddOnly);
}

DLL_EX_IM int fgpuReturnUnweightedSum(float *sumArr)
{
  return sFGPU.returnUnweightedSum(sumArr);
}

DLL_EX_IM void fgpuCleanup()
{
  sFGPU.cleanup();
}

DLL_EX_IM void fgpuRollAlignStack()
{
  sFGPU.rollAlignStack();
}

DLL_EX_IM void fgpuRollGroupStack()
{
  sFGPU.rollGroupStack();
}

DLL_EX_IM int fgpuSubtractAndFilterAlignSum(int stackInd, int groupRefine)
{
  return sFGPU.subtractAndFilterAlignSum(stackInd, groupRefine);
}

DLL_EX_IM int fgpuNewFilterMask(float *alignMask)
{
  return sFGPU.newFilterMask(alignMask);
}

DLL_EX_IM int fgpuShiftAddToAlignSum(int stackInd, float shiftX, float shiftY,
                                     int shiftSource)
{
  return sFGPU.shiftAddToAlignSum(stackInd, shiftX, shiftY, shiftSource);
}

DLL_EX_IM int fgpuCrossCorrelate(int aliInd, int refInd, float *subarea, int subXoffset,
                                 int subYoffset)
{
  return sFGPU.crossCorrelate(aliInd, refInd, subarea, subXoffset, subYoffset);
}

DLL_EX_IM int fgpuProcessAlignImage(float *binArr, int stackInd, int groupInd, 
                                    int stackOnGpu)
{
  return sFGPU.processAlignImage(binArr, stackInd, groupInd, stackOnGpu);
}

DLL_EX_IM void fgpuNumberOfAlignFFTs(int *numBinPad, int *numGroups)
{
  sFGPU.numberOfAlignFFTs(numBinPad, numGroups);
}

DLL_EX_IM int fgpuReturnAlignFFTs(float **saved, float **groups, float *alignSum, 
                                  float *workArr)
{
  return sFGPU.returnAlignFFTs(saved, groups, alignSum, workArr);
}

DLL_EX_IM int fgpuReturnStackedFrame(float *array, int *frameNum)
{
  return sFGPU.returnStackedFrame(array, frameNum);
}

DLL_EX_IM void fgpuCleanSumItems()
{
  sFGPU.cleanSumItems();
}

DLL_EX_IM void fgpuCleanAlignItems()
{
  sFGPU.cleanAlignItems();
}

DLL_EX_IM void fgpuZeroTimers()
{
  sFGPU.zeroTimers();
}

DLL_EX_IM void fgpuPrintTimers()
{
  sFGPU.printTimers();
}

DLL_EX_IM int fgpuClearAlignSum()
{
  return sFGPU.clearAlignSum();
}

DLL_EX_IM int fgpuSumIntoGroup(int stackInd, int groupInd)
{
  return sFGPU.sumIntoGroup(stackInd, groupInd);
}

DLL_EX_IM void fgpuSetGroupSize(int inVal)
{
  sFGPU.setGroupSize(inVal);
}

// These two will be called only if this is a dll
DLL_EX_IM void fgpuSetPrintFunc(CharArgType func)
{
  utilSetPrintFunc(func);
}

DLL_EX_IM int fgpuGetVersion(void)
{
  return GPUFRAME_VERSION;
}

#if defined(_WIN32) && defined(DELAY_LOAD_FGPU)
extern "C"
BOOL WINAPI DllMain(HINSTANCE hInstance, DWORD dwReason, LPVOID /*lpReserved*/)
{
  if (dwReason == DLL_PROCESS_ATTACH) {

    // This disables notifications about threads
    DisableThreadLibraryCalls(hInstance);
  } else if (dwReason == DLL_PROCESS_DETACH) {
  }
  return TRUE;  
}
#endif
