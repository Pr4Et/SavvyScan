#include "hip/hip_runtime.h"
/*
 *  gpuctf.cu  -  GPU module for CTF correction by ctfphaseflip
 *
 *  Author: David Mastronarde
 *
 *  Copyright (C) 2018 by  the Regents of the University of
 *  Colorado.  See dist/COPYRIGHT for full copyright notice.
 *
 *  $Id: ctfphaseflip.cpp,v 4d4a2804e2b6 2018/09/25 02:59:33 mast $
 */
#include <stdarg.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "b3dutil.h"
#include "gpuctf.h"
#include "frameutil.h"
#include "cppdefs.h"

// Static functions
//#define USE_DUMPS
#ifdef USE_DUMPS
static void dumpFFT(float *fft, int nxPad, int nyPad, const char *descrip, int doReal);
static void dumpImage(float *image, int nxDim, int nxPad, int nyPad, const char *descrip);
#endif
static int testReportErr(const char *mess);
static void pflerr(const char *format, ...);
static void freeCudaArray(float **array);
static void destroyPlan(hipfftHandle &plan);

/*
 * Static variables in CPU or device
 */
static hipfftHandle sForwardPlan = 0;
static hipfftHandle sInversePlan = 0;
static float *sFullSlice = NULL;
static float *sFullOnDev = NULL;
static float *sFullPadImage = NULL;
static float *sFullCopy1 = NULL;
static float *sFullCopy2 = NULL;
static float *sStripArray1 = NULL;
static float *sStripArray2 = NULL;
static float *sOutputArray = NULL;
static bool sDoFullImages;
static int sNxSlice;
static int sNySlice;
static bool sNeedFullXform = false;
static int sStripXdim = 0;
static int sNxPad;
static int sNyPad;
static int sDebug = 0;
static double sWallCopy = 0.;
static double sWallInterp = 0.;
static double sWallPrep = 0.;
static double sWallFFT = 0.;
static double sWallCorrect = 0.;
static double sWallStart, sWallNow;

// Macros to get time only when needed
#define START_TIMER  if (sDebug) sWallStart = wallTime();
#define ADD_TIME(a) if (sDebug)                                         \
  {sWallNow = wallTime(); a += sWallNow - sWallStart; sWallStart = sWallNow;}

#if CUDA_VERSION < 4000
#define hipDeviceSynchronize hipDeviceSynchronize
#endif

/*
 * KERNELS
 */

/*
 * Kernel to taper and pad an image
 */
__global__ void taperInPadKernel(float *fullImage, int nxDimIn, int ixStart, int nxBox,
                                 int iyStart, int nyBox, float *outArr, int nxDimOut,
                                 int nx, int ny, int nxTaper, int nyTaper, float dmean)
{
  int padInX, padInY, outInd, imageX, imageY, fullInd, xInTaper, yInTaper;
  float value, atten;
  int ix = blockIdx.x * blockDim.x + threadIdx.x;
  int iy = blockIdx.y * blockDim.y + threadIdx.y;
  if (ix >= nx || iy >= ny)
    return;
  padInX = nx / 2 - nxBox / 2;
  padInY = ny / 2 - nyBox / 2;
  outInd = ix + iy * nxDimOut;
  imageX = ix - padInX;
  imageY = iy - padInY;
  
  if (imageX < 0 || imageX >= nxBox || imageY < 0 || imageY >= nyBox) {
    outArr[outInd] = dmean;
  } else {
    fullInd = imageX + ixStart + nxDimIn * (imageY + iyStart);
    value = fullImage[fullInd];
    xInTaper = min(imageX, (nxBox - 1) - imageX);
    yInTaper = min(imageY, (nyBox - 1) - imageY);
    if (xInTaper < nxTaper || yInTaper < nyTaper) {
      atten = min((xInTaper + 1.f) / (nxTaper + 1.f), (yInTaper + 1.f) / (nyTaper + 1.f));
      value = atten * (value - dmean) + dmean;
    }
    outArr[outInd] = value;
  }
}


/*
 * Kernel to correct the CTF
 */
__global__ void correctCtfKernel
(float *curStrip, int stripXdim, int ny, float freq_scalex, float freq_scaley,
 float pointDefocus, float cosAstig, float sinAstig, float focusSum, float focusDiff, 
 float cutonAngstroms, float phaseFracFactor, float phaseShift, float ampAngle, float C1,
 float C2, float scaleByPower, int powerIsHalf, int generalPower, float firstZeroFreqSq,
 float attenStartFrac, float minAttenFreqSq)
{
  float gx, gy, f2, denom, cosSum, phaseFrac = 1., waveAberration, attenFrac, ctf;
  int fx = blockIdx.x * blockDim.x + threadIdx.x;
  int fy = blockIdx.y * blockDim.y + threadIdx.y;
  int index;
  if (fx >= stripXdim / 2 || fy >= ny)
    return;
  index = fy * stripXdim + 2 * fx;
  if (fy > ny / 2)
    fy -= ny;
  gy = fy * freq_scaley;
  gx = fx * freq_scalex;
  f2 = gx * gx + gy * gy;
  if (focusDiff && (fx || fy)) {
    denom = sqrtf(f2);
    cosSum = (cosAstig * gx + sinAstig * gy) / denom;
    pointDefocus = focusSum + focusDiff * (2. * cosSum * cosSum - 1.);
  }
  if (cutonAngstroms > 0.)
    phaseFrac = phaseFracFactor * (1. - exp(-sqrtf(f2) / cutonAngstroms));
  waveAberration = (C2 * f2 - C1 * pointDefocus) * f2 - phaseFrac * phaseShift;

  // Produce a positive ctf for consistency and so it can be used for scaling
  // (Here is the formal equation before simplifying)
  /*ctf = -(sqrt(1 - ampContrast * ampContrast)) * sin(waveAberration)
    + ampContrast * cos(waveAberration);*/
  ctf = -sin(waveAberration - ampAngle);
  if (scaleByPower > 0. && f2 > minAttenFreqSq) {
    if (powerIsHalf)
      ctf = sqrt(fabs(ctf)) * (ctf >= 0. ? 1. : -1.);
    else if (generalPower) 
      ctf = pow(fabs(ctf), scaleByPower) * (ctf >= 0. ? 1. : -1.);
    if (f2 < firstZeroFreqSq) {
      attenFrac = (firstZeroFreqSq - f2) / 
        ((1. - attenStartFrac) * firstZeroFreqSq);
      ctf = attenFrac + (1. - attenFrac) * ctf;
    }

    // *= did not work here
    curStrip[index] = curStrip[index] * ctf;
    curStrip[index + 1] = curStrip[index + 1] * ctf;
  } else if (ctf < 0) {
    curStrip[index] = -curStrip[index];
    curStrip[index + 1] = -curStrip[index + 1];
  }
}

/*
 * Kernel to copy columns
 */
__global__ void copyColumnsKernel(float *curStrip, float *restoredArray, int nxDim,
                                  int nyFile, int stripXdim, int xoff, int yoff, 
                                  int startCol, int endCol)
{
  int column = blockIdx.x * blockDim.x + threadIdx.x + startCol;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (column <= endCol && row < nyFile)
    restoredArray[row * nxDim + column] =
      curStrip[(row + yoff) * stripXdim + column + xoff];
}

/*
 * Kernel to interpolate between two strips
 */
__global__ void interpolateKernel(float *curStrip, float *lastStrip, float *restoredArray,
                                  int nxDim, int nyFile, int stripXdim, int yoff, 
                                  int stripStride, int stripMid, int halfStrip,
                                  int curOffset, int lastOffset)
{
  int stripDist0, stripDist1;
  float curFrac, lastFrac;
  int startCol = stripMid - stripStride + 1;
  int column = blockIdx.x * blockDim.x + threadIdx.x + startCol;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (column <= stripMid && row < nyFile) {
    stripDist0 = column - stripMid + stripStride - 1;
    stripDist1 = stripMid + 1 - column;
    curFrac = stripDist0 / (float)stripStride;
    lastFrac = stripDist1 / (float)stripStride;
    restoredArray[row * nxDim + column] =
      curFrac * curStrip[(row + yoff) * stripXdim + curOffset + column] +
      lastFrac * lastStrip[(row + yoff) * stripXdim + lastOffset + column];
  }
}

/*
 * Kernel to copy along diagonals from a full image
 */
__global__ void copyDiagonalKernel
(float *curStrip, float *restoredArray, int nx, int nyFile, int stripXdim, int xoff,
 int yoff, float sinViewAxis, float cosViewAxis, float lowLim, float highLim)
{
  float axisDist;
  float xPixCenter = nx / 2.f - 0.5f;
  float yPixCenter = nyFile / 2.f - 0.5f;
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (column < nx && row < nyFile) {
    axisDist = -sinViewAxis * (column - xPixCenter) + cosViewAxis * (row - yPixCenter);
    if (axisDist >= lowLim && axisDist <= highLim) {
      restoredArray[row * nx + column] =
        curStrip[(row + yoff) * stripXdim + column + xoff];
    }
  }
}

/*
 * Kernel to interpolate along diagonals between two full images
 */
__global__ void interpDiagonalsKernel
(float *curStrip, float *lastStrip, float *restoredArray, int nx, int nyFile,
 int stripXdim, int xoff, int yoff, float stripStride, float sinViewAxis,
 float cosViewAxis, float lastAxisDist, float curAxisDist)
{
  float axisDist, curAxFrac;
  float xPixCenter = nx / 2.f - 0.5f;
  float yPixCenter = nyFile / 2.f - 0.5f;
  int column = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if (column < nx && row < nyFile) {
    axisDist = -sinViewAxis * (column - xPixCenter) + cosViewAxis * (row - yPixCenter);
    if (axisDist >= lastAxisDist + 0.5f && axisDist <= curAxisDist + 0.5f) {
      axisDist = max(lastAxisDist, min(curAxisDist, axisDist));
      curAxFrac = (min(axisDist, curAxisDist) - lastAxisDist) / stripStride;
      restoredArray[row * nx + column] = 
        curAxFrac * curStrip[(row + yoff) * stripXdim + xoff + column] +
        (1. - curAxFrac) * lastStrip[(row + yoff) * stripXdim + xoff + column];
    }
  }
}

/*
 * EXTERNALLY CALLED FUNCTTIONS
 */

/*
 * Test whether a GPU is available, either a GPU of the given number if nGPU is
 * > 0, or the one with the best processing rate if nGPU is 0, and return the
 * memory in bytes.  Return value is 1 for success, 0 for failure.
 */
int gpuAvailable(int nGPU, float *memory, int debug)
{
  int current_device = 0;
  int device_count = 0;
  int totalCores, max_gflops_device;
  float gflops;
  struct hipDeviceProp_t device_properties, best_properties;

  // The Mac mini comes through with a clock rate of 0 so allow a 0 product
  float max_gflops = -1.;
  sDebug = debug;
  *memory = 0;
  hipGetDeviceCount( &device_count );
  if (debug) {
#if CUDA_VERSION >= 3000
    int version, version2;
    hipRuntimeGetVersion(&version2);
    hipDriverGetVersion(&version);
    utilPrint("CUDA version - driver: %d.%02d  runtime: %d.%02d\n", version / 1000,
              version % 1000, version2 / 1000, version2 % 1000);
#endif
    utilPrint("Device count = %d\n", device_count);
  }
  if (nGPU != 0) {
    if (nGPU < 0 || nGPU > device_count) {
      utilPrint("The requested GPU number, %d, is out of range; there are only %d "
                "devices\n", nGPU, device_count);
      return 0;
    }
    current_device = nGPU - 1;
    device_count = nGPU;
  }
  for (; current_device < device_count; current_device++) {
    if (hipGetDeviceProperties( &device_properties, current_device)
        != hipSuccess) {
      pflerr("Error returned from trying to get properties of GPU device %d",
             current_device);
      return 0;
    }
    totalCores = totalCudaCores(device_properties.major, device_properties.minor,
                                device_properties.multiProcessorCount);
    if (debug) {
      utilPrint("Device %d (%s): mp %d cores %d  cr %d  major %d minor %d  mem %.0f",
                current_device, device_properties.name,
                device_properties.multiProcessorCount, totalCores,
                device_properties.clockRate, device_properties.major,
                device_properties.minor, (float)device_properties.totalGlobalMem);
#if CUDA_VERSION >= 4000
      utilPrint("  tex1d %d", device_properties.maxTexture1DLinear);
#endif
      utilPrint("\n");
    }
    gflops = totalCores * (float)device_properties.clockRate;

    // Exclude emulation mode (?) which shows up on the Mac
    if( gflops > max_gflops && device_properties.major != 9999) {
      max_gflops = gflops;
      max_gflops_device = current_device;
      best_properties = device_properties;
    }
  }
    
  if (max_gflops_device >= 0) {
    *memory = best_properties.totalGlobalMem;
    if (hipSetDevice(max_gflops_device) != hipSuccess) {
      pflerr("Error selecting GPU device %d", max_gflops_device + 1);
      return 0;
    }
    return 1;
  }
  return 0;
}

/*
 * Initialize operations for one slice, doing some one-time operations in the first call
 * i.e., allocate arrays and set up FFT plans
 */
int gpuInitializeSlice(float *sliceData, int nxFile, int nyFile, int stripXdim, int nxPad, 
                       int nyPad, bool doFullImages)
{
  int error;
  int sliceBytes = nxFile * nyFile * sizeof(float);
  int padBytes = stripXdim * nyPad * sizeof(float);
  sFullSlice = sliceData;
  sDoFullImages = doFullImages;
  sNxSlice = nxFile;
  sNySlice = nyFile;
  
  if (!sFullOnDev) {
    // First slice: allocate array
    if (hipMalloc((void **)&sFullOnDev, sliceBytes) != hipSuccess || 
        hipMalloc((void **)&sOutputArray, sliceBytes) != hipSuccess) {
      pflerr("Failed to allocate full slice array on GPU");
      return 1;
    }
    
    // If doing full slice, also allocate the padded array and its two copies here
    if (doFullImages) {
      if (hipMalloc((void **)&sFullPadImage, padBytes) != hipSuccess || 
          hipMalloc((void **)&sFullCopy1, padBytes) != hipSuccess || 
          hipMalloc((void **)&sFullCopy2, padBytes) != hipSuccess) {
        pflerr("Failed to allocate padded full slice arrays on GPU");
        return 1;
      }
    }
  }

  if (!doFullImages && stripXdim != sStripXdim) {
    freeCudaArray(&sStripArray1);
    freeCudaArray(&sStripArray2);
    if (hipMalloc((void **)&sStripArray1, padBytes) != hipSuccess || 
        hipMalloc((void **)&sStripArray2, padBytes) != hipSuccess) {
      pflerr("Failed to allocate strip arrays on GPU");
      return 1;
    }
  }

  // Make FFT plans once or whenever strip size changes
  if (stripXdim != sStripXdim) {
    destroyPlan(sForwardPlan);
    destroyPlan(sInversePlan);
    error = hipfftPlan2d(&sForwardPlan, nyPad, nxPad, HIPFFT_R2C);

    if (error == HIPFFT_SUCCESS)
      error = hipfftPlan2d(&sInversePlan, nyPad, nxPad, HIPFFT_C2R);
    if (error != HIPFFT_SUCCESS) {
      utilPrint("Failed to make plan for FFTs (error %d)\n", error);
      return 1;
    }
  }    

  // Copy full array to the GPU
  START_TIMER;
  if (hipMemcpy(sFullOnDev, sFullSlice, sNxSlice * sNySlice * sizeof(float), 
                 hipMemcpyHostToDevice) != hipSuccess) {
    pflerr("Failed to copy slice image to GPU array");
    return 1;
  }
  ADD_TIME(sWallCopy);

  sNeedFullXform = sDoFullImages;
  sStripXdim = stripXdim;
  sNxPad = nxPad;
  sNyPad = nyPad;
  return 0;
}

/*
 * Extract a strip between stripBegin and stripEnd with the given tapering and padded
 * as specified previously, and get the transform.  If doing full images, do this operation
 * only on the first strip and after that, copy the transform to the appropriate buffer
 */
int gpuExtractAndTransform(int stripInd, int stripBegin, int stripEnd, int nxTaper,
                           int nyTaper)
{
  float *curStrip;
  float dmean;
  int err;
  int blockX = 32;
  if (sDoFullImages)
    curStrip = sFullPadImage;
  else
    curStrip = (stripInd % 2) ? sStripArray2 : sStripArray1;
    
  // Need to taper-pad into curStrip for the strips or the full padded image
  if (!sDoFullImages || sNeedFullXform) {
    if (sDoFullImages) {
      stripBegin = 0;
      stripEnd = sNxSlice - 1;
    }
    dmean = sliceEdgeMean(sFullSlice, sNxSlice, stripBegin, stripEnd, 0, sNySlice - 1);
    START_TIMER;
    dim3 blockSize(blockX, 8, 1);
    dim3 gridSize((sNxPad + blockSize.x - 1) / blockSize.x,
                  (sNyPad + blockSize.y - 1) / blockSize.y, 1);
    taperInPadKernel<<<gridSize, blockSize>>>
      (sFullOnDev, sNxSlice, stripBegin, stripEnd + 1 - stripBegin, 0, sNySlice, curStrip,
       sStripXdim, sNxPad, sNyPad, nxTaper, nyTaper, dmean);
    if (testReportErr("to taper-pad image on GPU"))
      return 1;
    ADD_TIME(sWallPrep);
    if (stripInd < 5) {
      //dumpImage(curStrip, sStripXdim, sNxPad, sNyPad, "taperpad");
    }
    // And then take the FFT
    err = hipfftExecR2C(sForwardPlan, curStrip, (hipfftComplex *)curStrip);
    hipDeviceSynchronize();
    if (err != HIPFFT_SUCCESS) {
      utilPrint("Failure in forward FFT on GPU (CUFFT error %d)\n", err);
      return 1;
    }
    ADD_TIME(sWallFFT);
    sNeedFullXform = false;
    if (stripInd < 5) {
      //dumpFFT(curStrip, sNxPad, sNyPad, "taperpad-fft", 0);
    }
  }
  
  // If doing full image, now copy to current copy array
  if (sDoFullImages) {
    START_TIMER;
    if (hipMemcpy((stripInd % 2) ? sFullCopy2 : sFullCopy1, sFullPadImage,
                   sStripXdim * sNyPad * sizeof(float), hipMemcpyDeviceToDevice) != 
        hipSuccess) {
      pflerr("Failed to copy full padded FFT to GPU current copy array");
      return 1;
    }
    ADD_TIME(sWallCopy);
  }
  
  return 0;
}

/*
 * Correct the CTF in the current strip and back-transform it
 */
int gpuCorrectCTF(int stripInd, float freq_scalex, float freq_scaley, float pointDefocus,
                  float cosAstig, float sinAstig, float focusSum, float focusDiff,
                  float cutonAngstroms, float phaseFracFactor, float phaseShift, 
                  float ampAngle, float C1, float C2, float scaleByPower, bool powerIsHalf,
                  bool generalPower, float firstZeroFreqSq, float attenStartFrac,
                  float minAttenFreqSq)
{
  float *curStrip;
  int err;
  int blockX = 32;
  if (sDoFullImages)
    curStrip = (stripInd % 2) ? sFullCopy2 : sFullCopy1;
  else
    curStrip = (stripInd % 2) ? sStripArray2 : sStripArray1;

  // Correct the CTF in the current strip
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((sStripXdim / 2 + blockSize.x - 1) / blockSize.x,
                (sNyPad + blockSize.y - 1) / blockSize.y, 1);
  correctCtfKernel<<<gridSize, blockSize>>>
    (curStrip, sStripXdim, sNyPad, freq_scalex, freq_scaley, pointDefocus, cosAstig,
     sinAstig, focusSum, focusDiff, cutonAngstroms, phaseFracFactor, phaseShift, 
     ampAngle, C1, C2, scaleByPower, powerIsHalf ? 1 : 0, generalPower ? 1 : 0,
     firstZeroFreqSq, attenStartFrac, minAttenFreqSq);
  if (testReportErr("to apply CTF corrections on GPU"))
    return 1;
  ADD_TIME(sWallCorrect);
  if (stripInd < 5) {
    //dumpFFT(curStrip, sNxPad, sNyPad, "corr-fft", 0);
  }
    
  // And then take the inverse FFT
  err = hipfftExecC2R(sInversePlan, (hipfftComplex *)curStrip, curStrip);
  hipDeviceSynchronize();
  if (err != HIPFFT_SUCCESS) {
    utilPrint("Failure in inverse FFT on GPU (CUFFT error %d)\n", err);
    return 1;
  }
  ADD_TIME(sWallFFT);
  //if (stripInd < 5) {
  //dumpImage(curStrip, sStripXdim, sNxPad, sNyPad, "corr-img");
    //}

  return 0;
}

/*
 * Interpolate columns between the last and current strip from stripMid - stripStride + 1 
 * to stripMid, where the offsets are ADDED when getting the strip coordinates 
 */
int gpuInterpolateColumns(int stripInd, int yoff, int stripStride, int stripMid,
                          int halfStrip, int curOffset, int lastOffset)
{
  float *curStrip, *lastStrip;
  int blockX = 32;
  if (sDoFullImages) {
    curStrip = (stripInd % 2) ? sFullCopy2 : sFullCopy1;
    lastStrip = (stripInd % 2) ? sFullCopy1 : sFullCopy2;
  } else {
    curStrip = (stripInd % 2) ? sStripArray2 : sStripArray1;
    lastStrip = (stripInd % 2) ? sStripArray1 : sStripArray2;
  }
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((stripStride + blockSize.x - 1) / blockSize.x,
                (sNySlice + blockSize.y - 1) / blockSize.y, 1);
  interpolateKernel<<<gridSize, blockSize>>>
    (curStrip, lastStrip, sOutputArray, sNxSlice, sNySlice, sStripXdim, yoff,
     stripStride, stripMid, halfStrip, curOffset, lastOffset);
  ADD_TIME(sWallInterp);
  return testReportErr("to interpolate columns on GPU");
}

/*
 * Copy columns from the current strip between startCol and endCol to the output, where
 * the offsets are ADDED to output coordinates to get the strip coordinates 
 */
int gpuCopyColumns(int stripInd, int xoff, int yoff, int startCol, int endCol)
{
  float *curStrip;
  int blockX = 32;
  if (sDoFullImages)
    curStrip = (stripInd % 2) ? sFullCopy2 : sFullCopy1;
  else
    curStrip = (stripInd % 2) ? sStripArray2 : sStripArray1;

  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize(((endCol + 1 - startCol) + blockSize.x - 1) / blockSize.x,
                (sNySlice + blockSize.y - 1) / blockSize.y, 1);
  copyColumnsKernel<<<gridSize, blockSize>>>
    (curStrip, sOutputArray, sNxSlice, sNySlice, sStripXdim, xoff, yoff, startCol,
     endCol);
  ADD_TIME(sWallInterp);
  return testReportErr("to copy columns on GPU");
}

/*
 * Interpolate along diagonals
 */
int gpuInterpDiagonals(int stripInd, int xoff, int yoff, int stripStride, 
                       float sinViewAxis, float cosViewAxis, float lastAxisDist,
                       float curAxisDist)
{
  float *curStrip, *lastStrip;
  int blockX = 32;
  curStrip = (stripInd % 2) ? sFullCopy2 : sFullCopy1;
  lastStrip = (stripInd % 2) ? sFullCopy1 : sFullCopy2;
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((sNxSlice + blockSize.x - 1) / blockSize.x,
                (sNySlice + blockSize.y - 1) / blockSize.y, 1);
  interpDiagonalsKernel<<<gridSize, blockSize>>>
    (curStrip, lastStrip, sOutputArray, sNxSlice, sNySlice, sStripXdim, xoff, yoff, 
     (float)stripStride, sinViewAxis, cosViewAxis, lastAxisDist, curAxisDist);
  ADD_TIME(sWallInterp);
  return testReportErr("to interpolate diagonals on GPU");
}

/*
 * Copy along diagonals
 */
int gpuCopyDiagonals(int stripInd, int xoff, int yoff, float sinViewAxis, 
                     float cosViewAxis, float lowLim, float highLim)
{
  float *curStrip;
  int blockX = 32;
  curStrip = (stripInd % 2) ? sFullCopy2 : sFullCopy1;
  START_TIMER;
  dim3 blockSize(blockX, 8, 1);
  dim3 gridSize((sNxSlice + blockSize.x - 1) / blockSize.x,
                (sNySlice + blockSize.y - 1) / blockSize.y, 1);
  copyDiagonalKernel<<<gridSize, blockSize>>>
    (curStrip, sOutputArray, sNxSlice, sNySlice, sStripXdim, xoff, yoff, sinViewAxis,
     cosViewAxis, lowLim, highLim);
  ADD_TIME(sWallInterp);
  return testReportErr("to copy diagonals on GPU");
}

/*
 * Return the corrected image
 */
int gpuReturnImage(float *finalImage)
{
  START_TIMER;
  if (hipMemcpy(finalImage, sOutputArray, sNxSlice * sNySlice * sizeof(float),
                 hipMemcpyDeviceToHost) != hipSuccess) {
    pflerr("Failed to copy restored image back from GPU");
    return 1;
  }
  ADD_TIME(sWallCopy);
  for (int i = 0; i < sNxSlice * sNySlice; i++)
    finalImage[i] /= sNxPad * sNyPad;
  return 0;
}

/*
 * Return the diagnostic times
 */
void gpuGetTimes(double &copy, double &prep, double &FFT, double &correct, double &interp)
{
  copy = sWallCopy;
  prep = sWallPrep;
  FFT = sWallFFT;
  correct = sWallCorrect;
  interp = sWallInterp;
}

/*
 * UTILITIES
 */

/*
 * Cleanup functions when changing sizes
 */
static void freeCudaArray(float **array)
{
  if (*array)
    hipFree(*array);
  *array = NULL;
}

static void destroyPlan(hipfftHandle &plan)
{
  if (plan)
    hipfftDestroy(plan);
  plan = 0;
}

/*
 * Test for an error launching threads then synchronize, which flushes out other errors
 */
static int testReportErr(const char *mess)
{
  hipError_t err;
  err = hipGetLastError();
  if (err != hipSuccess) {
    utilPrint("Error executing threads %s: %s\n", mess,
              hipGetErrorString(err));
    return 1;
  }
  if (hipDeviceSynchronize() != hipSuccess) {
    pflerr("Error return from synchronizing after running kernel %s", mess);
    return 1;
  }
  return 0;
}

/*
 * In case of error, find the error string and print it with message
 */
static void pflerr(const char *format, ...)
{
  hipError_t err;
  char errorMess[512];
  va_list args;
  va_start(args, format);
  vsprintf(errorMess, format, args);
  err = hipGetLastError();
  utilPrint("%s: %s\n", errorMess, hipGetErrorString(err));
  va_end(args);
}

/*
 * Wrappers to the utility dump functions
 */
#ifdef USE_DUMPS
static void dumpFFT(float *fft, int nxPad, int nyPad, const char *descrip, int doReal)
{
  int sizeTmp = (nxPad + 2) * nyPad;
  float *temp = B3DMALLOC(float, sizeTmp);
  if (!temp)
    return;
  if (hipMemcpy(temp, fft, sizeTmp * sizeof(float), hipMemcpyDeviceToHost) ==
      hipSuccess)
    utilDumpFFT(temp, nxPad, nyPad, descrip, doReal, 0, 1);
  free(temp);
}

static void dumpImage(float *image, int nxDim, int nxPad, int nyPad, const char *descrip)
{
  int sizeTmp = nxDim * nyPad;
  float *temp = B3DMALLOC(float, sizeTmp);
  if (!temp)
    return;
  if (hipMemcpy(temp, image, sizeTmp * sizeof(float), hipMemcpyDeviceToHost) == 
      hipSuccess)
    utilDumpImage(temp, nxDim, nxPad, nyPad, 0, descrip);
  free(temp);
}
#endif

